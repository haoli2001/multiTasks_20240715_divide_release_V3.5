#include "hip/hip_runtime.h"
#include "virtualface_gpu.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include "handlerror.h"
#include "hip/device_functions.h"

#define TILE_DIM 16

Radius AxistoRadius_gpu(Axis a, Radius b)		//������ֱ������ϵ���������ת��
{
	Radius c;
	double st, fi;					//st=��,fi=��
	double pi = acos(-1.0);
	st = b.Yst * pi / 180;
	fi = b.Zfi * pi / 180;
	c.Xr = sin(st)*cos(fi)*a.x + sin(st)*sin(fi)*a.y + cos(st)*a.z;
	c.Yst = cos(st)*cos(fi)*a.x + cos(st)*sin(fi)*a.y - sin(st)*a.z;
	c.Zfi = -sin(fi)*a.x + cos(fi)*a.y;
	return c;
}

Axis dSphericaltoRectangular(Radius a)				//���꣬�����굽ֱ������ϵ��ת��
{
	double pi = acos(-1.0);
	Axis b;
	double st, fi;					//st=��,fi=��
	st = a.Yst * pi / 180;
	fi = a.Zfi * pi / 180;

	b.x = a.Xr*sin(st)*cos(fi);
	b.y = a.Xr*sin(st)*sin(fi);
	b.z = a.Xr*cos(st);

	return b;
}

__device__ Axis dSphericaltoRectangular_gpu(Radius a)				//���꣬�����굽ֱ������ϵ��ת��
{
	double pi = acos(-1.0);
	Axis b;
	double st, fi;					//st=��,fi=��
	st = a.Yst * pi / 180;
	fi = a.Zfi * pi / 180;

	b.x = a.Xr*sin(st)*cos(fi);
	b.y = a.Xr*sin(st)*sin(fi);
	b.z = a.Xr*cos(st);

	return b;
}

__global__ void dAxistoRadius_gpu(Radius* d_spherical, Element* d_point, Radius b, int num)		//������ֱ������ϵ���������ת��
{
	double pi = acos(-1.0);
	double st, fi;					//st=��,fi=��
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;
	if (idx < num)
	{
		st = b.Yst * pi / 180;
		fi = b.Zfi * pi / 180;

		d_spherical[idx].Xr = sin(st)*cos(fi)*d_point[idx].point[0] + sin(st)*sin(fi)*d_point[idx].point[1] + cos(st)*d_point[idx].point[2];
		d_spherical[idx].Yst = cos(st)*cos(fi)*d_point[idx].point[0] + cos(st)*sin(fi)*d_point[idx].point[1] - sin(st)*d_point[idx].point[2];
		d_spherical[idx].Zfi = -sin(fi)*d_point[idx].point[0] + cos(fi)*d_point[idx].point[1];
	}

}


__device__ Axis dRadiustoAxis_gpu(Radius a, Radius b)		//�����������굽ֱ������ϵ��ת��
{
	double pi = acos(-1.0);
	Axis c;
	double st, fi;					//st=��,fi=��
	st = b.Yst * pi / 180;
	fi = b.Zfi * pi / 180;
	c.x = sin(st)*cos(fi)*a.Xr + cos(st)*cos(fi)*a.Yst - sin(fi)*a.Zfi;
	c.y = sin(st)*sin(fi)*a.Xr + cos(st)*sin(fi)*a.Yst + cos(fi)*a.Zfi;
	c.z = cos(st)*a.Xr - sin(st)*a.Yst;
	return c;
}

__device__ Axis_slx dRadiustoAxis_gpu_slx(Radius a, Radius b)		//�����������굽ֱ������ϵ��ת��
{
	double pi = acos(-1.0);
	Axis_slx c;
	double st, fi;					//st=��,fi=��
	st = b.Yst * pi / 180;
	fi = b.Zfi * pi / 180;
	c.p[0] = sin(st)*cos(fi)*a.Xr + cos(st)*cos(fi)*a.Yst - sin(fi)*a.Zfi;
	c.p[1] = sin(st)*sin(fi)*a.Xr + cos(st)*sin(fi)*a.Yst + cos(fi)*a.Zfi;
	c.p[2] = cos(st)*a.Xr - sin(st)*a.Yst;
	return c;
}

__global__ void dcreate_virtualface_gpu(Radius direction_radius, Direction *rays, Square *squares, int height, int width, float e_st_min, float e_fi_max, float lmd)
{
	Radius p_cent, p_corner;

	//direction_p = dSphericaltoRectangular_gpu(direction);//等相位面的法向量
	p_cent.Xr = p_corner.Xr = direction_radius.Xr;

	int idx = blockIdx.x * TILE_DIM + threadIdx.x;
	int idy = blockIdx.y * TILE_DIM + threadIdx.y;
	int id = idy * width + idx;
	//每根声线管束的五个射线编号
	if (idx < width && idy < height)
	{
		squares[id].CornerRayIndex.x = idy * width + idy + idx + 1;
		squares[id].CornerRayIndex.y = idy * width + idy + idx;
		squares[id].CornerRayIndex.z = (idy + 1) * width + idy + idx + 1;
		squares[id].CornerRayIndex.w = (idy + 1) * width + idy + idx + 2;
		squares[id].CenterRayIndex = (width + 1) * (height + 1) + idy * width + idx;
		squares[id].right = false;
	}

	if (idx < width && idy < height)//声线管束的中心射线
	{
		p_cent.Yst = e_st_min + (idy + 0.5)*lmd;
		p_cent.Zfi = e_fi_max - (idx + 0.5)*lmd;
		Axis_slx p_cent_axis = dRadiustoAxis_gpu_slx(p_cent, direction_radius);
		squares[id].CenterRay.x = p_cent_axis.p[0];
		squares[id].CenterRay.y = p_cent_axis.p[1];
		squares[id].CenterRay.z = p_cent_axis.p[2];
	}

	id = idy * (width + 1) + idx;
	if (idx < width + 1 && idy < height + 1)//声线管束的四个角顶射线
	{
		p_corner.Yst = e_st_min + idy * lmd;
		p_corner.Zfi = e_fi_max - idx * lmd;
		Axis_slx p_corner_axis = dRadiustoAxis_gpu_slx(p_corner, direction_radius);
		rays[id].p.x = p_corner_axis.p[0];
		rays[id].p.y = p_corner_axis.p[1];
		rays[id].p.z = p_corner_axis.p[2];
	}
}

void create_virtualface_gpu(Direction *rays2, Square *squares2, int dwidth, int dheight, float lmd, Radius direction_radius,
	float e_st_min, float e_fi_max, int device_id)
{
	dim3 gridsize((dwidth + 1) / TILE_DIM + 1, (dheight + 1) / TILE_DIM + 1);
	dim3 blocksize(TILE_DIM, TILE_DIM);
	dcreate_virtualface_gpu << <gridsize, blocksize >> >(direction_radius, rays2, squares2, dheight, dwidth, e_st_min, e_fi_max, lmd);
	HANDLE_ERROR(hipGetLastError());
}

__global__ void dStFiExtreme(Radius* d_spherical, float* d_st_min, float* d_st_max, float* d_fi_min, float* d_fi_max, int nodeNum)
{
	__shared__ float st_min_data[512];
	__shared__ float st_max_data[512];
	__shared__ float fi_min_data[512];
	__shared__ float fi_max_data[512];

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;
	if (idx < nodeNum)
	{
		st_min_data[tid] = d_spherical[idx].Yst;
		st_max_data[tid] = d_spherical[idx].Yst;
		fi_min_data[tid] = d_spherical[idx].Zfi;
		fi_max_data[tid] = d_spherical[idx].Zfi;
	}
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s){
			st_min_data[tid] = (st_min_data[tid] < st_min_data[tid + s]) ? st_min_data[tid] : st_min_data[tid + s];
			st_max_data[tid] = (st_max_data[tid] > st_max_data[tid + s]) ? st_max_data[tid] : st_max_data[tid + s];
			fi_min_data[tid] = (fi_min_data[tid] < fi_min_data[tid + s]) ? fi_min_data[tid] : fi_min_data[tid + s];
			fi_max_data[tid] = (fi_max_data[tid] > fi_max_data[tid + s]) ? fi_max_data[tid] : fi_max_data[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		d_st_min[blockIdx.x] = st_min_data[0];
		d_st_max[blockIdx.x] = st_max_data[0];
		d_fi_min[blockIdx.x] = fi_min_data[0];
		d_fi_max[blockIdx.x] = fi_max_data[0];
	}
}

__global__ void dStFiExtreme2(float* d_st_min, float* d_st_max, float* d_fi_min, float* d_fi_max, int nodeNum)
{
	__shared__ float st_min_data[512];
	__shared__ float st_max_data[512];
	__shared__ float fi_min_data[512];
	__shared__ float fi_max_data[512];

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;
	if (idx < nodeNum)
	{
		st_min_data[tid] = d_st_min[idx];
		st_max_data[tid] = d_st_max[idx];
		fi_min_data[tid] = d_fi_min[idx];
		fi_max_data[tid] = d_fi_max[idx];
	}
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s){
			st_min_data[tid] = (st_min_data[tid] < st_min_data[tid + s]) ? st_min_data[tid] : st_min_data[tid + s];
			st_max_data[tid] = (st_max_data[tid] > st_max_data[tid + s]) ? st_max_data[tid] : st_max_data[tid + s];
			fi_min_data[tid] = (fi_min_data[tid] < fi_min_data[tid + s]) ? fi_min_data[tid] : fi_min_data[tid + s];
			fi_max_data[tid] = (fi_max_data[tid] > fi_max_data[tid + s]) ? fi_max_data[tid] : fi_max_data[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		d_st_min[blockIdx.x] = st_min_data[0];
		d_st_max[blockIdx.x] = st_max_data[0];
		d_fi_min[blockIdx.x] = fi_min_data[0];
		d_fi_max[blockIdx.x] = fi_max_data[0];
	}
}
void getWidthHeight(float far_dis,Element *points, int num, int *width_calc, int *height_calc,int st, float fi, float lmd_calc, int *dheight_calc, float *de_st_min, float *de_fi_max, int device_num, int* divided_num, int max_pipeline_capicity)
{
	Radius direction = { far_dis, st, fi };	//射线表示用角度（r，st，fi）  θ =st，和z轴的夹角；为从正z轴来看自x轴按逆时针方向转到OM所转过的角  // snw 不能接立即数
	
	Radius* d_direction;
	HANDLE_ERROR(hipMalloc((void**)&d_direction, sizeof(Radius)));
	HANDLE_ERROR(hipMemcpy(d_direction, &direction, sizeof(Radius), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipDeviceSynchronize());

	dim3 threadSize(512, 1, 1);
	dim3 blockSize(num / 512 + 1, 1, 1);

	int numItem = 0;
	Element* d_point;
	HANDLE_ERROR(hipMalloc((void**)&d_point, num * sizeof(Element)));
	//HANDLE_ERROR(hipMemset(d_point, 0, num * sizeof(Element)));
	HANDLE_ERROR(hipMemcpy(d_point, points, num * sizeof(Element), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipDeviceSynchronize());

	Radius* d_spherical;//读取数据时，球面坐标系矢量
	HANDLE_ERROR(hipMalloc((void**)&d_spherical, num * sizeof(Radius)));
	HANDLE_ERROR(hipMemset(d_spherical, 0, num * sizeof(Radius)));

	//将模型各点的坐标转换到球坐标系
	dAxistoRadius_gpu << <blockSize, threadSize >> >(d_spherical, d_point, direction, num);
	HANDLE_ERROR(hipGetLastError());
	HANDLE_ERROR(hipDeviceSynchronize());

	float* d_st_min;
	float* d_fi_max;
	float* d_st_max;
	float* d_fi_min;
	HANDLE_ERROR(hipMalloc((void**)&d_st_min, blockSize.x * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_fi_max, blockSize.x * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_st_max, blockSize.x * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_fi_min, blockSize.x * sizeof(float)));
	HANDLE_ERROR(hipMemset(d_st_min, 0, blockSize.x * sizeof(float)));
	HANDLE_ERROR(hipMemset(d_fi_max, 0, blockSize.x * sizeof(float)));
	HANDLE_ERROR(hipMemset(d_st_max, 0, blockSize.x * sizeof(float)));
	HANDLE_ERROR(hipMemset(d_fi_min, 0, blockSize.x * sizeof(float)));

	//两级归约求st,fi的最大最小值
	dStFiExtreme << <blockSize, threadSize >> >(d_spherical, d_st_min, d_st_max, d_fi_min, d_fi_max, num);
	HANDLE_ERROR(hipGetLastError());
	HANDLE_ERROR(hipDeviceSynchronize());
	numItem = blockSize.x;
	blockSize.x = numItem / threadSize.x + 1;
	while (numItem > 1)
	{
		dStFiExtreme2 << <blockSize, threadSize >> >(d_st_min, d_st_max, d_fi_min, d_fi_max, numItem);
		hipDeviceSynchronize();
		numItem = blockSize.x;
		blockSize.x = numItem / threadSize.x + 1;
	}

	//读取三角面元的位置信息，转化到虚拟孔径面找到最值
	float e_st_min, e_st_max, e_fi_min, e_fi_max;
	HANDLE_ERROR(hipMemcpy(&e_st_min, d_st_min, sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(&e_st_max, d_st_max, sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(&e_fi_min, d_fi_min, sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(&e_fi_max, d_fi_max, sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipDeviceSynchronize());
	*width_calc = (e_fi_max - e_fi_min + lmd_calc - 0.001) / lmd_calc;
	*height_calc = (e_st_max - e_st_min + lmd_calc - 0.001) / lmd_calc;
	*dheight_calc = (*height_calc + device_num - 0.001) / device_num;
	//printf("e_fi_max:%f, e_fi_min:%f",e_fi_max, e_fi_min);
	//printf("e_st_max:%f, e_st_min:%f",e_st_max, e_st_min);
	*de_st_min = e_st_min;
	*de_fi_max = e_fi_max;

	*divided_num = ceil((float)*width_calc * (float)*height_calc / (float)max_pipeline_capicity);
	

	hipFree(d_direction);
	hipFree(d_point);
	hipFree(d_spherical);
	hipFree(d_st_min);
	hipFree(d_fi_max);
	hipFree(d_st_max);
	hipFree(d_fi_min);
}

void getWidthHeight_Sun(float far_dis,Box b, int *width, int *height, int st, float fi, float lmd, int *dheight, float *de_st_min, float *de_fi_max, int device_num)
{
	float  distance = far_dis;//snw
	float p[3];
	float dir[3];
	p[2] = distance*cos(st*PI/180);
	p[1] = distance*sin(st*PI / 180)*cos(fi*PI / 180);
	p[0] = distance*sin(st*PI / 180)*sin(fi*PI / 180);

	float s = sqrt(p[0] * p[0] + p[1] * p[1] + p[2] * p[2]);
	dir[0] = -p[0] / s;
	dir[1] = -p[1] / s;
	dir[2] = -p[2] / s;
	float dirx[3], diry[3];
	dirx[0] = dir[1] / (sqrt(dir[0] * dir[0] + dir[1] * dir[1]));
	dirx[1] = -dir[0] / (sqrt(dir[0] * dir[0] + dir[1] * dir[1]));
	dirx[2] = 0;

	diry[0] = dirx[1] * dir[2] - dirx[2] * dir[1];
	diry[1] = dirx[2] * dir[0] - dirx[0] * dir[2];
	diry[2] = dirx[0] * dir[1] - dirx[1] * dir[0];


	float D = -(dir[0] * p[0]) - (dir[1] * p[1]) - (dir[2] * p[2]);

	float point[8][3];
	point[0][0] = b.bmin[0]; point[0][1] = b.bmin[1], point[0][2] = b.bmin[2];
	point[1][0] = b.bmax[0]; point[1][1] = b.bmin[1], point[1][2] = b.bmin[2];
	point[2][0] = b.bmin[0]; point[2][1] = b.bmax[1], point[2][2] = b.bmin[2];
	point[3][0] = b.bmin[0]; point[3][1] = b.bmin[1], point[3][2] = b.bmax[2];
	point[4][0] = b.bmax[0]; point[4][1] = b.bmax[1], point[4][2] = b.bmin[2];
	point[5][0] = b.bmax[0]; point[5][1] = b.bmin[1], point[5][2] = b.bmax[2];
	point[6][0] = b.bmin[0]; point[6][1] = b.bmax[1], point[6][2] = b.bmax[2];
	point[7][0] = b.bmax[0]; point[7][1] = b.bmax[1], point[7][2] = b.bmax[2];

	float xmin=999999999, xmax=-999999999, ymin=9999999999, ymax=-9999999999;
	for (int i = 0; i < 8; i++)
	{
		float t = (dir[0] * point[i][0] + dir[1] * point[i][1] + dir[2] * point[i][2] + D)
			/ (dir[0] * dir[0] + dir[1] * dir[1] + dir[2] * dir[2]);
		float dir_t[3];
		float x = point[i][0] - t*dir[0];
		float y = point[i][1] - t*dir[1];
		float z = point[i][2] - t*dir[2];

		dir_t[0] = x - p[0];
		dir_t[1] = y - p[1];
		dir_t[2] = z - p[2];

		float yy = dir_t[2] / diry[2];
		float xx = (dir_t[0] - y*diry[0]) / dirx[0];
		if (xx > xmax)
			xmax = xx;
		if (xx < xmin)
			xmin = xx;
		if (yy > ymax)
			ymax = yy;
		if (yy < ymin)
			ymin = yy;
	}
	*height = (ymax - ymin) / lmd;
	*width = (xmax - xmin) / lmd;
	*dheight = (*height + device_num - 1) / device_num;
	*de_st_min = ymin;
	*de_fi_max = xmax;

	return;
}

void ConstructCore(DynamicPlane *node, float e_st_min, float e_fi_max, int width, int height)
{
	node->st_min = e_st_min;
	node->fi_max = e_fi_max;
	node->height = height;
	node->width = width;
}

void ConstructNodeLeft(DynamicPlane parent, DynamicPlane* child, BinaryTimeTree* pre_angle_time, float lmd)
{
	if (parent.width >= parent.height)
	{
		child->flag = 0; // 列向划分

		int width;
		if (pre_angle_time->runtime == 0 || pre_angle_time->flag == 1)
		{
			// 第一度仿真或与前一度划分方向不一样时
			child->GPUnum = ceil(parent.GPUnum / 2); // ceil:返回大于等于它的最小整数
			child->ratio = ceil(parent.GPUnum / 2) / parent.GPUnum;
			width = ceil(parent.width * child->ratio);
		}
		else
		{
			int deltaW = (int)(0.5 * (pre_angle_time->rightchild->runtime * pre_angle_time->leftchild->GPUnum - pre_angle_time->leftchild->runtime * pre_angle_time->rightchild->GPUnum) / \
				(pre_angle_time->AveTimeLeft * pre_angle_time->rightchild->GPUnum + pre_angle_time->AveTimeRight * pre_angle_time->leftchild->GPUnum));
			width = (int)(parent.width * pre_angle_time->ratio) + deltaW;
			child->ratio = (float)width / parent.width;
		}
		float fi_max = parent.fi_max - (parent.width - width) * lmd;
		ConstructCore(child, parent.st_min, fi_max, width, parent.height);
	}
	else
	{
		child->flag = 1; // 行向划分

		int height;
		if (pre_angle_time->runtime == 0 || pre_angle_time->flag == 0)
		{
			// 第一度仿真或与前一度划分方向不一样时
			child->GPUnum = ceil(parent.GPUnum / 2); // ceil:返回大于等于它的最小整数
			child->ratio = ceil(parent.GPUnum / 2) / parent.GPUnum;
			height = ceil(parent.height * child->ratio);
		}
		else
		{
			int deltaH = (int)(0.5 * (pre_angle_time->rightchild->runtime * pre_angle_time->leftchild->GPUnum - pre_angle_time->leftchild->runtime * pre_angle_time->rightchild->GPUnum) / \
				(pre_angle_time->AveTimeLeft * pre_angle_time->rightchild->GPUnum + pre_angle_time->AveTimeRight * pre_angle_time->leftchild->GPUnum));
			height = (int)(parent.height*pre_angle_time->ratio) + deltaH;
			child->ratio = (float)height / parent.height;
		}
		float st_min = parent.st_min + (parent.height - height) * lmd;
		ConstructCore(child, st_min, parent.fi_max, parent.width, height);
	}
}

void ConstructNodeRight(DynamicPlane parent, DynamicPlane* child, BinaryTimeTree* pre_angle_time)
{
	if (parent.width >= parent.height)
	{
		child->flag = 0; // 按列

		int width;
		if (pre_angle_time->runtime == 0 || pre_angle_time->flag == 1)
		{
			// 第一度仿真或与前一度划分方向不一样时
			child->GPUnum = floor(parent.GPUnum / 2); // floor:返回小于等于它的最小整数
			width = parent.width - ceil(parent.width * ceil(parent.GPUnum / 2) / parent.GPUnum);
		}
		else
		{
			int deltaW = (int)(0.5 * (pre_angle_time->leftchild->runtime * pre_angle_time->rightchild->GPUnum - pre_angle_time->rightchild->runtime * pre_angle_time->leftchild->GPUnum) / \
				(pre_angle_time->AveTimeLeft * pre_angle_time->rightchild->GPUnum + pre_angle_time->AveTimeRight * pre_angle_time->leftchild->GPUnum));
			width = parent.width - (int)(parent.width * pre_angle_time->ratio) + deltaW;
		}
		ConstructCore(child, parent.st_min, parent.fi_max, width, parent.height);
	}
	else
	{
		child->flag = 1; // 按行

		int height;
		if (pre_angle_time->runtime == 0 || pre_angle_time->flag == 0)
		{
			child->GPUnum = floor(parent.GPUnum / 2); // floor:返回小于等于它的最小整数
			height = parent.height - ceil(parent.height * ceil(parent.GPUnum / 2) / parent.GPUnum);
		}
		else
		{
			int deltaH = (int)(0.5 * (pre_angle_time->leftchild->runtime * pre_angle_time->rightchild->GPUnum - pre_angle_time->rightchild->runtime * pre_angle_time->leftchild->GPUnum) / \
				(pre_angle_time->AveTimeLeft * pre_angle_time->rightchild->GPUnum + pre_angle_time->AveTimeRight * pre_angle_time->leftchild->GPUnum));
			height = parent.height - (int)(parent.height * pre_angle_time->ratio) + deltaH;
		}
		ConstructCore(child, parent.st_min, parent.fi_max, parent.width, height);
	}
}

/**************************
名称：DynamicPlane* ConstructVirtualFace()
描述：动态生成子孔径面
参数：BinaryTimeTree** pre_angle_time:前一角度的计算时间; int DeviceCount:GPU数量; float e_st_min, float e_fi_max, int width, int height:虚拟孔径面信息; float lmd:划分步长
返回值：DynamicPlane* dData:各GPU卡上的子孔径面边界信息
***************************/
void ConstructVirtualFace(DynamicPlane* array, DynamicPlane* dData, BinaryTimeTree** pre_angle_time, int DeviceCount, float e_st_min, float e_fi_max, int width, int height, float lmd)
{
	int NodeNum = 2 * DeviceCount - 1;

	//DynamicPlane* array = (DynamicPlane*)malloc(NodeNum * sizeof(DynamicPlane));
	int index = 1;

	while (index <= NodeNum)
	{
		if (index == 1) // 创建根节点
		{
			array[index - 1].st_min = e_st_min;
			array[index - 1].fi_max = e_fi_max;
			array[index - 1].height = height;
			array[index - 1].width = width;
			array[index - 1].GPUnum = DeviceCount;
			
			if (index >= DeviceCount && index <= NodeNum) // 单卡
			{
				dData[index - DeviceCount] = array[index - 1];
			}
			index++;
			continue;
		}

		if (index % 2 == 0) // 左子节点
		{
			ConstructNodeLeft(array[index / 2 - 1], &array[index - 1], pre_angle_time[index / 2 - 1], lmd);
		}
		else // 右子节点
		{
			ConstructNodeRight(array[index / 2 - 1], &array[index - 1], pre_angle_time[index / 2 - 1]);
		}

		if (index >= DeviceCount && index <= NodeNum)
		{
			dData[index - DeviceCount] = array[index - 1];
		}
		index++;
	}
}

/**************************
名称：BinaryTimeTree** ConstructTimeTree()
描述：将各卡的计算时间存储成二叉树结构
参数：DynamicPlane* plane:子孔径面信息; float *runtime:各卡计算时间; int DeviceCount:GPU数量
返回值：BinaryTimeTree** timetree:前一角度的各卡计算时间
***************************/
BinaryTimeTree** ConstructTimeTree(DynamicPlane* plane, float *runtime, int DeviceCount)
{
	int NodeNum = 2 * DeviceCount - 1;
	BinaryTimeTree** timetree = (BinaryTimeTree**)malloc(NodeNum * sizeof(BinaryTimeTree*));
	int i = 1;
	while (i <= NodeNum)
	{
		if (i == 1)
		{
			BinaryTimeTree *root = (BinaryTimeTree*)malloc(sizeof(BinaryTimeTree));
			root->runtime = 0;
			timetree[i - 1] = root;
			i++;
			continue;
		}
		timetree[i - 1] = (BinaryTimeTree*)malloc(sizeof(BinaryTimeTree));
		BinaryTimeTree *temp = new BinaryTimeTree();
		if (i % 2 == 0)
			(timetree[i / 2 - 1])->leftchild = temp;
		else
			(timetree[i / 2 - 1])->rightchild = temp;
		timetree[i - 1] = temp;

		if (i >= DeviceCount && i <= NodeNum)
		{
			(timetree[i - 1])->runtime = runtime[i - DeviceCount];
			(timetree[i - 1])->GPUnum = 1;
		}
		i++;
	}
	i = NodeNum - 1;
	while (i > 0)
	{
		(timetree[i / 2 - 1])->flag = plane[i].flag;
		(timetree[i / 2 - 1])->runtime = (timetree[i])->runtime + (timetree[i - 1])->runtime;
		(timetree[i / 2 - 1])->ratio = plane[i - 1].ratio;
		(timetree[i / 2 - 1])->GPUnum = (timetree[i])->GPUnum + (timetree[i - 1])->GPUnum;
		(timetree[i / 2 - 1])->AveTimeRight = (timetree[i])->runtime / (plane[i].flag == 0 ? plane[i].width : plane[i].height);
		(timetree[i / 2 - 1])->AveTimeLeft = (timetree[i - 1])->runtime / (plane[i - 1].flag == 0 ? plane[i - 1].width : plane[i - 1].height);
		i -= 2;
	}
	return timetree;
}