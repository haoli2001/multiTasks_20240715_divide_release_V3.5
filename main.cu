#include <stdio.h>
#include <pthread.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <semaphore.h>
#include <pthread.h>
#ifdef linux
#include <sys/types.h>
#include <netinet/in.h>
#include <sys/socket.h>
#include <sys/wait.h>
#include <unistd.h>
#include <netinet/tcp.h>

#endif
#ifdef _UNIX
#include <sys/types.h>
#include <netinet/in.h>
#include <sys/socket.h>
#include <sys/wait.h>
#include <unistd.h>
#include <netinet/tcp.h>
#endif
#ifdef __WINDOWS_
#include <winsock2.h>

#endif
#ifdef _WIN32
#include <winsock2.h>

#endif


#pragma comment(lib, "Ws2_32.lib")

#include "socketFunctions.h"
#include "common_struct.h"
#include "recvThreadFunction.h"
#include "GPUWatchThreadFun.h"

#define PORT 3490  //����Ĭ�϶˿�Ϊ3490�˿�


//������socket�����ӵĿͻ���socket 
int socketSevice;


int main()
{
   
	pthread_t recvThread;       //�����̣߳�
	pthread_t GPUWatchThread;   //GPU �����߳�
	int socketClient[10];       //�ͻ���socket
	int curIndex = 0;           //��ǰ���ӵ�id
    
    
	//socket�ĳ�ʼ�����󶨣������Լ��ȴ�����
	init_socket();
    //�����׽���
	socketSevice = create_socket();
    //���������׽��ְ󶨵�ָ���˿�
	if (-1 == bind_listen(socketSevice, PORT))
	{ 
		printf("bind&listen wrong!\n");
		return 0;
	}
	
    //ѭ������������Ϣ
    while(true) 
    {
        //�������ӣ�������������accept_client �������أ���������
        socketClient[curIndex] = accept_client(socketSevice);
        
        int deviceCount;
        //��ȡ������GPU�豸��
        hipGetDeviceCount(&deviceCount); 
        for (int dev = 0; dev < deviceCount; dev++)
        {
            hipSetDevice(dev);
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, dev);
            Frame frame;
            DeviceInfo gpuInfo;
            strcpy(frame.command, "DeviceInfo");
            gpuInfo.deviceID = dev;
            gpuInfo.deviceCount = deviceCount;
            strcpy(gpuInfo.deviceName, deviceProp.name);
            //��ȡ�豸ÿ���ദ�����ĺ�������
            gpuInfo.coresPreMutiprocess = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor); 
            //�ദ�������� 
            gpuInfo.mutiprocessCount = deviceProp.multiProcessorCount;          
            frame.length = sizeof(DeviceInfo);
            memcpy(frame.data, (char*)&gpuInfo, sizeof(gpuInfo));
            //��ͻ��˷����豸��Ϣ
            send_frame(socketClient[curIndex], (char*)&frame, sizeof(Frame));     
        }
        pthread_create(&recvThread, NULL, recvThreadFunction, (void*)&(socketClient[curIndex]));    //�����������ݵ��߳�
        pthread_create(&GPUWatchThread,NULL , GPUWatchThreadFun, (void*)&(socketClient[curIndex]));  //����GPUʵʱ��ص��߳�
        curIndex++;
    }
	
    //����������ر�socket
#ifdef linux
	for(int i = curIndex-1;i>=0;i--)
	{	
		shutdown(socketClient[i], SHUT_RDWR);
		close(socketClient[i]);
	}
	shutdown(socketSevice, SHUT_RDWR);
	close(socketSevice);
#endif
#ifdef _UNIX
	for(int i = curIndex-1;i>=0;i--)
	{	
		shutdown(socketClient[i], SHUT_RDWR);
		close(socketClient[i]);
	}
	shutdown(socketClient, SHUT_RDWR);
	shutdown(socketSevice, SHUT_RDWR);
	close(socketClient);
	close(socketSevice);
#endif
#ifdef __WINDOWS_
	for(int i = curIndex-1;i>=0;i--)
	{	
		close(socketClient[i]);
	}
	close(socketSevice);
#endif
#ifdef _WIN32
	for(int i = curIndex-1;i>=0;i--)
	{	
		close(socketClient[i]);
	}
	closesocket(socketSevice);
#endif
	return 0;
}
