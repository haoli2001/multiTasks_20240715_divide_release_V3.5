
#include <hip/hip_runtime.h>

//矩阵相乘cpu版本
//输入：x,y相乘的两矩阵，一维数组。	rowNum，colNum分别为两矩阵的维数。 result为矩阵相乘结果，一维数组
//返回值：-1失败   1成功
int martixMulti(float* x,int rowNum_X,int colNum_X,float* y,int rowNum_Y,int colNum_Y,float* result)
{
	if(colNum_X != rowNum_Y)
		return -1;
	
	memset(result,0,sizeof(float) * rowNum_X*colNum_Y);

	for(int i=0;i<rowNum_X;i++)
	{
		for(int j=0;j<colNum_Y;j++)
		{
			for(int idx=0;idx<colNum_X;idx++)
			{
				result[i*colNum_Y +j] += x[i*colNum_X + idx] * y[idx*colNum_Y + j];
			}
		}
	}
	return 1;
}
