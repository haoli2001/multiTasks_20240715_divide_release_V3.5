#include "GPUWatchThreadFun.h"

//ʹ��NVIDIA�ٷ��ṩ��nvml���ȡGPU״̬��Ϣ
#include <nvml.h>
#include "common_struct.h"
#include <memory.h>
#include <malloc.h>
#include <string.h>
#include <unistd.h>
#include "socketFunctions.h"

#ifdef __WINDOWS_
#include <Windows.h>
#endif
#ifdef _WIN32
#include <Windows.h>
#endif



void *GPUWatchThreadFun(void *argv)
{
	int socketClient = *(int*)argv;
	nvmlInit();
	unsigned int deviceCount;

	/*@return
		*-\ref NVML_SUCCESS                 if \a deviceCount has been set
		*         -\ref NVML_ERROR_UNINITIALIZED     if the library has not been successfully initialized
		*         -\ref NVML_ERROR_INVALID_ARGUMENT  if \a deviceCount is NULL
		*         -\ref NVML_ERROR_UNKNOWN           on any unexpected error
	*/
	if (NVML_ERROR_UNINITIALIZED == nvmlDeviceGetCount(&deviceCount))
	{
		printf("the library has not been successfully initialized");
		return NULL;
	}
	if (NVML_ERROR_INVALID_ARGUMENT == nvmlDeviceGetCount(&deviceCount))
	{
		printf("\a deviceCount is NULL");
		return NULL;
	}
	if (NVML_ERROR_UNKNOWN == nvmlDeviceGetCount(&deviceCount))
	{
		printf("on any unexpected error");
		return NULL;
	}
	nvmlDevice_t *device = (nvmlDevice_t*)malloc(sizeof(nvmlDevice_t)*deviceCount);

	nvmlUtilization_t *utilization = (nvmlUtilization_t*)malloc(sizeof(nvmlUtilization_t)*deviceCount);

	while (true)
	{
		GPUWatchStruct *gpuwatchstruct = (GPUWatchStruct*)malloc(sizeof(GPUWatchStruct)*deviceCount);
		for (int i = 0; i < deviceCount; i++)
		{
			nvmlDeviceGetHandleByIndex(i, &device[i]);
			nvmlDeviceGetUtilizationRates(device[i], &utilization[i]);
			gpuwatchstruct[i].device_id = i;
			gpuwatchstruct[i].gpu = utilization[i].gpu;
			gpuwatchstruct[i].memory = utilization[i].memory;
			nvmlDeviceGetTemperatureThreshold(device[i], NVML_TEMPERATURE_THRESHOLD_SHUTDOWN, (unsigned int*)&gpuwatchstruct[i].shutdown_temp);
			nvmlDeviceGetTemperatureThreshold(device[i], NVML_TEMPERATURE_THRESHOLD_SLOWDOWN, (unsigned int*)&gpuwatchstruct[i].slowdown_temp);
			nvmlDeviceGetTemperature(device[i], NVML_TEMPERATURE_GPU, (unsigned int*)&gpuwatchstruct[i].temp);
			nvmlMemory_t memory;
			nvmlDeviceGetMemoryInfo(device[i], &memory);
			gpuwatchstruct[i].total = memory.total;
			gpuwatchstruct[i].used = memory.used;
			gpuwatchstruct[i].free = memory.free;
		}
		Frame frame;
		
		for (int i = 0; i < deviceCount; i++)
		{
			strcpy(frame.command, "GPUWatch");
			frame.length = sizeof(GPUWatchStruct);
			memcpy(frame.data, (char*)&gpuwatchstruct[i], sizeof(GPUWatchStruct));
			//send_frame(socketClient, (char*)&frame, sizeof(Frame));
		}
	
        //ÿ��һ���ȡһ����Ϣ
#ifdef linux
		sleep(1);
#endif
#ifdef _UNIX
		sleep(1);
#endif
#ifdef __WINDOWS_
		Sleep(1000);
#endif
#ifdef _WIN32
		Sleep(1000);
#endif
	}
}

