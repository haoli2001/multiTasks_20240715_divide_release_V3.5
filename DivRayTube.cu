#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<>
#include<hip/device_functions.h>
#include "common_struct.h"
#include "virtualface_gpu.h"
#include "handlerror.h"
#include<stdio.h>


/*__global__ void CopyRayTube(Square *d_squares, int RayTubeNum, Square *d_squares2, Direction* d_rays2, Direction* d_rays, int* d_squares_pred, int* d_DivRayTubeNum, int RayTubeNumNew)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	//ԭ���߹����е����ߣ�����Ҫ����׷��
	if (id < RayTubeNum && d_squares[id].IsDivRayTube == true)
	{
		int index = 4 * (d_DivRayTubeNum[blockIdx.x] + d_squares_pred[id]);

		int4 corner_ray_idx = d_squares[id].CornerRayIndex;
		int center_ray_idx = d_squares[id].CenterRayIndex;
		
		Direction ray0 = d_rays[corner_ray_idx.x];
		Direction ray1 = d_rays[corner_ray_idx.y];
		Direction ray2 = d_rays[corner_ray_idx.z];
		Direction ray3 = d_rays[corner_ray_idx.w];
		Direction ray4 = d_rays[center_ray_idx];
		
		d_rays2[index + RayTubeNumNew] = ray0;
		d_rays2[index + 1 + RayTubeNumNew] = ray1;
		d_rays2[index + 2 + RayTubeNumNew] = ray2;
		d_rays2[index + 3 + RayTubeNumNew] = ray3;
		d_rays2[index/4 + RayTubeNumNew*2] = ray4;
	}
}*/

__global__ void DivRayTubeCenter(Square *d_squares2, Direction* d_rays2, int RayTubeNumNew, int DivRayTubeNum, Axis direction)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = id + RayTubeNumNew * 2 + DivRayTubeNum;

	//ÿ�����߹����������ߵ�����
	if (id < RayTubeNumNew)
	{
		//d_rays2[idx].caled = false;//20190427
		//d_rays2[idx].flag = false;
		//d_rays2[idx].distance = 0;
		//d_rays2[idx].times = 0;
		//d_rays2[idx].triangle_index = -1;
		//d_rays2[idx].dir[0] = -direction.x;
		//d_rays2[idx].dir[1] = -direction.y;
		//d_rays2[idx].dir[2] = -direction.z;

		int4 cr = d_squares2[id].CornerRayIndex;

		if ((id % 2) == 0)
		{
			float3 ray1 = d_rays2[cr.y].p;
			float3 ray3 = d_rays2[cr.w].p;

			d_squares2[id].CenterRay.x = 0.5 * (ray1.x + ray3.x);
			d_squares2[id].CenterRay.y = 0.5 * (ray1.y + ray3.y);
			d_squares2[id].CenterRay.z = 0.5 * (ray1.z + ray3.z);
		}
		if ((id % 2) == 1)
		{
			float3 ray0 = d_rays2[cr.x].p;
			float3 ray2 = d_rays2[cr.z].p;

			d_squares2[id].CenterRay.x = 0.5 * (ray0.x + ray2.x);
			d_squares2[id].CenterRay.y = 0.5 * (ray0.y + ray2.y);
			d_squares2[id].CenterRay.z = 0.5 * (ray0.z + ray2.z);
		}
	}
}

__global__ void CreateRayTubeInfo(Square *d_squares, Direction* d_rays, int RayTubeNum, Direction* d_rays2, int* d_squares_pred, 
	int* d_DivRayTubeNum, float lmd, float fi, Axis direction, int RayTubeNumNew)
{

	int id = blockIdx.x * blockDim.x + threadIdx.x;

	double ang = (double)fi;

	//�����ɵ���Ҫ����׷�ٵ�����
	if (id < RayTubeNum && d_squares[id].IsDivRayTube == true)
	{
		int index = 4 * (d_DivRayTubeNum[blockIdx.x] + d_squares_pred[id]);

		float3 CenterRay = d_squares[id].CenterRay;
		int4 corner_ray_idx = d_squares[id].CornerRayIndex;
		int center_ray_idx = d_squares[id].CenterRayIndex;

		Direction ray0 = d_rays[corner_ray_idx.x];
		Direction ray1 = d_rays[corner_ray_idx.y];
		Direction ray2 = d_rays[corner_ray_idx.z];
		Direction ray3 = d_rays[corner_ray_idx.w];
		Direction ray4 = d_rays[center_ray_idx];

		//����������
		d_rays2[index].p.x = CenterRay.x;
		d_rays2[index].p.y = CenterRay.y;
		d_rays2[index].p.z = CenterRay.z + lmd;

		d_rays2[index + 1].p.x = CenterRay.x - lmd * sin(ang * 3.1415926 / 180);
		d_rays2[index + 1].p.y = CenterRay.y + lmd * cos(ang * 3.1415926 / 180);
		d_rays2[index + 1].p.z = CenterRay.z;

		d_rays2[index + 2].p.x = CenterRay.x;
		d_rays2[index + 2].p.y = CenterRay.y;
		d_rays2[index + 2].p.z = CenterRay.z - lmd;

		d_rays2[index + 3].p.x = CenterRay.x + lmd * sin(ang * 3.1415926 / 180);
		d_rays2[index + 3].p.y = CenterRay.y - lmd * cos(ang * 3.1415926 / 180);
		d_rays2[index + 3].p.z = CenterRay.z;

		//��������Ϣ����
		d_rays2[index + RayTubeNumNew] = ray0; // 20191223 ��CopyRayTube��CreateRayTubeInfo�ϲ�Ϊһ���˺���
		d_rays2[index + 1 + RayTubeNumNew] = ray1;
		d_rays2[index + 2 + RayTubeNumNew] = ray2;
		d_rays2[index + 3 + RayTubeNumNew] = ray3;
		d_rays2[index / 4 + RayTubeNumNew * 2] = ray4;
	}
}

__global__ void CreateRayTubeIndex(Square *d_squares2, int RayTubeNumNew, int DivRayTubeNum)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	//ÿ�����߹������ĸ��Ƕ����߱��
	if (id < RayTubeNumNew)
	{
		if ((id % 4) == 0)
		{
			d_squares2[id].CornerRayIndex.x = id + RayTubeNumNew;
			d_squares2[id].CornerRayIndex.y = id;
			d_squares2[id].CornerRayIndex.z = RayTubeNumNew * 2 + id / 4;
			d_squares2[id].CornerRayIndex.w = id + 3;
			d_squares2[id].CenterRayIndex = RayTubeNumNew * 2 + DivRayTubeNum + id;
		}
		if ((id % 4) == 1)
		{
			d_squares2[id].CornerRayIndex.x = id - 1;
			d_squares2[id].CornerRayIndex.y = id + RayTubeNumNew;
			d_squares2[id].CornerRayIndex.z = id;
			d_squares2[id].CornerRayIndex.w = RayTubeNumNew * 2 + id / 4;
			d_squares2[id].CenterRayIndex = RayTubeNumNew * 2 + DivRayTubeNum + id;
		}
		if ((id % 4) == 2)
		{
			d_squares2[id].CornerRayIndex.x = RayTubeNumNew * 2 + id / 4;
			d_squares2[id].CornerRayIndex.y = id - 1;
			d_squares2[id].CornerRayIndex.z = id + RayTubeNumNew;
			d_squares2[id].CornerRayIndex.w = id;
			d_squares2[id].CenterRayIndex = RayTubeNumNew * 2 + DivRayTubeNum + id;
		}
		if ((id % 4) == 3)
		{
			d_squares2[id].CornerRayIndex.x = id;
			d_squares2[id].CornerRayIndex.y = RayTubeNumNew * 2 + id / 4;
			d_squares2[id].CornerRayIndex.z = id - 1;
			d_squares2[id].CornerRayIndex.w = id + RayTubeNumNew;
			d_squares2[id].CenterRayIndex = RayTubeNumNew * 2 + DivRayTubeNum + id;
		}
	}
}

void DivRayTube(Direction *d_rays, Square *d_squares, Direction *d_rays2, Square *d_squares2, int* d_DivRayTubeNum, int DivRayTubeNum, int* d_sum_gmem, 
	int* d_squares_pred, int RayTubeNum, float lmd, float fi, Axis direction)
{
	int RayTubeNumNew = DivRayTubeNum * 4;
	//printf("RayTubeNumNew %d\n", RayTubeNumNew);
	
	dim3 SizeOfthread(512, 1, 1);
	dim3 SizeOfblock(RayTubeNumNew / 512 + 1, 1, 1);
	CreateRayTubeIndex << <SizeOfblock, SizeOfthread >> >(d_squares2, RayTubeNumNew, DivRayTubeNum);
	
	/*Square* squares2;
	squares2 = (Square*)malloc(3998000 * sizeof(Square));
	HANDLE_ERROR(hipMemcpy(squares2, d_squares2, 3998000 * sizeof(Square), hipMemcpyDeviceToHost));
	int count=0;
	for(int i=841659; i< 841660; i++)
	{
		printf("%d %d %d %d %d\n", squares2[i].ray_index[0], squares2[i].ray_index[1], squares2[i].ray_index[2], squares2[i].ray_index[3], squares2[i].ray_index[4]);
	}*/
	
	dim3 threadSize(512, 1, 1);
	dim3 blockSize(RayTubeNum / 512 + 1, 1, 1);
	CreateRayTubeInfo << <blockSize, threadSize >> >(d_squares, d_rays, RayTubeNum, d_rays2, d_squares_pred, d_DivRayTubeNum, lmd, fi, direction, RayTubeNumNew);
	
	/*Direction* rays2;
	rays2 = (Direction*)malloc(8000000 * sizeof(Direction));
	HANDLE_ERROR(hipMemcpy(rays2, d_rays2, 8000000 * sizeof(Direction), hipMemcpyDeviceToHost));
	printf("rays %d: %f %f %f\n", 841659, rays2[841659].p[0], rays2[841659].p[1], rays2[841659].p[2]);
	printf("rays %d: %f %f %f\n", 1893734, rays2[1893734].p[0], rays2[1893734].p[1], rays2[1893734].p[2]);
	printf("rays %d: %f %f %f\n", 841658, rays2[841658].p[0], rays2[841658].p[1], rays2[841658].p[2]);
	printf("rays %d: %f %f %f\n", 1683319, rays2[1683319].p[0], rays2[1683319].p[1], rays2[1683319].p[2]);
	printf("rays %d: %f %f %f\n", 2735394, rays2[2735394].p[0], rays2[2735394].p[1], rays2[2735394].p[2]);
	printf("-------------------------------------------------------\n");
	printf("rays %d: %f %f %f\n", 0, rays2[0].p[0], rays2[0].p[1], rays2[0].p[2]);
	printf("rays %d: %f %f %f\n", 1, rays2[1].p[0], rays2[1].p[1], rays2[1].p[2]);
	printf("rays %d: %f %f %f\n", 2, rays2[2].p[0], rays2[2].p[1], rays2[2].p[2]);
	printf("rays %d: %f %f %f\n", 3, rays2[3].p[0], rays2[3].p[1], rays2[3].p[2]);
	printf("rays %d: %f %f %f\n", 4, rays2[4].p[0], rays2[4].p[1], rays2[4].p[2]);
	printf("-------------------------------------------------------\n");*/
	
	DivRayTubeCenter << <SizeOfblock, SizeOfthread >> >(d_squares2, d_rays2, RayTubeNumNew, DivRayTubeNum, direction);
	
	/*//Direction* rays2;
	//rays2 = (Direction*)malloc(8000000 * sizeof(Direction));
	HANDLE_ERROR(hipMemcpy(rays2, d_rays2, 8000000 * sizeof(Direction), hipMemcpyDeviceToHost));
	printf("rays %d: %f %f %f\n", 841659, rays2[841659].p[0], rays2[841659].p[1], rays2[841659].p[2]);
	printf("rays %d: %f %f %f\n", 1893734, rays2[1893734].p[0], rays2[1893734].p[1], rays2[1893734].p[2]);
	printf("rays %d: %f %f %f\n", 841658, rays2[841658].p[0], rays2[841658].p[1], rays2[841658].p[2]);
	printf("rays %d: %f %f %f\n", 1683319, rays2[1683319].p[0], rays2[1683319].p[1], rays2[1683319].p[2]);
	printf("rays %d: %f %f %f\n", 2735394, rays2[2735394].p[0], rays2[2735394].p[1], rays2[2735394].p[2]);
	printf("-------------------------------------------------------\n");*/

	//CopyRayTube << <blockSize, threadSize >> >(d_squares, RayTubeNum, d_squares2, d_rays2, d_rays, d_squares_pred, d_DivRayTubeNum, RayTubeNumNew);
	
	/*//Direction* rays2;
	//rays2 = (Direction*)malloc(8000000 * sizeof(Direction));
	HANDLE_ERROR(hipMemcpy(rays2, d_rays2, 8000000 * sizeof(Direction), hipMemcpyDeviceToHost));
	printf("rays %d: %f %f %f\n", 841659, rays2[841659].p[0], rays2[841659].p[1], rays2[841659].p[2]);
	printf("rays %d: %f %f %f\n", 1893734, rays2[1893734].p[0], rays2[1893734].p[1], rays2[1893734].p[2]);
	printf("rays %d: %f %f %f\n", 841658, rays2[841658].p[0], rays2[841658].p[1], rays2[841658].p[2]);
	printf("rays %d: %f %f %f\n", 1683319, rays2[1683319].p[0], rays2[1683319].p[1], rays2[1683319].p[2]);
	printf("rays %d: %f %f %f\n", 2735394, rays2[2735394].p[0], rays2[2735394].p[1], rays2[2735394].p[2]);
	printf("-------------------------------------------------------\n");
	printf("rays %d: %f %f %f\n", 841659, rays2[841659].dir[0], rays2[841659].dir[1], rays2[841659].dir[2]);
	printf("rays %d: %f %f %f\n", 1893734, rays2[1893734].dir[0], rays2[1893734].dir[1], rays2[1893734].dir[2]);
	printf("rays %d: %f %f %f\n", 841658, rays2[841658].dir[0], rays2[841658].dir[1], rays2[841658].dir[2]);
	printf("rays %d: %f %f %f\n", 1683319, rays2[1683319].dir[0], rays2[1683319].dir[1], rays2[1683319].dir[2]);
	printf("rays %d: %f %f %f\n", 2735394, rays2[2735394].dir[0], rays2[2735394].dir[1], rays2[2735394].dir[2]);
	printf("-------------------------------------------------------\n");*/

}