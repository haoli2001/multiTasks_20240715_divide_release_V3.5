#include "hip/hip_runtime.h"
#include "ReflectCoeff_2.h"

__device__ __host__ comp1 cdiv(comp1 z1,comp1 z2)       
{ double x1,x2,y1,y2;
  comp1 z;
  x1=z1.re;
  x2=z2.re;
  y1=z1.im;
  y2=z2.im;
  z.re=(x1*x2+y1*y2)/(x2*x2+y2*y2);
  z.im=(x2*y1-y2*x1)/(x2*x2+y2*y2);
  return z;
}

__device__ __host__ comp1 cmul(comp1 z1,comp1 z2)
{ double x1,x2,y1,y2;
  comp1 z;
  x1=z1.re;
  x2=z2.re;
  y1=z1.im;
  y2=z2.im;
  z.re=x1*x2-y1*y2;
  z.im=x1*y2+y1*x2;
  return z;
}

__device__ __host__ double cabs(comp1 z)                      
{
	double x,y;
	x=z.re;
	y=z.im;
	return sqrt(x*x+y*y);
}

//����˫������T��ķ���ϵ��
__host__ float ReflectCoeff_2(float f, float theta)
{
	double theta0, theta1, theta2;
	double c0, c1, c2, c3;
	double rou0, rou1, rou2, rou3;
	double Ee2;
	double Z0, Z3, Z4;
	double eta1, eta2;
	double k1, k2, k3;
	double d1, d2, d3;
	double h1, h2, h3;
	double R, AR;
	double phi;
	double AR0, AR1;
	comp1 temp1, temp2, Z23, Z12, Z01, Zin, cw1, cw2, Z1, Z2;

	theta0 = theta;  //����Ƕ�
	rou0 = 1000;     //ˮ�ܶȣ���λkg/m^3��
	c0 = 1500;       //ˮ�����٣���λm/s��
	Z0 = rou0 * c0;  //ˮ���迹

	rou1 = 1039;     //�𽺣�����1���ܶ�
	c1 = 1470;     //����1�еĵ�Ч����
	eta1 = 0.4;      //�������
	phi = atan(eta1);
	cw1.re = c1 * pow((1 * 1 + eta1*eta1), 0.25)*cos(phi / 2);
	cw1.im = c1 * pow((1 * 1 + eta1*eta1), 0.25)*sin(phi / 2);//ճ�����ϲ���
	Z1.re = rou1 * cw1.re;    //���ϵ��ݲ����迹
	Z1.im = rou1 * cw1.im;    //���ϵ��ݲ����迹
	k1 = 2 * 180 * A2R * f / c1;//����1����

	rou2 = 1090;     //�𽺣�����1���ܶ�
	eta2 = 0.5;      //�������
	phi = atan(eta2);
	Ee2 = 1e9;      //����2������ģ��
	c2 = sqrt(Ee2 / rou2);//����2�еĵ�Ч����
	cw2.re = c2 * pow((1 * 1 + eta2*eta2), 0.25)*cos(phi / 2);
	cw2.im = c2 * pow((1 * 1 + eta2*eta2), 0.25)*sin(phi / 2);//ճ�����ϲ���
	Z2.re = rou2 * cw2.re;    //���ϵ��ݲ����迹
	Z2.im = rou2 * cw2.im;    //���ϵ��ݲ����迹
	k2 = 2 * 180 * A2R * f / c2;//���ʲ���

	rou3 = 7850;     //�֣��ĵף��ܶ�
	c3 = 5200;       //����2�еĵ�Ч����
	Z3 = rou3 * c3;  //�ĵ�(�ֵ��迹����ˮ��20��)
	k3 = 2 * 180 * A2R * f / c3;//���ʲ���,�ĵ�Ϊ�ְ�ʱ�Ĵ����ٶ�

	d1 = 2e-3;       //����1���
	d2 = 2e-3;       //����2���
	d3 = 3e-3;       //�ĵ׺��

					 //////����Snell���䶨�����������ʲ��е�����Ƕ�
	AR0 = sin(theta0) * c0 / c1;  //����ȫ����ϵ��
	if (fabs(AR0) > 1)
	{
		theta1 = 90 * A2R;
	}
	else
	{
		theta1 = asin(sin(theta0) * c0 / c1);
	}

	AR1 = sin(theta1) * c1 / c2;  //����ȫ����ϵ��
	if (fabs(AR1) > 1)
	{
		theta2 = 90 * A2R;
	}
	else
	{
		theta2 = asin(sin(theta1) * c1 / c2);
	}
	h1 = tan(k1 * cos(theta0) * d1);
	h2 = tan(k2 * cos(theta1) * d2);
	h3 = tan(k3 * cos(theta2) * d3);
	Z4 = Z0;

	temp1.im = Z3 * h3;
	temp1.re = Z4;
	temp2.im = Z4 * h3;
	temp2.re = Z3;
	Z23 = cdiv(temp1, temp2);
	Z23.re = Z3 * Z23.re;
	Z23.im = Z3 * Z23.im;


	temp1.im = Z23.im + Z2.re * h2;
	temp1.re = Z23.re - h2*Z2.im;
	temp2.im = Z23.re * h2 + Z2.im;
	temp2.re = Z2.re - Z23.im * h2;
	Z12 = cdiv(temp1, temp2);
	Z12 = cmul(Z12, Z2);

	temp1.im = Z12.im + Z1.re * h1;
	temp1.re = Z12.re - h1*Z1.im;
	temp2.im = Z1.im + Z12.re * h1;
	temp2.re = Z1.re - Z12.im * h1;
	Z01 = cdiv(temp1, temp2);
	Z01 = cmul(Z01, Z1);
	Zin = Z01;
	//Zin = Z1 * (Z3 * (Z2 - h1 * h2 * Z1) + jay * Z2 * (Z2 * h2 + h1 * Z1))/(Z2 * (Z1 - h1 * h2 * Z2) + jay * Z3 * (Z1 * h2 + h1 * Z2));  //�����迹
	R = fabs((cabs(Zin) - Z0) / (cabs(Zin) + Z0)); //����ϵ��
	return R;
}
