#include "hip/hip_runtime.h"
﻿#include "calcThreadFunction.h"

#include <omp.h>
#include <semaphore.h>
#include <hip/hip_runtime.h>
#include "handlerror.h"
#include "kd_struct.h"
#include "integral_gpu.h"
#include "raystrace.h"
#include "virtualface_gpu.h"
#include "cudaMallocFree.h"
#include "tree2vector.h"
#include "socketFunctions.h"
#include "simple_time.h"
#include "scalfuc.h"
#include "ReflectCoeff_2.h"
#include "martixMulti.h"
#include <pthread.h>
#define MAX_PIPELINE_CAPICITY 150000000
/**************************
名称：struct PreBlockData
描述：每一个Block中申请的内存指针
***************************/
struct PreBlockData
{
	KD_Node_V *d_tree;
	Prim_Box *d_out_array;
	Element *d_points;
	Triangle *d_triangles;


	Axis direction;
	Direction* d_rays1;
	Direction* d_rays2;
	Square* d_squares1;
	Square* d_squares2;
	RayBeamInfo* d_effrays;
	Vector* d_center;
	Vector* d_axis;
	MatStruct* d_transMat;
	ReimOutput* d_reim;
	float* d_sum_re;
	float* d_sum_im;
	float* d_sum_sre; 
	float* d_sum_sim; 
	int* d_DivRayTubeNum;
	int* d_sum_gmem;
	int* d_sum_Gmem;
	int* d_sum;
	int* d_squares_pred;
	int DivRayTubeNum1st;
	int DivRayTubeNum2nd;
	int DivRayTubeNum3rd;

};

/**************************
名称：struct FreePoint
描述：存储申请的内存指针， 当做参数传递给清理函数cleanup，
      用来在线程被中途杀掉后释放内存
***************************/
struct FreePoint
{
	ConfigStruct config;
	KD_Node_V **h_tree;
	Prim_Box **h_out_array;
	float **pre_triangle_result;
	float **pre_triangle_result_reim;
	PreBlockData **plan;
	int **pre_device_height;
	int **pre_device_width;
	int **height;

	float **e_st_min;
	float **e_fi_max;
	int **divided_num;
    int **divided_width;
    int **divided_height;
    float **divided_st_min;
    float **divided_fi_max;
	ReimOutput*** h_reim;
	comp*** h_TSOfPerTriangle;
	
	DynamicPlane **SubAperturePlane;
	DynamicPlane **AperturePlane;
	BinaryTimeTree ***PreAngelTime;
	
	Element **h_points;	//存储坐标变换后模型point数据的指针
};

/**************************
名称：void cleanup(void* argv)
描述：清理函数，用来在线程被中途杀掉后释放内存
参数：void* argv:   FreePoint 结构体指针
返回值：无
***************************/
void cleanup(void* argv)
{
	FreePoint *fp = (FreePoint*)argv;
	ConfigStruct config = fp->config;
	if((*fp->plan)!=NULL)
	{
		for(int i=0;i<config.card_num;i++)
		{
			HANDLE_ERROR(hipSetDevice(config.select_device_list[i]));

			FreeOnGPU((*fp->plan)[i].d_rays1, (*fp->plan)[i].d_squares1, (*fp->plan)[i].d_rays2, (*fp->plan)[i].d_squares2, 
				(*fp->plan)[i].d_effrays, (*fp->plan)[i].d_center, (*fp->plan)[i].d_axis, (*fp->plan)[i].d_transMat, (*fp->plan)[i].d_reim, (*fp->plan)[i].d_sum_re, (*fp->plan)[i].d_sum_im, (*fp->plan)[i].d_sum_sre, (*fp->plan)[i].d_sum_sim,
				(*fp->plan)[i].d_DivRayTubeNum, (*fp->plan)[i].d_sum_gmem, (*fp->plan)[i].d_sum_Gmem, (*fp->plan)[i].d_squares_pred);

			free_data((*fp->plan)[i].d_out_array, (*fp->plan)[i].d_tree, (*fp->plan)[i].d_points, (*fp->plan)[i].d_triangles);
		}
	}
	//host Free
	if(*fp->plan!=NULL)
	{
		free(*fp->plan);
	}
	if(*fp->pre_device_height!=NULL)
	{
		free(*fp->pre_device_height);
	}
	if(*fp->pre_device_width!=NULL)
	{
		free(*fp->pre_device_width);
	}
	if(*fp->height!=NULL)
	{
		free(*fp->height);
	}
	if(*fp->e_fi_max!=NULL)
	{
		free(*fp->e_fi_max);
	}
	if(*fp->e_st_min!=NULL)
	{
		free(*fp->e_st_min);
	}
	
	if(*fp->divided_num!=NULL)
	{
		free(*fp->divided_num);
	}
	if(*fp->divided_width!=NULL)
	{
		free(*fp->divided_width);
	}
	if(*fp->divided_height!=NULL)
	{
		free(*fp->divided_height);
	}
	if(*fp->divided_st_min!=NULL)
	{
		free(*fp->divided_st_min);
	}
	if(*fp->divided_fi_max!=NULL)
	{
		free(*fp->divided_fi_max);
	}
	if(*fp->h_out_array!=NULL)
	{
		free(*fp->h_out_array);
	}
	if(*fp->h_tree!=NULL)
	{
		free(*fp->h_tree);
	}
	
	if(*fp->h_reim!=NULL)
	{
		for(int i=0;i<config.card_num;i++)
			free((*fp->h_reim)[i]);
		free(*fp->h_reim);
	}
	if(*fp->h_TSOfPerTriangle!=NULL)
	{
		for(int i=0;i<config.card_num;i++)
			free((*fp->h_TSOfPerTriangle)[i]);
		free(*fp->h_TSOfPerTriangle);
	}
	if(*fp->pre_triangle_result!=NULL)
	{
		free(*fp->pre_triangle_result);
	}
	if(*fp->pre_triangle_result_reim!=NULL)
	{
		free(*fp->pre_triangle_result_reim);
	}
	if(*fp->SubAperturePlane!=NULL)
	{
		free(*fp->SubAperturePlane);
	}
	if(*fp->AperturePlane!=NULL)
	{
		free(*fp->AperturePlane);
	}
	if(*fp->PreAngelTime!=NULL)
	{
		for(int i=0;i < 2 * config.card_num - 1;i++)
			free((*fp->PreAngelTime)[i]);
		free(*fp->PreAngelTime);
	}
	if(*fp->h_points!=NULL)
	{
		free(*fp->h_points);
	}
}

/**************************
名称：void* calcThreadFunction(void *argv)
描述：计算线程函数
参数：void* argv:   传递给计算线程的参数
返回值：无
***************************/
void* calcThreadFunction(void *argv)
{
    //计算线程参数
	CalcInfo calcInfo = *(CalcInfo*)argv;
	
	int socketClient = calcInfo.socket;     
	KD_Node_V *h_tree = NULL;
	Prim_Box *h_out_array = NULL;
	Element *h_points = NULL;
	Element *h_points_old = NULL;
	Triangle *h_triangles = NULL;
	float *pre_triangle_result = NULL;
	float *pre_triangle_result_reim = NULL;
	PreBlockData *plan = NULL;
	int *pre_device_height = NULL;
	int *pre_device_width = NULL;
	int *height = NULL;
	int *divided_num = NULL;
	int *divided_width = NULL;
	int *divided_height = NULL;
	float *divided_st_min = NULL;
	float *divided_fi_max = NULL;
	float *e_st_min = NULL;
	float *e_fi_max = NULL;
	ReimOutput** h_reim = NULL;
	comp** h_TSOfPerTriangle = NULL;
	DynamicPlane *SubAperturePlane = NULL;
	DynamicPlane *AperturePlane = NULL;
	BinaryTimeTree **PreAngelTime = NULL;
	
    //中途退出需要释放的内存结构体
	FreePoint freePt;
	freePt.config= calcInfo.config;
	freePt.h_tree=&h_tree;
	freePt.h_out_array=&h_out_array;
	freePt.pre_triangle_result=&pre_triangle_result;
	freePt.pre_triangle_result_reim=&pre_triangle_result_reim;
	freePt.plan=&plan;
	freePt.pre_device_height=&pre_device_height;
	freePt.pre_device_width=&pre_device_width;
	freePt.height=&height;

	freePt.e_st_min=&e_st_min;
	freePt.e_fi_max=&e_fi_max;
	freePt.divided_num=&divided_num;
	freePt.divided_width=&divided_width;
	freePt.divided_height=&divided_height;
	freePt.divided_st_min=&divided_st_min;
	freePt.divided_fi_max=&divided_fi_max;
	freePt.h_reim=&h_reim;
	freePt.h_TSOfPerTriangle=&h_TSOfPerTriangle;
	
	freePt.SubAperturePlane=&SubAperturePlane;
	freePt.AperturePlane=&AperturePlane;
	freePt.PreAngelTime=&PreAngelTime;
	freePt.h_points = &h_points;

    //设置退出线程时调用cleanup清理内存
	pthread_cleanup_push(cleanup,&freePt);
    //允许退出线程 
	pthread_setcancelstate(PTHREAD_CANCEL_ENABLE, NULL);
    //下一个监视点取消    
	pthread_setcanceltype(PTHREAD_CANCEL_DEFERRED, NULL);  

	int h_tree_length;
	int h_out_array_length;
	int h_points_length;
	int h_triangle_length;
	
	//每一个三角形积分结果，用于伪彩图显示
	pre_triangle_result = (float*)malloc(sizeof(float)*calcInfo.triangles_length);
	memset(pre_triangle_result,0,sizeof(float)*calcInfo.triangles_length);
	pre_triangle_result_reim = (float*)malloc(2*sizeof(float)*calcInfo.triangles_length);
	memset(pre_triangle_result_reim,0,2*sizeof(float)*calcInfo.triangles_length);
    
	//根据船体晃动的三个角度，生成坐标旋转矩阵，旋转模型坐标
	//并根据用户的输入水线深度，计算模型坐标中的绝对水线z坐标
	float rolX[9] = {1,0,0,0,cos(calcInfo.config.sway_phi),sin(calcInfo.config.sway_phi),0,-sin(calcInfo.config.sway_phi),cos(calcInfo.config.sway_phi)};
	float rolY[9] = {cos(calcInfo.config.sway_theta),0,-sin(calcInfo.config.sway_theta),0,1,0,sin(calcInfo.config.sway_theta),0,cos(calcInfo.config.sway_theta)};
	float rolZ[9] = {cos(calcInfo.config.sway_psi),sin(calcInfo.config.sway_psi),0,-sin(calcInfo.config.sway_psi),cos(calcInfo.config.sway_psi),0,0,0,1};
	float Change_Corrd_Matrix[9];
	float tempMatrix[9];
	martixMulti(rolX, 3, 3, rolZ, 3, 3, tempMatrix);
	martixMulti(tempMatrix, 3, 3, rolY, 3, 3, Change_Corrd_Matrix);
	float abs_waterLine_axis = INT_MAX;
	h_points_old = calcInfo.points;
	h_points_length = calcInfo.points_length;
	h_points = (Element*)malloc(sizeof(Element) * h_points_length);
	for(int i=0;i<h_points_length;i++)
	{
		if(h_points_old[i].point[2] < abs_waterLine_axis)
			abs_waterLine_axis = h_points_old[i].point[2];
		martixMulti(Change_Corrd_Matrix, 3, 3, h_points_old[i].point, 3, 1, h_points[i].point);
	}
	abs_waterLine_axis = abs_waterLine_axis + calcInfo.config.water_line;
	printf("\n模型坐标转换完毕，水线深度：%f，模型中水线z坐标：%f\n",calcInfo.config.water_line,abs_waterLine_axis);


	//记录KDTree 建树时间
	simple_time simpleTimeKDTree;
	float timeKDTree;

    //建立KDTree
	printf("\n\n*************KD_Tree build begin*************\n");
	simpleTimeKDTree.Time_Start();
	{
		int Out_Arr_Length = 0;
		int index = 0;
		
		KD_Node *Root_Node = (KD_Node*)malloc(sizeof(KD_Node));
		Prim_Box *Triangles_Box = (Prim_Box*)malloc(calcInfo.triangles_length * sizeof(Prim_Box));

		Triangle *Triangles;
		Element *Points;
		Prim_Box *Out_Array;
		KD_Infomation KD_Info;

		Out_Array= (Prim_Box *)malloc( 4 * calcInfo.triangles_length * sizeof(Prim_Box));

		Sort_Box(calcInfo.points, calcInfo.triangles, Triangles_Box, calcInfo.triangles_length);
		memset(Root_Node, 0, sizeof(struct KD_Node));//��ʼ�����ڵ�Ϊ0
		memset(&KD_Info, 0, sizeof(struct KD_Infomation));//��ʼ��ͳ����ϢΪ0
		KD_Node_init(Root_Node, Triangles_Box, calcInfo.triangles_length);//��������Ķ�����Ϣ�����г�ʼ����ֵ


		Build_BigNode(Root_Node, Triangles_Box, Out_Array, calcInfo.triangles_length, &Out_Arr_Length);
		Optimization_Rope(Root_Node);//����ֵ���Ż�

        //将KDTree之前的树状存储结构转换为连续存储的数组形式
		KD_Node_V *root_v = tree2vector(Root_Node, &index);
		
        //销毁树状结构的KDTree
		Destroy_Tree(Root_Node);
		
		free(Triangles_Box);
	
		h_out_array = Out_Array;
		h_out_array_length = Out_Arr_Length;
		h_tree = root_v;
		h_tree_length = index;
		h_triangles = calcInfo.triangles;
		h_triangle_length = calcInfo.triangles_length;
	}
	timeKDTree = simpleTimeKDTree.Time_End();

	//有时KDTree的length计算会出错，使得该值非常大，因而增加此判断
	if(h_tree_length > 100000000)
		exit(0);

	printf("发送KDTree,建树时间，节点个数等信息\n");
	{
		struct KDTreeInfo
		{
			float time;
			int length;
		};
		KDTreeInfo kdtreeinfo;
		kdtreeinfo.time = timeKDTree;
		kdtreeinfo.length = h_tree_length;
		Frame frame;
		strcpy(frame.command, "KDTreeTime");
		frame.length = sizeof(KDTreeInfo);
		memcpy(frame.data, &kdtreeinfo, sizeof(KDTreeInfo));
		send_frame(socketClient, (char*)&frame, sizeof(Frame));
	}
    
    //线程退出的监视点
	pthread_testcancel(); 
	pthread_setcancelstate(PTHREAD_CANCEL_DISABLE, NULL);
    
	printf("发送KDTree数据！\n");
	{
		int sendedLength = 0;
		Frame frame;

		while (true)
		{
			strcpy(frame.command, "KDTreeDate");
			if (sendedLength + 1024 < h_tree_length*sizeof(KD_Node_V))
			{
				memcpy(frame.data, (char*)h_tree + sendedLength, 1024);
				frame.length = 1024;
				send_frame(socketClient, (char*)&frame, sizeof(Frame));
				sendedLength += 1024;
			}
			else
			{
				memcpy(frame.data, (char*)h_tree + sendedLength, h_tree_length*sizeof(KD_Node_V) - sendedLength);
				frame.length = h_tree_length*sizeof(KD_Node_V) - sendedLength;
				send_frame(socketClient, (char*)&frame, sizeof(Frame));
				break;
			}
		}
	}
	
	pthread_setcancelstate(PTHREAD_CANCEL_ENABLE, NULL);
    //线程退出的监视点
	pthread_testcancel(); 
	pthread_setcancelstate(PTHREAD_CANCEL_DISABLE, NULL);

	plan = (PreBlockData*)malloc(sizeof(PreBlockData)*calcInfo.config.card_num);
	memset(plan, 0, sizeof(PreBlockData)*calcInfo.config.card_num);

	//int sumAngleNum = calcInfo.config.end_alpha - calcInfo.config.start_alpha + 1;
	//保证初始角度不为0时仍能正确计算,  220304 jzy
	int sumAngleNum = calcInfo.config.end_alpha + 1;
	//每一个卡上的宽
	pre_device_height = (int*)malloc(sizeof(int)*sumAngleNum);
	//每一个卡上的高
	pre_device_width = (int*)malloc(sizeof(int)*sumAngleNum);
	divided_num = (int*)malloc(sizeof(int)*sumAngleNum);
	height = (int*)malloc(sizeof(int)*sumAngleNum);

	e_st_min = (float*)malloc(sizeof(float)*sumAngleNum);
	e_fi_max = (float*)malloc(sizeof(float)*sumAngleNum);
	int d_width_max = -1;
	int d_height_max = -1;
    float far_dis = calcInfo.config.far_distance;


	//多卡上的malloc 和树，顶点，三角面片，包围盒,使用OpenMp实现多线程
	omp_set_num_threads(calcInfo.config.card_num);
	#pragma omp parallel
	{
		int i = omp_get_thread_num();
		hipSetDevice(calcInfo.config.select_device_list[i]);

		HANDLE_ERROR(hipMalloc((void**)&plan[i].d_tree, sizeof(KD_Node_V)*h_tree_length));
		HANDLE_ERROR(hipMalloc((void**)&plan[i].d_out_array, sizeof(Prim_Box)*h_out_array_length));
		HANDLE_ERROR(hipMalloc((void**)&plan[i].d_points, sizeof(Element)*h_points_length));
		HANDLE_ERROR(hipMalloc((void**)&plan[i].d_triangles, sizeof(Triangle)*h_triangle_length));
		hipMemcpy(plan[i].d_tree, h_tree, h_tree_length*sizeof(KD_Node_V), hipMemcpyHostToDevice);
		hipMemcpy(plan[i].d_out_array, h_out_array, h_out_array_length*sizeof(Prim_Box), hipMemcpyHostToDevice);
		hipMemcpy(plan[i].d_points, h_points, h_points_length*sizeof(Element), hipMemcpyHostToDevice);
		hipMemcpy(plan[i].d_triangles, h_triangles, h_triangle_length*sizeof(Triangle), hipMemcpyHostToDevice);
	}

	float fai_angle = calcInfo.config.start_beta;
	//获取每一度虚拟孔径面所需宽高，通过先得到每一度的宽高，按最大的开辟空间，减少了每一度扫描的空间开辟时间
	for (int i = calcInfo.config.start_alpha; i <= calcInfo.config.end_alpha; i++)
	{
		//注意：此处angle没有考虑浮点角度！另：pre_device_height无用	jzy 2023.4.4
		float angle = i;
		getWidthHeight(far_dis,h_points, h_points_length, &pre_device_width[i], &height[i], si_angle, angle, calcInfo.config.pipe_size*calcInfo.config.wave_length, &pre_device_height[i], &e_st_min[i], &e_fi_max[i], calcInfo.config.card_num, &divided_num[i], MAX_PIPELINE_CAPICITY);
	}
	//printf("d_height_max=%d, d_width_max=%d\n", d_height_max, d_width_max);
	d_height_max = 1975;
	d_width_max = 10000;

	pthread_setcancelstate(PTHREAD_CANCEL_ENABLE, NULL);
    //线程退出的监视点	
	pthread_testcancel(); 
    pthread_setcancelstate(PTHREAD_CANCEL_DISABLE, NULL);

	//根据最大的宽高开辟空间
	omp_set_dynamic(0);
	omp_set_num_threads(calcInfo.config.card_num);  // create as many CPU threads as there are CUDA devices
	#pragma omp parallel
	{

		int i = omp_get_thread_num();
		hipSetDevice(calcInfo.config.select_device_list[i]);

		MallocOnGPU(d_width_max, d_height_max, &plan[i].d_rays1, &plan[i].d_squares1, &plan[i].d_rays2, &plan[i].d_squares2,
			&plan[i].d_effrays, &plan[i].d_center, &plan[i].d_axis, &plan[i].d_transMat, &plan[i].d_reim, &plan[i].d_sum_re, &plan[i].d_sum_im, &plan[i].d_sum_sre, &plan[i].d_sum_sim,
			&plan[i].d_DivRayTubeNum, &plan[i].d_sum_gmem, &plan[i].d_sum_Gmem, &plan[i].d_squares_pred);
	}

	h_reim = (ReimOutput**)malloc(sizeof(ReimOutput*)*calcInfo.config.card_num);
	for (int t = 0; t < calcInfo.config.card_num; t++)
	{
		h_reim[t] = (ReimOutput*)malloc(sizeof(ReimOutput) * d_width_max * d_height_max);
		memset(h_reim[t], 0, sizeof(ReimOutput) * d_width_max * d_height_max);
	}

	h_TSOfPerTriangle = (comp**)malloc(sizeof(comp*) * calcInfo.config.card_num);
	for (int p = 0; p < calcInfo.config.card_num; p++)
	{
		h_TSOfPerTriangle[p] = (comp*)malloc(sizeof(comp) * calcInfo.triangles_length);
		memset(h_TSOfPerTriangle[p], 0, sizeof(comp) * calcInfo.triangles_length);
	}
	
	int NodeNum = 2 * calcInfo.config.card_num - 1;
	SubAperturePlane = (DynamicPlane*)malloc(calcInfo.config.card_num * sizeof(DynamicPlane));
	AperturePlane = (DynamicPlane*)malloc(NodeNum * sizeof(DynamicPlane));
	PreAngelTime = (BinaryTimeTree**)malloc(NodeNum * sizeof(BinaryTimeTree*));
	for (int i = 0; i < NodeNum; i++)
	{
		PreAngelTime[i] = (BinaryTimeTree*)malloc(sizeof(BinaryTimeTree));
		memset(PreAngelTime[i], 0, sizeof(BinaryTimeTree));
	}

	float CSpeed = 1500.0;//20200919
	float pi = 3.1415926535898;
	float D2R = pi / 180.0;
	float fend = calcInfo.config.time_end_frequency;//姬梓遇20210831
	float fbeg = calcInfo.config.time_start_frequency;////姬梓遇20210831
	float Tao = calcInfo.config.tao/1000;  //原单位为ms，此处转为s 姬梓遇20210831
	float fs = calcInfo.config.sampling_frequency;//caiyanglv 姬梓遇20210831
	float velocity1 = calcInfo.config.velocity1;//鱼雷速度1 姬梓遇
	float velocity2 = calcInfo.config.velocity2;//目标速度1 姬梓遇
    float velocity12 = velocity1 - velocity2;//相对投影速度 LV 20220720
    //float velocity12 = velocity1 * cos(calcInfo.config.H_axis_angle * D2R)  -  velocity2 * cos(calcInfo.config.? + calcInfo.config.H_axis_angle * D2R);//相对投影速度 LV 20220720
	float band = fend - fbeg;
	float KO = band / Tao;//wangying	
    float dopGene = (CSpeed + velocity12) / (CSpeed - velocity12);//多普勒压缩因子
	if(fabs(band) > 0.001)  //宽带信号则考虑脉冲宽度压缩
		Tao = Tao / dopGene;
	fbeg = fbeg * dopGene;//调制后起始频率
	fend = fbeg + band * dopGene * dopGene;   // LV20221229 原有计算，错误： fend = fbeg + band;

	int taosize = int(fs * Tao);
	int maxsize;
	int totalsize= int(calcInfo.config.sampling_width * fs);//20210402

	//FILE* fileresult_1200 = fopen("result_1200", "w");//wangying
	FILE* fileresult_1200;
    FILE* fileresult_TS = fopen("TS.txt", "w");

	char nameout[50];
	float* result_1200_re;
	//memset(result_1200_re, 0, maxsize * sizeof(float));
	float* result_1200_im;
	//memset(result_1200_im, 0, maxsize * sizeof(float));//20200920
	int minZeroNum=INT_MAX,maxZeroNum=INT_MIN;//20210308

	float* result_1200;//20210402
	comp result={0,0};
	float* s_sum_re;//20200919 面积积分结果
	float* s_sum_im;//20200919 面积积分结果

	float* d_m_sum_re[calcInfo.config.card_num];//20210331
	float* d_m_sum_im[calcInfo.config.card_num];//20210331


	//生成多点接收阵列20220527jzy
	int sumnum = calcInfo.config.recvPointsNum;
	float d = 0.026;
	float theta = 0;
	float psi = 0;
	float fai = 0;
	//Axis_slx* receive_points = (Axis_slx*)malloc(sizeof(Axis_slx) * sumnum);
	Axis_slx* receive_points = calcInfo.recvPoints;
	Axis_slx* New_receive_points = (Axis_slx*)malloc(sizeof(Axis_slx) * sumnum);
	printf("雷体坐标系下各坐标\n");
	for(int idx=0;idx<sumnum;idx++)
	{		
            printf("%f %f %f\n",receive_points[idx].p[0],receive_points[idx].p[1],receive_points[idx].p[2]);
	}
	//生成多点接收阵列结束20220527jzy




	//循环计算每一个频率和角度
	float start_alpha;
	if(fabs(calcInfo.config.continue_alpha + 1) < 0.00001)
		start_alpha = calcInfo.config.start_alpha;
	else
		start_alpha = calcInfo.config.continue_alpha;
	

	for(int recv_index = calcInfo.config.recvPointsStartIdx; recv_index < sumnum; recv_index ++)
	{
		float angle = start_alpha;
		for (float f = calcInfo.config.start_frequency; f <= calcInfo.config.end_frequency; f += 0.1)
		{
			for (int i = start_alpha; i <= calcInfo.config.end_alpha; i++)
			{
				float lmd = 1.5 / f;          // 声波波长
				float time_all=0;
				result.re=0;
				result.im=0;
				memset(result_1200, 0, totalsize * sizeof(float));

				divided_width = (int*)malloc(sizeof(int) * divided_num[i]);
				divided_height = (int*)malloc(sizeof(int) * divided_num[i]);
				divided_st_min = (float*)malloc(sizeof(float) * divided_num[i]);
				divided_fi_max = (float*)malloc(sizeof(float) * divided_num[i]);
				//分割虚拟孔径面
				divide_module_virtualface(divided_num[i], calcInfo.config.pipe_size*calcInfo.config.wave_length, sumAngleNum, pre_device_width[i], height[i], e_st_min[i], e_fi_max[i], divided_width, divided_height, divided_st_min, divided_fi_max);






            	//多卡的时间统计 同时启动
            	simple_time *runSimpleTime = new simple_time[calcInfo.config.card_num];
           		float *calcTime = (float*)malloc(sizeof(float)*calcInfo.config.card_num);
           		memset(calcTime,0,sizeof(float)*calcInfo.config.card_num);
				// = fopen("i", "w");//wangying
				sprintf(nameout,"./data/%.2f_recv%d.txt",angle,recv_index);
				fileresult_1200=fopen(nameout,"w");//20200924
		
				pthread_setcancelstate(PTHREAD_CANCEL_ENABLE, NULL);
            	//线程退出的监视点
				pthread_testcancel(); 
				//避免线程在其他地方cancel，以保证上位机能正常暂停计算 2022.3.24 jzy
				pthread_setcancelstate(PTHREAD_CANCEL_DISABLE, NULL);
            	//先将数据清零，避免上一度的结果影响
				for (int p = 0; p < calcInfo.config.card_num; p++)
				{
					memset(h_TSOfPerTriangle[p], 0, sizeof(comp) * calcInfo.triangles_length);
				}
				comp *rnt_sum = (comp *)malloc(sizeof(comp)*calcInfo.config.card_num);
				memset(rnt_sum, 0,sizeof(comp)*calcInfo.config.card_num);
				float lmd = 1.5 / f;          // 声波波长


				//多点接收阵列坐标转换到目标坐标系
				float R0 = far_dis;
				float alpha = angle * D2R;
				float beta = fai_angle * D2R;
				//float P0_B_A = R0 * sin(beta) * cos(alpha)
				float H_Axis_Angle = calcInfo.config.H_axis_angle * D2R;
				float V_Axis_Angle = calcInfo.config.V_axis_angle * D2R;
		
				printf("H_Axis_Angle = %f\n",H_Axis_Angle);	//调试用
				if((alpha >=0* D2R)&&(alpha <90*D2R))
				{
					H_Axis_Angle = H_Axis_Angle;
				}
				else if((alpha >=90* D2R)&&(alpha <=180*D2R))
				{
					H_Axis_Angle = -H_Axis_Angle;
				}
				else
				{}

				if((beta  >=0* D2R)&&(beta  <90*D2R))
				{
					V_Axis_Angle = -V_Axis_Angle;
				}
				else if((beta  >90* D2R)&&(beta  <=180*D2R))
				{
					V_Axis_Angle = V_Axis_Angle;
				}
				else
				{}

				psi = (beta + V_Axis_Angle - pi/2);
				theta = -(pi - (alpha - H_Axis_Angle));
				//生成坐标转换矩阵


				//float rolX[9] = {1,0,0,0,cos(fai),sin(fai),0,-sin(fai),cos(fai)};
				//float rolY[9] = {cos(theta),0,-sin(theta),0,1,0,sin(theta),0,cos(theta)};
				//float rolZ[9] = {cos(psi),sin(psi),0,-sin(psi),cos(psi),0,0,0,1};


				float rolX[9] = {1,0,0,0,cos(fai),sin(fai),0,-sin(fai),cos(fai)};
				float rolY[9] = {cos(theta),0,-sin(theta),0,1,0,sin(theta),0,cos(theta)};
				float rolZ[9] = {cos(psi),sin(psi),0,-sin(psi),cos(psi),0,0,0,1};

				float Change_Corrd_Matrix[9];
				float tempMatrix[9];
				martixMulti(rolX, 3, 3, rolZ, 3, 3, tempMatrix);
				martixMulti(tempMatrix, 3, 3, rolY, 3, 3, Change_Corrd_Matrix);
				int jj;
				/*
				printf("fai:%f\n",fai);
				printf("theta:%f\n",theta);
				printf("psi:%f\n",psi);

			
				printf("rolX旋转矩阵各坐标\n");
				for(jj = 0;jj<9;jj++)
				{
					printf("%f\n",rolX[jj]);
				}
				printf("rolY旋转矩阵各坐标\n");
				for(jj = 0;jj<9;jj++)
				{
					printf("%f\n",rolY[jj]);
				}
				for(jj = 0;jj<9;jj++)
				{
					printf("%f\n",rolZ[jj]);
				}
				printf("旋转矩阵各坐标\n");
				for(jj = 0;jj<9;jj++)
				{
					printf("%f\n",Change_Corrd_Matrix[jj]);
				}
				printf("目标坐标系下基阵几何中心坐标\n");	
				printf("%f %f %f\n",R0 * sin(beta) * cos(alpha),R0 * cos(beta),R0 * sin(beta) * sin(alpha));
				*/
				printf("目标坐标系下各坐标\n");	
				for(int idx=0;idx<sumnum;idx++)
				{
					martixMulti(Change_Corrd_Matrix, 3, 3, receive_points[idx].p, 3, 1, New_receive_points[idx].p);
					printf("%f %f %f\n",New_receive_points[idx].p[0],New_receive_points[idx].p[1],New_receive_points[idx].p[2]);
					New_receive_points[idx].p[0] += R0 * sin(beta) * cos(alpha);
					New_receive_points[idx].p[1] += R0 * cos(beta);
					New_receive_points[idx].p[2] += R0 * sin(beta) * sin(alpha);

					//互换y、z坐标，保证一致
					float temp_index = 0;
					temp_index = New_receive_points[idx].p[1];
					New_receive_points[idx].p[1] = New_receive_points[idx].p[2];
					New_receive_points[idx].p[2] = temp_index;

				}
			
				//多点接收阵列坐标转换到目标坐标系结束


				//	float *s_sum_re;//20200919 面积积分结果
				//float* s_sum_im;//20200919 面积积分结果

				// 动态生成子孔径面边界信息
				ConstructVirtualFace(AperturePlane, SubAperturePlane, PreAngelTime, calcInfo.config.card_num, e_st_min[i], e_fi_max[i], pre_device_width[i], height[i], calcInfo.config.pipe_size * calcInfo.config.wave_length);
			
				// 开始计时
				for(int index=0;index<calcInfo.config.card_num;index++)
            	{
               	 	runSimpleTime[index].Time_Start();
            	}
			
				//多卡并行计算
            	simple_time simple_time_TS_compute;
            	simple_time_TS_compute.Time_Start();
				omp_set_num_threads(calcInfo.config.card_num);  // create as many CPU threads as there are CUDA device
				#pragma omp parallel
				{
				int j = omp_get_thread_num();                        //目前线程id,即卡id
				int num_threads = omp_get_num_threads();             //获取卡的数量
				hipSetDevice(calcInfo.config.select_device_list[j]);
				//hipDeviceEnablePeerAccess(j, 0);

				Radius direction_radius = { far_dis, fai_angle, angle }; //snw 接入远场距离
				//printf("direction_radius={%f,%f,%f}\n",direction_radius.Xr,direction_radius.Yst,direction_radius.Zfi);
				plan[j].direction = dSphericaltoRectangular(direction_radius);//等相位面的法向量

				MemsetOnGPU(d_width_max, d_height_max, &plan[j].d_rays1, &plan[j].d_squares1, &plan[j].d_rays2, &plan[j].d_squares2,
					&plan[j].d_effrays, &plan[j].d_center, &plan[j].d_axis, &plan[j].d_transMat, &plan[j].d_reim, &plan[j].d_sum_re, &plan[j].d_sum_im, &plan[j].d_sum_sre, &plan[j].d_sum_sim,
					&plan[j].d_DivRayTubeNum, &plan[j].d_sum_gmem, &plan[j].d_sum_Gmem, &plan[j].d_squares_pred);
					
				//printf("GPU%d: width=%d, height=%d, st_min=%e, fi_max=%e\n", j, SubAperturePlane[j].width, SubAperturePlane[j].height, SubAperturePlane[j].st_min, SubAperturePlane[j].fi_max);

                //创建虚拟孔径面
				create_virtualface_gpu(plan[j].d_rays1, plan[j].d_squares1, SubAperturePlane[j].width, SubAperturePlane[j].height,
					calcInfo.config.pipe_size * calcInfo.config.wave_length, direction_radius, SubAperturePlane[j].st_min, SubAperturePlane[j].fi_max, j);
                
                //射线追踪
				allraystrace_v2(plan[j].d_rays1, plan[j].d_squares1, SubAperturePlane[j].width, SubAperturePlane[j].height,
					plan[j].d_tree, plan[j].d_out_array, plan[j].d_points, plan[j].d_triangles, plan[j].d_DivRayTubeNum,
					&(plan[j].DivRayTubeNum1st), plan[j].d_sum_gmem, plan[j].d_sum_Gmem, plan[j].d_squares_pred, plan[j].direction, angle, abs_waterLine_axis);
                
                //声场积分
				RayBeamInfo* c_effrays = (RayBeamInfo*)malloc(SubAperturePlane[j].width * SubAperturePlane[j].height  * sizeof(RayBeamInfo));

				comp sum = sound_field_integral_gpu(plan[j].d_rays1, plan[j].d_squares1, lmd, SubAperturePlane[j].width * SubAperturePlane[j].height, plan[j].d_effrays, plan[j].d_center,
					plan[j].d_axis, plan[j].d_transMat, plan[j].d_reim, plan[j].d_sum_re, plan[j].d_sum_im, fai_angle, angle, &(calcInfo.config));//20210831姬梓遇
                calcTime[j]+=runSimpleTime[j].Time_End();

				HANDLE_ERROR(hipMemcpy(c_effrays, plan[j].d_effrays, SubAperturePlane[j].width * SubAperturePlane[j].height * sizeof(RayBeamInfo), hipMemcpyDeviceToHost)); 


				//copy出各声线的d_center,用于下面计算时域积分的maxsize	20220610
				Vector* c_center = (Vector*)malloc(SubAperturePlane[j].width * SubAperturePlane[j].height * sizeof(Vector));
				HANDLE_ERROR(hipMemcpy(c_center, plan[j].d_center, SubAperturePlane[j].width * SubAperturePlane[j].height * sizeof(Vector), hipMemcpyDeviceToHost)); 

				for(int idx=0;idx<SubAperturePlane[j].width * SubAperturePlane[j].height;idx++)
				{
					float recv_p_cent_distance = sqrt(pow(c_center[idx].x - New_receive_points[recv_index].p[0], 2) + pow(c_center[idx].y - New_receive_points[recv_index].p[1], 2) + pow(c_center[idx].z - New_receive_points[recv_index].p[2], 2));
					int pendZeroNum = int( (recv_p_cent_distance + c_effrays[idx].p_cent_distance) / CSpeed * fs) ;//wangying  snw:pendZeroNum从2倍距离算起
					if(pendZeroNum!=0&&pendZeroNum<minZeroNum) minZeroNum=pendZeroNum;
					if(pendZeroNum>maxZeroNum) maxZeroNum=pendZeroNum;
				}//20210308
				free(c_effrays);
        		free(c_center);	   
                //拷贝出积分结果，用来伪彩图显示
				hipMemcpy(h_reim[j], plan[j].d_reim, d_width_max * d_height_max * sizeof(ReimOutput), hipMemcpyDeviceToHost);
				for (int n = 0; n < d_width_max * d_height_max; n++)
				{
					if (h_reim[j][n].triangle_index >= 0)
					{
						h_TSOfPerTriangle[j][h_reim[j][n].triangle_index].re += h_reim[j][n].re;
						h_TSOfPerTriangle[j][h_reim[j][n].triangle_index].im += h_reim[j][n].im;
					}
				}
				float reflect_coeff = calcInfo.config.reflect_coeff_Auto_flag ? ReflectCoeff_2(f,i) : calcInfo.config.reflect_coeff; //计算反射系数 姬梓遇
				sum.re = sum.re * reflect_coeff;//积分结果乘反射系数
				sum.im = sum.im * reflect_coeff;//积分结果乘反射系数
				rnt_sum[j].im = sum.im;
				rnt_sum[j].re = sum.re;

				maxsize = maxZeroNum + taosize - minZeroNum + 400;

				printf("maxsize, minZeroNum, maxZeroNum+taosize: %d, %d, %d\n",maxsize, minZeroNum, maxZeroNum+taosize);

				//float *d_m_sum_re,*d_m_sum_im;
				
				//20210331
				HANDLE_ERROR(hipMalloc((void** )&d_m_sum_re[j], maxsize * sizeof(float)));
				HANDLE_ERROR(hipMalloc((void** )&d_m_sum_im[j], maxsize * sizeof(float)));
				
				for (int ig = 0; ig <  maxsize; ig++) 
				{//wangying 20210308
					float tao0 = ig * (1 / fs);
					float wavelength = 2*PI*(fbeg+KO*tao0)/CSpeed;
					scalfuc(plan[j].d_effrays, SubAperturePlane[j].width * SubAperturePlane[j].height, ig + minZeroNum - 200, d_m_sum_re[j] + ig, d_m_sum_im[j] + ig, calcInfo.config, plan[j].d_center, New_receive_points[recv_index]);//20210331 20210831姬梓遇
                    //scalfuc(plan[j].d_effrays, SubAperturePlane[j].width * SubAperturePlane[j].height, ig + minZeroNum - 200, d_m_sum_re[j] + ig, d_m_sum_im[j] + ig, calcInfo.config, d_one_beam_result[j] + ig);//20210904snw   
					//printf("ig * sizeof(float): %d\n", ig * sizeof(float));

					//plan[j].d_m_sum_re[ig]=*plan[j].d_sum_sre;
					//plan[j].d_m_sum_im[ig]=*plan[j].d_sum_sim;
				}

				

			}
		
			s_sum_re = (float* )malloc(maxsize * sizeof(float) * calcInfo.config.card_num);//20200919 面积积分结果
			s_sum_im = (float* )malloc(maxsize * sizeof(float) * calcInfo.config.card_num);//20200919 面积积分结果	
			//20210331
			omp_set_num_threads(calcInfo.config.card_num);  // create as many CPU threads as there are CUDA devices
			#pragma omp parallel
			{
				int j = omp_get_thread_num();
				HANDLE_ERROR(hipSetDevice(calcInfo.config.select_device_list[j]));
		
				HANDLE_ERROR(hipMemcpy(s_sum_re+ j * maxsize, d_m_sum_re[j], maxsize * sizeof(float), hipMemcpyDeviceToHost));
				HANDLE_ERROR(hipMemcpy(s_sum_im+ j * maxsize, d_m_sum_im[j], maxsize * sizeof(float), hipMemcpyDeviceToHost));
				hipFree(d_m_sum_re[j]);
				hipFree(d_m_sum_im[j]);
			}	
			
            //simple_time simple_time_TS_compute;
            //simple_time_TS_compute.Time_Start();
            //合并每一块卡的积分结果
			comp all_rnt_sum = {0,0};
			for(int index=0;index<calcInfo.config.card_num;index++)
			{
				all_rnt_sum.re += rnt_sum[index].re;
				all_rnt_sum.im += rnt_sum[index].im;
			}

			//memset(result_1200_re, 0, maxsize * sizeof(float));
			//memset(result_1200_im, 0, maxsize * sizeof(float));
			
			free(rnt_sum);

		 	result_1200_re = (float*)malloc(maxsize * sizeof(float));
			memset(result_1200_re, 0, maxsize * sizeof(float));
			result_1200_im = (float*)malloc(maxsize * sizeof(float));
			memset(result_1200_im, 0, maxsize * sizeof(float));//20200920

			for (int ig =  0; ig <  maxsize; ig++)//wangying 20210308 
			{
				float all_s_sum_re = 0;
				float all_s_sum_im = 0;
				for (int index = 0; index < calcInfo.config.card_num; index++)
				{
					all_s_sum_re += s_sum_re[index * maxsize + ig ];
					all_s_sum_im += s_sum_im[index * maxsize + ig ];
				}
				result_1200_re[ig] = all_s_sum_re;
				result_1200_im[ig] = all_s_sum_im;
			}



			free(s_sum_re);
			free(s_sum_im);


			result_1200 = (float*)malloc( totalsize* sizeof(float));//20210402
			memset(result_1200, 0, totalsize * sizeof(float));
			for(int ig =  0; ig <  maxsize; ig++){
				result_1200[ig+minZeroNum-200]=result_1200_re[ig];//20220719 考虑-200
				}




			for (int k = 0; k < totalsize; k++)//20210402
			{
				//fprintf(fileresult_1200, "%d	%d	%e	%e\n", i,k, result_1200_re[k], result_1200_im[k]);
				fprintf(fileresult_1200, "%e\n", result_1200[k]);
			}
			
			fclose(fileresult_1200);
            //计算TS值
			float result = TS_compute(far_dis,all_rnt_sum, lmd);
            
            float time_TS_compute = simple_time_TS_compute.Time_End();
			
			float maxcalcTime = -1;
            for(int index=0;index<calcInfo.config.card_num;index++)
			{
				if(calcTime[index]>maxcalcTime)
                    maxcalcTime = calcTime[index];
			}
            
            float time_all = maxcalcTime+time_TS_compute;
            
            
            //计算每一个三角面片的积分结果，用于伪彩图显示
			memset(pre_triangle_result,0,sizeof(float)*calcInfo.triangles_length);
			for (int j = 0; j < calcInfo.config.card_num; j++)
			{
				for (int index = 0; index < calcInfo.triangles_length; index++)
				{
					pre_triangle_result_reim[2*index] += h_TSOfPerTriangle[j][index].re;
					pre_triangle_result_reim[2*index+1] += h_TSOfPerTriangle[j][index].im;
					if (h_TSOfPerTriangle[j][index].re != 0 || h_TSOfPerTriangle[j][index].im != 0)
					{				
						pre_triangle_result[index] += sqrt(h_TSOfPerTriangle[j][index].re*h_TSOfPerTriangle[j][index].re + h_TSOfPerTriangle[j][index].im*h_TSOfPerTriangle[j][index].im);						
					}
				}
			}
			
			// 将各卡计算时间存储成二叉树结构，用于调整下一度子孔径面划分
			PreAngelTime = ConstructTimeTree(AperturePlane, calcTime, calcInfo.config.card_num);
		
			CalcResult calcResult;
			calcResult.angle = angle;
			calcResult.calc_time = time_all;
			calcResult.freq = f;
			calcResult.raysnum = pre_device_width[i] * height[i] + (pre_device_width[i] + 1)*(height[i] + 1);
			calcResult.squarenum = pre_device_width[i] * height[i];
			calcResult.TS = result;
			calcResult.height = height[i];
			calcResult.width = pre_device_width[i];
			calcResult.recvIdx = recv_index;

            fprintf(fileresult_TS,"%e\n",calcResult.TS);  //snw

			Frame frame;
			strcpy(frame.command, "CalcResult");
			frame.length = sizeof(CalcResult);
			memcpy(frame.data, &calcResult, sizeof(CalcResult));
            //计算结果发送到客户端
			send_frame(socketClient, (char*)&frame, sizeof(frame));

			int sendedLength = 0;
            //将每一个三角面片的积分结果模值发送到客户端
			while (true)
			{
				strcpy(frame.command, "PreTriangleResult");
				if (sendedLength + 1024 < calcInfo.triangles_length*sizeof(float))
				{
					memcpy(frame.data, (char*)pre_triangle_result + sendedLength, 1024);
					frame.length = 1024;
					send_frame(socketClient, (char*)&frame, sizeof(Frame));
					sendedLength += 1024;
				}
				else
				{
					memcpy(frame.data, (char*)pre_triangle_result + sendedLength, calcInfo.triangles_length*sizeof(float) - sendedLength);
					frame.length = calcInfo.triangles_length*sizeof(float) - sendedLength;
					send_frame(socketClient, (char*)&frame, sizeof(Frame));
					break;
				}
			 }
			//将每一个三角面片的积分结果（实部虚部）发送到客户端
			sendedLength = 0;
			while (true)
			{
				strcpy(frame.command, "TriangleResultReIm");
				if (sendedLength + 1024 < 2*calcInfo.triangles_length*sizeof(float))
				{
					memcpy(frame.data, (char*)pre_triangle_result_reim + sendedLength, 1024);
					frame.length = 1024;
					send_frame(socketClient, (char*)&frame, sizeof(Frame));
					sendedLength += 1024;
				}
				else
				{
					memcpy(frame.data, (char*)pre_triangle_result_reim + sendedLength, 2*calcInfo.triangles_length*sizeof(float) - sendedLength);
					frame.length = 2*calcInfo.triangles_length*sizeof(float) - sendedLength;
					send_frame(socketClient, (char*)&frame, sizeof(Frame));
					break;
				}
			 }
			//将时域积分结果发送回客户端  姬梓遇20210913
			sendedLength = 0;
			while (true)
			{
				strcpy(frame.command, "re");
				if (sendedLength + 1024 < totalsize*sizeof(float))
				{
					memcpy(frame.data, (char*)result_1200 + sendedLength, 1024);
					frame.length = 1024;
					send_frame(socketClient, (char*)&frame, sizeof(Frame));
					sendedLength += 1024;
				}
				else
				{
					memcpy(frame.data, (char*)result_1200 + sendedLength, totalsize*sizeof(float) - sendedLength);
					frame.length = totalsize*sizeof(float) - sendedLength;
					send_frame(socketClient, (char*)&frame, sizeof(Frame));
					break;
				}
			}




			printf("%d\t%d\t%d\t%f\t%d\t%d\t%f\t%f\n", i, pre_device_width[i], height[i], calcInfo.config.pipe_size, 0, 0, result, time_all);
			//避免线程在其他地方cancel，以保证上位机能正常暂停计算 2022.3.24 jzy
			pthread_setcancelstate(PTHREAD_CANCEL_ENABLE, NULL);
			angle ++;          
			}
		
		}
		start_alpha = calcInfo.config.start_alpha;
		printf("第%d个阵元计算完毕\n",recv_index);
	}
    fclose(fileresult_TS);
	free(result_1200_re);
	free(result_1200_im);
	free(result_1200);
	//hipFree(d_s_sum_re);
	//hipFree(d_s_sum_re);
	//fclose(fileresult_1200);//20200919
	//hipFree(plan.d_m_sum_re);
	//hipFree(plan.d_m_sum_im);


	omp_set_num_threads(calcInfo.config.card_num);  // create as many CPU threads as there are CUDA devices
	#pragma omp parallel
	{
		int i = omp_get_thread_num();
		HANDLE_ERROR(hipSetDevice(calcInfo.config.select_device_list[i]));

		FreeOnGPU(plan[i].d_rays1, plan[i].d_squares1, plan[i].d_rays2, plan[i].d_squares2, 
			plan[i].d_effrays, plan[i].d_center, plan[i].d_axis, plan[i].d_transMat, plan[i].d_reim, plan[i].d_sum_re, plan[i].d_sum_im, plan[i].d_sum_sre, plan[i].d_sum_sim,
			plan[i].d_DivRayTubeNum, plan[i].d_sum_gmem, plan[i].d_sum_Gmem, plan[i].d_squares_pred);

		free_data(plan[i].d_out_array, plan[i].d_tree, plan[i].d_points, plan[i].d_triangles);
	}

	//host Free

	if(plan!=NULL)
		free(plan);
	if(pre_device_height!=NULL)
		free(pre_device_height);
	if(pre_device_width!=NULL)
		free(pre_device_width);
	if(height!=NULL)
		free(height);
	if(e_fi_max!=NULL)
		free(e_fi_max);
	if(e_st_min!=NULL)
		free(e_st_min);
	if(h_out_array!=NULL)
		free(h_out_array);
	if(h_tree!=NULL)
		free(h_tree);
	
	if(h_reim!=NULL)
	{
		for(int i=0;i<calcInfo.config.card_num;i++)
			free(h_reim[i]);
		free(h_reim);
	}
	if(h_TSOfPerTriangle!=NULL)
	{
		for(int i=0;i<calcInfo.config.card_num;i++)
			free(h_TSOfPerTriangle[i]);
		free(h_TSOfPerTriangle);
	}
	if(pre_triangle_result!=NULL)
		free(pre_triangle_result);
	if(pre_triangle_result_reim!=NULL)
		free(pre_triangle_result_reim);
	if(SubAperturePlane!=NULL)
		free(SubAperturePlane);	
	if(AperturePlane!=NULL)
		free(AperturePlane);
	if(PreAngelTime!=NULL)
	{
		for(int i=0;i<2*calcInfo.config.card_num-1;i++)
			free(PreAngelTime[i]);
		free(PreAngelTime);
	}		
	if(New_receive_points!=NULL)
		free(New_receive_points);
	if(h_points!=NULL)
		free(h_points);
	Frame frame;
	strcpy(frame.command, "CalcOver");
	frame.length = 0;

	send_frame(socketClient, (char*)&frame, sizeof(frame));
	printf("calc over\n");
	pthread_cleanup_pop(0);


	return NULL;

}
