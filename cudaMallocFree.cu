#include "simple_time.h"
#include "raystrace.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "tree2vector.h"
#include <math.h>
#include <stdio.h>
#include "handlerror.h"
#include "virtualface_gpu.h"
#include"integral_gpu.h"

void MallocOnGPU(int dwidth, int dheight, Direction** d_rays1, Square** d_squares1, Direction** d_rays2, Square** d_squares2,
	RayBeamInfo** d_effrays, Vector** d_center, Vector** d_axis, MatStruct** d_transMat, ReimOutput** d_reim,
	float** d_sum_re, float** d_sum_im, float** d_sum_sre,float** d_sum_sim,int** d_DivRayTubeNum, int** d_sum_gmem, int** d_sum_Gmem, int** d_squares_pred)
{
	//printf("dwidth_malloc=%d, dheight_malloc=%d\n",dwidth,dheight);

	HANDLE_ERROR(hipMalloc((void**)d_rays1, sizeof(Direction)*((dwidth + 1)*(dheight + 1) + dwidth*dheight)));//0.82GB//4850000
	HANDLE_ERROR(hipMalloc((void**)d_rays2, sizeof(Direction)*((dwidth + 1)*(dheight + 1) + dwidth*dheight+4850000)));//0.82GB
	HANDLE_ERROR(hipMalloc((void**)d_squares1, sizeof(Square)*dwidth*dheight));//0.46GB
	HANDLE_ERROR(hipMalloc((void**)d_squares2, sizeof(Square)*dwidth*dheight));//0.46GB

	//printf("sizeof d_rays1:%d\n",sizeof(Direction)*((dwidth + 1)*(dheight + 1) + dwidth*dheight));\
	//printf("sizeof d_squares1:%d\n",sizeof(Square)*dwidth*dheight);

	int raysBeamNum = dwidth * dheight;
	//printf("raysBeamNum malloc = %d\n",raysBeamNum);
	//printf("raysNum malloc = %d\n",(dwidth + 1)*(dheight + 1) + dwidth*dheight);
	dim3 blockSize((raysBeamNum + 511)/ 512, 1, 1);
	
	//3.12GB
	HANDLE_ERROR(hipMalloc((void**)d_effrays, raysBeamNum * sizeof(RayBeamInfo)));//1.64GB
	HANDLE_ERROR(hipMalloc((void**)d_center, raysBeamNum * sizeof(Vector)));//0.11GB
	HANDLE_ERROR(hipMalloc((void**)d_axis, 3 * raysBeamNum * sizeof(Vector)));//0.33GB
	HANDLE_ERROR(hipMalloc((void**)d_transMat, raysBeamNum * sizeof(MatStruct)));//0.45GB
	HANDLE_ERROR(hipMalloc((void**)d_reim, raysBeamNum * sizeof(ReimOutput)));//0.075GB
	HANDLE_ERROR(hipMalloc((void**)d_sum_re, blockSize.x * sizeof(float)));//0.076MB
	HANDLE_ERROR(hipMalloc((void**)d_sum_im, blockSize.x * sizeof(float)));//0.076MB
	HANDLE_ERROR(hipMalloc((void**)d_sum_sre, blockSize.x * sizeof(float)));//0.076MB
	HANDLE_ERROR(hipMalloc((void**)d_sum_sim, blockSize.x * sizeof(float)));//0.076MB

	HANDLE_ERROR(hipMalloc((void**)d_DivRayTubeNum, blockSize.x * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)d_sum_gmem, 35 * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)d_sum_Gmem, 35 * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)d_squares_pred, raysBeamNum * sizeof(int)));

	/*HANDLE_ERROR(hipMemset(*d_effrays, 0, raysBeamNum * sizeof(RayBeamInfo)));
	HANDLE_ERROR(hipMemset(*d_center, 0, raysBeamNum * sizeof(Vector)));
	HANDLE_ERROR(hipMemset(*d_axis, 0, 3 * raysBeamNum * sizeof(Vector)));
	HANDLE_ERROR(hipMemset(*d_transMat, 0, raysBeamNum * sizeof(MatStruct)));
	HANDLE_ERROR(hipMemset(*d_reim, 0, raysBeamNum * sizeof(comp)));
	HANDLE_ERROR(hipMemset(*d_sum_re, 0, blockSize.x * sizeof(float)));
	HANDLE_ERROR(hipMemset(*d_sum_im, 0, blockSize.x * sizeof(float)));
	HANDLE_ERROR(hipMemset(*d_DivRayTubeNum, 0, blockSize.x * sizeof(int)));
	HANDLE_ERROR(hipMemset(*d_sum_gmem, 0, 35 * sizeof(int)));
	HANDLE_ERROR(hipMemset(*d_sum_Gmem, 0, 35 * sizeof(int)));
	HANDLE_ERROR(hipMemset(*d_squares_pred, 0, raysBeamNum * sizeof(int)));*/
}

void MemsetOnGPU1(int dwidth, int dheight, Direction** d_rays1, Square** d_squares1, RayBeamInfo** d_effrays, Vector** d_center, Vector** d_axis, 
	MatStruct** d_transMat, ReimOutput** d_reim, float** d_sum_re, float** d_sum_im,
	int** d_DivRayTubeNum, int** d_sum_gmem, int** d_sum_Gmem, int** d_squares_pred)
{
	int raysBeamNum = dwidth * dheight;
	dim3 threadSize(512, 1, 1);
	dim3 blockSize(raysBeamNum / 512 + 1, 1, 1);

	HANDLE_ERROR(hipMemset(*d_squares1, 0, dwidth*dheight * sizeof(Square)));
	HANDLE_ERROR(hipMemset(*d_rays1, 0, ((dwidth + 1)*(dheight + 1) + dwidth*dheight) * sizeof(Direction)));
	HANDLE_ERROR(hipMemset(*d_effrays, 0, raysBeamNum * sizeof(RayBeamInfo)));
	HANDLE_ERROR(hipMemset(*d_center, 0, raysBeamNum * sizeof(Vector)));
	HANDLE_ERROR(hipMemset(*d_axis, 0, 3 * raysBeamNum * sizeof(Vector)));
	HANDLE_ERROR(hipMemset(*d_transMat, 0, raysBeamNum * sizeof(MatStruct)));
	HANDLE_ERROR(hipMemset(*d_reim, 0, raysBeamNum * sizeof(ReimOutput)));
	HANDLE_ERROR(hipMemset(*d_sum_re, 0, blockSize.x * sizeof(float)));
	HANDLE_ERROR(hipMemset(*d_sum_im, 0, blockSize.x * sizeof(float)));

	HANDLE_ERROR(hipMemset(*d_DivRayTubeNum, 0, blockSize.x * sizeof(int)));
	HANDLE_ERROR(hipMemset(*d_sum_gmem, 0, 35 * sizeof(int)));
	HANDLE_ERROR(hipMemset(*d_sum_Gmem, 0, 35 * sizeof(int)));
	HANDLE_ERROR(hipMemset(*d_squares_pred, 0, raysBeamNum * sizeof(int)));
}

void MemsetOnGPU2(int dwidth, int dheight, Direction** d_rays2, Square** d_squares2, RayBeamInfo** d_effrays, Vector** d_center, Vector** d_axis, 
	MatStruct** d_transMat, ReimOutput** d_reim, float** d_sum_re, float** d_sum_im,
	int** d_DivRayTubeNum, int** d_sum_gmem, int** d_sum_Gmem, int** d_squares_pred)
{
	int raysBeamNum = dwidth * dheight;
	dim3 threadSize(512, 1, 1);
	dim3 blockSize(raysBeamNum / 512 + 1, 1, 1);

	HANDLE_ERROR(hipMemset(*d_squares2, 0, dwidth*dheight * sizeof(Square)));
	HANDLE_ERROR(hipMemset(*d_rays2, 0, ((dwidth + 1)*(dheight + 1) + dwidth*dheight+4850000) * sizeof(Direction)));
	HANDLE_ERROR(hipMemset(*d_effrays, 0, raysBeamNum * sizeof(RayBeamInfo)));
	HANDLE_ERROR(hipMemset(*d_center, 0, raysBeamNum * sizeof(Vector)));
	HANDLE_ERROR(hipMemset(*d_axis, 0, 3 * raysBeamNum * sizeof(Vector)));
	HANDLE_ERROR(hipMemset(*d_transMat, 0, raysBeamNum * sizeof(MatStruct)));
	HANDLE_ERROR(hipMemset(*d_reim, 0, raysBeamNum * sizeof(ReimOutput)));
	HANDLE_ERROR(hipMemset(*d_sum_re, 0, blockSize.x * sizeof(float)));
	HANDLE_ERROR(hipMemset(*d_sum_im, 0, blockSize.x * sizeof(float)));
	HANDLE_ERROR(hipMemset(*d_DivRayTubeNum, 0, blockSize.x * sizeof(int)));
	HANDLE_ERROR(hipMemset(*d_sum_gmem, 0, 35 * sizeof(int)));
	HANDLE_ERROR(hipMemset(*d_sum_Gmem, 0, 35 * sizeof(int)));
	HANDLE_ERROR(hipMemset(*d_squares_pred, 0, raysBeamNum * sizeof(int)));
}

void MemsetOnGPU3(int dwidth, int dheight, RayBeamInfo** d_effrays, Vector** d_center, Vector** d_axis, MatStruct** d_transMat, ReimOutput** d_reim, float** d_sum_re, float** d_sum_im)
{
	int raysBeamNum = dwidth * dheight;
	dim3 threadSize(512, 1, 1);
	dim3 blockSize(raysBeamNum / 512 + 1, 1, 1);

	HANDLE_ERROR(hipMemset(*d_effrays, 0, raysBeamNum * sizeof(RayBeamInfo)));
	HANDLE_ERROR(hipMemset(*d_center, 0, raysBeamNum * sizeof(Vector)));
	HANDLE_ERROR(hipMemset(*d_axis, 0, 3 * raysBeamNum * sizeof(Vector)));
	HANDLE_ERROR(hipMemset(*d_transMat, 0, raysBeamNum * sizeof(MatStruct)));
	HANDLE_ERROR(hipMemset(*d_reim, 0, raysBeamNum * sizeof(ReimOutput)));
	HANDLE_ERROR(hipMemset(*d_sum_re, 0, blockSize.x * sizeof(float)));
	HANDLE_ERROR(hipMemset(*d_sum_im, 0, blockSize.x * sizeof(float)));
}

void MemsetOnGPU(int dwidth, int dheight, Direction** d_rays1, Square** d_squares1, Direction** d_rays2, Square** d_squares2,
	RayBeamInfo** d_effrays, Vector** d_center, Vector** d_axis, MatStruct** d_transMat, ReimOutput** d_reim,
	float** d_sum_re, float** d_sum_im, float** d_sum_sre,float** d_sum_sim, int** d_DivRayTubeNum, int** d_sum_gmem, int** d_sum_Gmem, int** d_squares_pred)
{
	int raysBeamNum = dwidth * dheight;
	dim3 threadSize(512, 1, 1);
	dim3 blockSize(raysBeamNum / 512 + 1, 1, 1);

	HANDLE_ERROR(hipMemset(*d_squares2, 0, dwidth*dheight * sizeof(Square)));
	HANDLE_ERROR(hipMemset(*d_rays2, 0, ((dwidth + 1)*(dheight + 1) + dwidth*dheight+4850000) * sizeof(Direction)));
	HANDLE_ERROR(hipMemset(*d_squares1, 0, dwidth*dheight * sizeof(Square)));
	HANDLE_ERROR(hipMemset(*d_rays1, 0, ((dwidth + 1)*(dheight + 1) + dwidth*dheight) * sizeof(Direction)));
	HANDLE_ERROR(hipMemset(*d_effrays, 0, raysBeamNum * sizeof(RayBeamInfo)));
	HANDLE_ERROR(hipMemset(*d_center, 0, raysBeamNum * sizeof(Vector)));
	HANDLE_ERROR(hipMemset(*d_axis, 0, 3 * raysBeamNum * sizeof(Vector)));
	HANDLE_ERROR(hipMemset(*d_transMat, 0, raysBeamNum * sizeof(MatStruct)));
	HANDLE_ERROR(hipMemset(*d_reim, 0, raysBeamNum * sizeof(ReimOutput)));
	HANDLE_ERROR(hipMemset(*d_sum_re, 0, blockSize.x * sizeof(float)));
	HANDLE_ERROR(hipMemset(*d_sum_im, 0, blockSize.x * sizeof(float)));
	HANDLE_ERROR(hipMemset(*d_sum_sre, 0, blockSize.x * sizeof(float)));//20200919
HANDLE_ERROR(hipMemset(*d_sum_sim, 0, blockSize.x * sizeof(float)));//20200919
	HANDLE_ERROR(hipMemset(*d_DivRayTubeNum, 0, blockSize.x * sizeof(int)));
	HANDLE_ERROR(hipMemset(*d_sum_gmem, 0, 35 * sizeof(int)));
	HANDLE_ERROR(hipMemset(*d_sum_Gmem, 0, 35 * sizeof(int)));
	HANDLE_ERROR(hipMemset(*d_squares_pred, 0, raysBeamNum * sizeof(int)));
}

void free_virtualface(Direction *rays, Square *squares)
{
	hipFree(rays);
	hipFree(squares);
}

void FreeOnGPU(Direction* d_rays1, Square* d_squares1, Direction* d_rays2, Square* d_squares2, RayBeamInfo* d_effrays, Vector* d_center,
	Vector* d_axis, MatStruct* d_transMat, ReimOutput* d_reim, float* d_sum_re, float* d_sum_im, float* d_sum_sre,float* d_sum_sim, int* d_DivRayTubeNum, int* d_sum_gmem, int* d_sum_Gmem, int* d_squares_pred)
{
	hipFree(d_rays1);
	hipFree(d_rays2);
	hipFree(d_squares1);
	hipFree(d_squares2);
	
	hipFree(d_center);
	hipFree(d_axis);
	hipFree(d_transMat);
	hipFree(d_effrays);
	hipFree(d_reim);
	hipFree(d_sum_re);
	hipFree(d_sum_im);
	hipFree(d_sum_sre);
hipFree(d_sum_sim);
	hipFree(d_DivRayTubeNum);
	hipFree(d_sum_gmem);
	hipFree(d_sum_Gmem);
	hipFree(d_squares_pred);
}

void free_data(Prim_Box *d_array, KD_Node_V *d_root, Element *d_points, Triangle *d_triangles)
{
	hipFree(d_array);
	hipFree(d_root);
	hipFree(d_points);
	hipFree(d_triangles);
}

void Destroy_Tree(KD_Node *root)
{
	if (root)
	{
		Destroy_Tree(root->LeftChild);
		Destroy_Tree(root->RightChild);
	}
	free(root);
	root = NULL;
}
