﻿#include "recvThreadFunction.h"

#include <stdio.h>
#include <semaphore.h>
#include <pthread.h>
#include<signal.h>
#include <unistd.h>
#include <string.h>
#include <malloc.h>
#include "calcThreadFunction.h"
#include "common_struct.h"
#include "socketFunctions.h"
extern pthread_mutex_t socket_mutex;//lihao 20240711传输锁
void *recvThreadFunction(void *argv)
{
	CalcInfo calcInfo;                //解算参数配置
	int socketClient = *(int*)argv;   //接收套接字
    
	calcInfo.socket = socketClient; 

	pthread_t calcThread;             //计算线程    
	memset(&calcThread, 0, sizeof(pthread_t));
    
    //循环接收数据
	while (true)
	{
		Frame frame;
        //读取一个数据帧
		recv_data(socketClient, (char*)&frame, sizeof(Frame));

		if (!strcmp(frame.command, "Triangles"))
		{

			//接收triangles数据
			if(calcInfo.triangles != NULL)
			{
				free(calcInfo.triangles);
			}
			printf("triangles data recv start!\n");
			calcInfo.triangles_length = frame.length / sizeof(Triangle);
			calcInfo.triangles = (Triangle *)malloc(frame.length);
			recv_data(socketClient, (char*)calcInfo.triangles, frame.length);
			printf("triangles data recv over!\n");
		}
		if (!strcmp(frame.command, "Elements"))
		{
			//接收Element数据
			if(calcInfo.points != NULL)
			{
				free(calcInfo.points);
			}
			printf("elements data recv start!\n");
			calcInfo.points_length = frame.length / sizeof(Element);
			calcInfo.points = (Element *)malloc(frame.length);
			recv_data(socketClient, (char*)calcInfo.points, frame.length);
			printf("elements data recv over!\n");

			//数据接收完毕后，发送等待配置帧
			Frame frame;
			strcpy(frame.command, "WaitForConfig");
			frame.length = 0;
#ifdef linux
			pthread_mutex_lock(&socket_mutex);//lihao 20240711 
#endif
			send_frame(socketClient, (char*)&frame, sizeof(Frame));
#ifdef linux
			pthread_mutex_unlock(&socket_mutex);//lihao 20240711  发送数据时一直占有锁
#endif
		}
		if (!strcmp(frame.command, "RecvPoints"))
		{

			//接收RecvPoints数据
			if(calcInfo.recvPoints != NULL)
			{
				free(calcInfo.recvPoints);
			}
			printf("recvPoints data recv start!\n");
			calcInfo.recvPoints = (Axis_slx *)malloc(frame.length);
			recv_data(socketClient, (char*)calcInfo.recvPoints, frame.length);
			printf("recvPoints data recv over!\n");
		}
		if (!strcmp(frame.command, "Configuration"))
		{
			//配置并开始
			memcpy(&calcInfo.config, frame.data, sizeof(ConfigStruct));

			//开始执行计算线程，当计算线程正在执行时，则先关闭线程后再重新执行
			if (calcThread!=0 && pthread_kill(calcThread, 0) == 0)
			{
				pthread_cancel(calcThread);
				pthread_join(calcThread,NULL);
				printf("restart");
			}
			pthread_create(&calcThread, NULL, calcThreadFunction, (void*)&calcInfo);
		}
		if (!strcmp(frame.command, "Stop"))
		{
            //停止命令，退出线程
			if (calcThread!=0 && pthread_kill(calcThread, 0) == 0)
			{
				pthread_cancel(calcThread);
				pthread_join(calcThread,NULL);
			}
			//2020.3.24 jzy
			memset(&calcThread, 0, sizeof(pthread_t));
		}
		if (!strcmp(frame.command, "Exit"))
		{
            //断开连接，退出接收循环
			if (calcThread!=0 && pthread_kill(calcThread, 0) == 0)
			{
				pthread_cancel(calcThread);
				pthread_join(calcThread,NULL);
			}
			break;
		}
	}
	return NULL;
}
