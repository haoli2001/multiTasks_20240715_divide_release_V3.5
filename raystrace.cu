#include "hip/hip_runtime.h"
#include "common_struct.h"
#include <math.h>
#include <stdio.h>
#include "handlerror.h"
#include "raystrace.h"

#define TPB 128
__device__ int count1[181] = { 0 };
__device__ int count2[181] = { 0 };
__device__ int count3[181] = { 0 };

__device__
bool RayIntersectBox_In(KD_Node_V *node, Direction *ray, float *intersect_point)
{
	float ptOnPlane[3];                //射线与包围盒某一个面的交点
	float *minPoint = node->box.bmin;  //包围盒的最小点
	float *maxPoint = node->box.bmax;  //包围盒的最大点

	float3 origin = ray->p;
	float3 dir = ray->dir;

	float t;
	if (dir.x != 0.f)
	{
		if (dir.x > 0)
			t = (minPoint[0] - origin.x) / dir.x;
		else
			t = (maxPoint[0] - origin.x) / dir.x;
		if (t >= 0.f)
		{
			ptOnPlane[0] = origin.x + t*dir.x;
			ptOnPlane[1] = origin.y + t*dir.y;
			ptOnPlane[2] = origin.z + t*dir.z;
			if (minPoint[1] <= ptOnPlane[1] && ptOnPlane[1] <= maxPoint[1] && minPoint[2] <= ptOnPlane[2] && ptOnPlane[2] <= maxPoint[2])
			{
				intersect_point[0] = ptOnPlane[0];
				intersect_point[1] = ptOnPlane[1];
				intersect_point[2] = ptOnPlane[2];
				return true;
			}
		}
	}
	if (dir.y != 0.f)
	{
		if (dir.y > 0)
			t = (minPoint[1] - origin.y) / dir.y;
		else
			t = (maxPoint[1] - origin.y) / dir.y;
		if (t >= 0.f)
		{
			ptOnPlane[0] = origin.x + t*dir.x;
			ptOnPlane[1] = origin.y + t*dir.y;
			ptOnPlane[2] = origin.z + t*dir.z;
			if (minPoint[2] <= ptOnPlane[2] && ptOnPlane[2] <= maxPoint[2] && minPoint[0] <= ptOnPlane[0] && ptOnPlane[0] <= maxPoint[0])
			{
				intersect_point[0] = ptOnPlane[0];
				intersect_point[1] = ptOnPlane[1];
				intersect_point[2] = ptOnPlane[2];
				return true;
			}
		}

	}
	if (dir.z != 0.f)
	{
		if (dir.z > 0)
			t = (minPoint[2] - origin.z) / dir.z;
		else
			t = (maxPoint[2] - origin.z) / dir.z;
		if (t >= 0.f)
		{
			ptOnPlane[0] = origin.x + t*dir.x;
			ptOnPlane[1] = origin.y + t*dir.y;
			ptOnPlane[2] = origin.z + t*dir.z;
			if (minPoint[1] <= ptOnPlane[1] && ptOnPlane[1] <= maxPoint[1] && minPoint[0] <= ptOnPlane[0] && ptOnPlane[0] <= maxPoint[0])
			{
				intersect_point[0] = ptOnPlane[0];
				intersect_point[1] = ptOnPlane[1];
				intersect_point[2] = ptOnPlane[2];
				return true;
			}
		}
	}
	return false;
}

__device__
bool RayIntersectBox_Out(KD_Node_V *node, Direction *ray, float *intersect_point, int *whichface)
{
	int front_or_behind = -1;
	float ptOnPlane[3];                   //射线与包围盒某一个面的交点
	float *minPoint = node->box.bmin;     //包围盒的最小点
	float *maxPoint = node->box.bmax;     //包围盒的最大点

	float3 origin = ray->p;
	float3 dir = ray->dir;

	float t;
	if (dir.x != 0.f)
	{
        //如果找出口点的话,t为远点处的t
		if (dir.x > 0)
		{
			t = (maxPoint[0] - origin.x) / dir.x;
			front_or_behind = 1;
		}
		else
		{
			t = (minPoint[0] - origin.x) / dir.x;
			front_or_behind = 0;
		}
		if (t >= 0.f)
		{
			ptOnPlane[0] = origin.x + t*dir.x;
			ptOnPlane[1] = origin.y + t*dir.y;
			ptOnPlane[2] = origin.z + t*dir.z;
			if (minPoint[1] <= ptOnPlane[1] && ptOnPlane[1] <= maxPoint[1] && minPoint[2] <= ptOnPlane[2] && ptOnPlane[2] <= maxPoint[2])
			{
				intersect_point[0] = ptOnPlane[0];
				intersect_point[1] = ptOnPlane[1];
				intersect_point[2] = ptOnPlane[2];
				*whichface = 0 + front_or_behind;
				return true;
			}
		}
	}
	if (dir.y != 0.f)
	{
		if (dir.y > 0)
		{
			t = (maxPoint[1] - origin.y) / dir.y;
			front_or_behind = 1;
		}
		else
		{
			t = (minPoint[1] - origin.y) / dir.y;
			front_or_behind = 0;
		}
		if (t >= 0.f)
		{
			ptOnPlane[0] = origin.x + t*dir.x;
			ptOnPlane[1] = origin.y + t*dir.y;
			ptOnPlane[2] = origin.z + t*dir.z;
			if (minPoint[2] <= ptOnPlane[2] && ptOnPlane[2] <= maxPoint[2] && minPoint[0] <= ptOnPlane[0] && ptOnPlane[0] <= maxPoint[0])
			{
				intersect_point[0] = ptOnPlane[0];
				intersect_point[1] = ptOnPlane[1];
				intersect_point[2] = ptOnPlane[2];
				*whichface = 2 + front_or_behind;
				return true;
			}
		}

	}
	if (dir.z != 0.f)
	{
		if (dir.z > 0)
		{
			t = (maxPoint[2] - origin.z) / dir.z;
			front_or_behind = 1;
		}
		else
		{
			t = (minPoint[2] - origin.z) / dir.z;
			front_or_behind = 0;
		}
		if (t >= 0.f)
		{
			ptOnPlane[0] = origin.x + t*dir.x;
			ptOnPlane[1] = origin.y + t*dir.y;
			ptOnPlane[2] = origin.z + t*dir.z;
			if (minPoint[1] <= ptOnPlane[1] && ptOnPlane[1] <= maxPoint[1] && minPoint[0] <= ptOnPlane[0] && ptOnPlane[0] <= maxPoint[0])
			{
				intersect_point[0] = ptOnPlane[0];
				intersect_point[1] = ptOnPlane[1];
				intersect_point[2] = ptOnPlane[2];
				*whichface = 4 + front_or_behind;
				return true;
			}
		}
	}
	return false;
}

__device__
bool RayIntersectTriangle(Direction *ray, float *P0, float *P1, float *P2, float *distance, float *intersect_point)
{
	//三角形法线  N=(P1-P0)x(P2-P0)
	float N[3];
	N[0] = (P1[1] - P0[1])*(P2[2] - P0[2]) - (P2[1] - P0[1])*(P1[2] - P0[2]);
	N[1] = (P1[2] - P0[2])*(P2[0] - P0[0]) - (P2[2] - P0[2])*(P1[0] - P0[0]);
	N[2] = (P1[0] - P0[0])*(P2[1] - P0[1]) - (P2[0] - P0[0])*(P1[1] - P0[1]);

	//三角形平面到原点距离 ，平面方程 N*P+d=0;
	float d = (-N[0] * P0[0]) + (-N[1] * P0[1]) + (-N[2] * P0[2]);
	if (N[0] * ray->dir.x + N[1] * ray->dir.y + N[2] * ray->dir.z == 0)
	{
		return false;
	}

	//射线方程 O+t*dir=P(t);
	float t = (-d - (N[0] * ray->p.x + N[1] * ray->p.y + N[2] * ray->p.z)) /
		(N[0] * ray->dir.x + N[1] * ray->dir.y + N[2] * ray->dir.z);
	if (t <= 0)
		return false;

	//射线与三角形平面交点
	float P_intersect[3];
	P_intersect[0] = ray->p.x + t*ray->dir.x;
	P_intersect[1] = ray->p.y + t*ray->dir.y;
	P_intersect[2] = ray->p.z + t*ray->dir.z;

	//计算交点是否在三角形内，参考博客https://blog.csdn.net/ZJU_fish1996/article/details/52276987
	float R[3];
	R[0] = P_intersect[0] - P0[0]; R[1] = P_intersect[1] - P0[1]; R[2] = P_intersect[2] - P0[2];
	float Q1[3];
	Q1[0] = P1[0] - P0[0]; Q1[1] = P1[1] - P0[1]; Q1[2] = P1[2] - P0[2];
	float Q2[3];
	Q2[0] = P2[0] - P0[0]; Q2[1] = P2[1] - P0[1]; Q2[2] = P2[2] - P0[2];

	float fm = (Q1[0] * Q1[0] + Q1[1] * Q1[1] + Q1[2] * Q1[2])*(Q2[0] * Q2[0] + Q2[1] * Q2[1] + Q2[2] * Q2[2]) -
		(Q1[0] * Q2[0] + Q1[1] * Q2[1] + Q1[2] * Q2[2])*(Q1[0] * Q2[0] + Q1[1] * Q2[1] + Q1[2] * Q2[2]);

	float w1, w2;
	w1 = ((Q2[0] * Q2[0] + Q2[1] * Q2[1] + Q2[2] * Q2[2])*(R[0] * Q1[0] + R[1] * Q1[1] + R[2] * Q1[2]) -
		(Q1[0] * Q2[0] + Q1[1] * Q2[1] + Q1[2] * Q2[2])*(R[0] * Q2[0] + R[1] * Q2[1] + R[2] * Q2[2])) / fm;
	w2 = ((Q1[0] * Q1[0] + Q1[1] * Q1[1] + Q1[2] * Q1[2])*(R[0] * Q2[0] + R[1] * Q2[1] + R[2] * Q2[2]) -
		(Q1[0] * Q2[0] + Q1[1] * Q2[1] + Q1[2] * Q2[2])*(R[0] * Q1[0] + R[1] * Q1[1] + R[2] * Q1[2])) / fm;
        
    //备注 由于float精度的问题，会造成部分求交失败
	if (w1 >= 0 && w2 >= 0 && w1 + w2 <= 1)
	{

		intersect_point[0] = P_intersect[0];
		intersect_point[1] = P_intersect[1];
		intersect_point[2] = P_intersect[2];
		*distance = sqrt((ray->p.x - intersect_point[0])*(ray->p.x - intersect_point[0])
			+ (ray->p.y - intersect_point[1])*(ray->p.y - intersect_point[1])
			+ (ray->p.z - intersect_point[2])*(ray->p.z - intersect_point[2]));
	}
	else
	{
		return  false;
	}
	return true;
}

__device__
void RayReflectAndUpdateRayDir(Direction *ray, float *P0, float *P1, float *P2)
{
	//求法线
	float N[3];
	N[0] = (P1[1] - P0[1])*(P2[2] - P0[2]) - (P2[1] - P0[1])*(P1[2] - P0[2]);
	N[1] = (P1[2] - P0[2])*(P2[0] - P0[0]) - (P2[2] - P0[2])*(P1[0] - P0[0]);
	N[2] = (P1[0] - P0[0])*(P2[1] - P0[1]) - (P2[0] - P0[0])*(P1[1] - P0[1]);

	//法线归一化
	float NLen = sqrt(pow(N[0], 2) + pow(N[1], 2) + pow(N[2], 2));
	N[0] = N[0] / NLen;
	N[1] = N[1] / NLen;
	N[2] = N[2] / NLen;
	if (N[0] * ray->dir.x + N[1] * ray->dir.y + N[2] * ray->dir.z <= 0)
	{
		N[0] = -N[0]; N[1] = -N[1]; N[2] = -N[2];
	}
	float LxN = N[0] * ray->dir.x + N[1] * ray->dir.y + N[2] * ray->dir.z;
	ray->dir.x = ray->dir.x - 2 * LxN * N[0];
	ray->dir.y = ray->dir.y - 2 * LxN * N[1];
	ray->dir.z = ray->dir.z - 2 * LxN * N[2];
}



__device__
void SingleRayTrace(KD_Node_V *root, Direction *ray, Prim_Box *arrays, Element *points, Triangle *triangles, float water_line)
{
	float intersect_point[3];
    //与根节点求交失败
	if (false == RayIntersectBox_In(root, ray, intersect_point))
	{
		return;
	}
	KD_Node_V leaf_node = root[0];
	int max_cycle_count=1000;          //设置最大循环次数，防止未知原因造成死循环
	while (max_cycle_count--)
	{
        //循环遍历KDTree 直到找到底层包围盒
		while ((leaf_node.IsLeaf | leaf_node.IsEmpty)==false)
		{
			if ((intersect_point[leaf_node.Split_Axis] - leaf_node.SplitPos.point[leaf_node.Split_Axis])>0.000001)
			{
				leaf_node = root[leaf_node.RightIndex];
			}
			else /*if ((intersect_point[leaf_node.Split_Axis] - leaf_node.SplitPos.point[leaf_node.Split_Axis])<0.000001)*/
			{
				leaf_node = root[leaf_node.LeftIndex];
			}
		}
		
		//到达底层包围盒
		if (leaf_node.IsEmpty)
		{
			int whichface = -1;
            
			if (false == RayIntersectBox_Out(&leaf_node, ray, intersect_point, &whichface))
			{                     
                //从包围盒出去的时候出错了，理论不该有，防止由于精度等位置原因造成这种错误。
				return;
			}
			int out_index = leaf_node.RopeIndex[whichface];
			if (out_index != -1)
			{
                //空节点，穿过
				leaf_node = root[leaf_node.RopeIndex[whichface]];
			}
			else
			{
				//到达出口，退出
				return;
			}
		}
		else
		{
			//进入非空叶节点
			int calced_triangle_id = -1;
			while (ray->times <= 2)
			{
				float near_triangle_distance = 999999999;
				int intersect_triangle_id = -1;
				float current_intersect_point[3];
				for (int triIndex = leaf_node.begin; triIndex <= leaf_node.end; triIndex++)
				{
					if (triIndex == calced_triangle_id)
						continue;
					float P0[3], P1[3], P2[3];
					float intersect_point_on_triangle[3];
					float distance;
					P0[0] = points[triangles[arrays[triIndex].Box_Index].Points[0]].point[0];
					P1[0] = points[triangles[arrays[triIndex].Box_Index].Points[1]].point[0];
					P2[0] = points[triangles[arrays[triIndex].Box_Index].Points[2]].point[0];
					P0[1] = points[triangles[arrays[triIndex].Box_Index].Points[0]].point[1];
					P1[1] = points[triangles[arrays[triIndex].Box_Index].Points[1]].point[1];
					P2[1] = points[triangles[arrays[triIndex].Box_Index].Points[2]].point[1];
					P0[2] = points[triangles[arrays[triIndex].Box_Index].Points[0]].point[2];
					P1[2] = points[triangles[arrays[triIndex].Box_Index].Points[1]].point[2];
					P2[2] = points[triangles[arrays[triIndex].Box_Index].Points[2]].point[2];

					if (true == RayIntersectTriangle(ray, P0, P1, P2, &distance, intersect_point_on_triangle))
					{
                        //判断交点是否在包围盒内，如果不在，说明它在隔壁的包围盒，跳过
						if (intersect_point_on_triangle[0] >= leaf_node.box.bmin[0]
							&& intersect_point_on_triangle[0] <= leaf_node.box.bmax[0]   
							&& intersect_point_on_triangle[1] >= leaf_node.box.bmin[1]
							&& intersect_point_on_triangle[1] <= leaf_node.box.bmax[1]
							&& intersect_point_on_triangle[2] >= leaf_node.box.bmin[2]
							&& intersect_point_on_triangle[2] <= leaf_node.box.bmax[2])
						{
							if (0.01<distance && distance < near_triangle_distance)
							{
								near_triangle_distance = distance;
								intersect_triangle_id = triIndex;
								current_intersect_point[0] = intersect_point_on_triangle[0];
								current_intersect_point[1] = intersect_point_on_triangle[1];
								current_intersect_point[2] = intersect_point_on_triangle[2];
							}
						}
					}
				}
                //不等于-1的时候就说明射线与包围盒中的三角形有交点，然后就需要更新ray的原点，方向和据远点距离
				if (intersect_triangle_id != -1)
				{
					float P0[3], P1[3], P2[3];
					P0[0] = points[triangles[arrays[intersect_triangle_id].Box_Index].Points[0]].point[0];
					P1[0] = points[triangles[arrays[intersect_triangle_id].Box_Index].Points[1]].point[0];
					P2[0] = points[triangles[arrays[intersect_triangle_id].Box_Index].Points[2]].point[0];
					P0[1] = points[triangles[arrays[intersect_triangle_id].Box_Index].Points[0]].point[1];
					P1[1] = points[triangles[arrays[intersect_triangle_id].Box_Index].Points[1]].point[1];
					P2[1] = points[triangles[arrays[intersect_triangle_id].Box_Index].Points[2]].point[1];
					P0[2] = points[triangles[arrays[intersect_triangle_id].Box_Index].Points[0]].point[2];
					P1[2] = points[triangles[arrays[intersect_triangle_id].Box_Index].Points[1]].point[2];
					P2[2] = points[triangles[arrays[intersect_triangle_id].Box_Index].Points[2]].point[2];
					
					//若相交点在水线上，则抛弃之 22.11.7 jzy
					if(current_intersect_point[2]>water_line)
						break;
					
					RayReflectAndUpdateRayDir(ray, P0, P1, P2);
					ray->p.x = current_intersect_point[0];
					ray->p.y = current_intersect_point[1];
					ray->p.z = current_intersect_point[2];
					ray->distance += near_triangle_distance;
					ray->times++;
					ray->triangle_index = arrays[intersect_triangle_id].Box_Index;
					//ray->triangle_index = intersect_triangle_id;

					calced_triangle_id = intersect_triangle_id;
				}
				else
				{
                    //搜索完毕，与非空包围盒中的三角形都没有交点，退出非空包围盒
					break;
				}
			}
			if (ray->times == 3)
			{
			    //射线反射了三次，退出
				return;
			}
			int whichface;
			if (false == RayIntersectBox_Out(&leaf_node, ray, intersect_point, &whichface))
			{
                //求出口包围盒失败，如从这里退出，则是异常状况
				return;
			}
			int next_index = leaf_node.RopeIndex[whichface];
			if (next_index < 0)
			{
			     //从包围盒出来之后就出去了，正常返回
				return;
			}
				
			leaf_node = root[next_index];
		}
	}
	return;

}


//__device__ 
//bool IfValid(int a, int b, int c, int d)
//{
//	/*if ((a == b && b == c &&a != -1) || (a == b&&b == d&&a != -1) || (b == c&&c == d&&c != -1) || (a == c&&c == d&&a != -1))*/
//	if (a == b && b == c &&a != -1 && c == d)
//		return true;
//	else
//		return false;
//}
//
//
//__global__ 
//void sideraytracekernel_1d_gpu(KD_Node_V *d_root, Direction *d_rays, Square *d_squares, Prim_Box *d_array, Element *d_points, Triangle *d_triangles, int width, int height)
//{
//	int i = blockIdx.x*blockDim.x + threadIdx.x;
//	if (i >= (width + 1)*(height + 1))
//		return;
//	Direction ray = d_rays[i];
//	SingleRayTrace(d_root, &ray, d_array, d_points, d_triangles);
//	d_rays[i] = ray;
//}
//
//__global__ 
//void centraytracekernel_1d_gpu(KD_Node_V *d_root, Direction *d_rays, Square *d_squares, Prim_Box *d_array, Element *d_points, Triangle *d_triangles, int width, int height)
//{
//	int i = blockIdx.x*blockDim.x + threadIdx.x;
//
//	if (i >= width*height)
//		return;
//
//	int a = d_rays[d_squares[i].ray_index[0]].triangle_index;
//	int b = d_rays[d_squares[i].ray_index[1]].triangle_index;
//	int c = d_rays[d_squares[i].ray_index[2]].triangle_index;
//	int d = d_rays[d_squares[i].ray_index[3]].triangle_index;
//
//	bool value = IfValid(a, b, c, d);
//	if (value)
//	{
//		Direction ray = d_rays[d_squares[i].ray_index[4]];
//		SingleRayTrace(d_root, &ray, d_array, d_points, d_triangles);
//		d_rays[d_squares[i].ray_index[4]] = ray;
//	}
//	d_squares[i].right = value;
//	
//}
//
//
//void allraystrace_v2(Direction *d_rays, Square *d_squares, int width, int height,
//	KD_Node_V *d_root, Prim_Box *d_array, int prim_boxnum,
//	Element *d_points, int pointsnum, Triangle *d_triangles, int trianglesnum)
//{
//	const int gridSize1 = ((width + 1)*(height + 1) + TPB - 1) / TPB;
//	const int blockSize1 = TPB;
//
//	sideraytracekernel_1d_gpu << <gridSize1, blockSize1 >> >(d_root, d_rays, d_squares, d_array, d_points, d_triangles, width, height);
//
//	HANDLE_ERROR(hipGetLastError());
//	//HANDLE_ERROR(hipDeviceSynchronize());
//
//	const int gridSize2 = (width * height + TPB - 1) / TPB;
//	const int blockSize2 = TPB;
//
//	centraytracekernel_1d_gpu << <gridSize2, blockSize2>> >(d_root, d_rays, d_squares, d_array, d_points, d_triangles, width, height);
//
//	HANDLE_ERROR(hipGetLastError());
//	//HANDLE_ERROR(hipDeviceSynchronize());
//}

//声线管束有效性判断,四个顶点在同一个三角面元内即为有效
__device__ bool ValidRayTube(int a, int b, int c, int d)
{
	if (a == b && b == c && a != -1 && c == d)
		return true;
	else
		return false;
}

__device__ bool InvalidRayTube(int a, int b, int c, int d)
{
	return (a == b && b == c && a == -1 && c == d);
}



__global__ void centraytracekernel_1d_gpu(KD_Node_V *d_root, Direction *d_rays, Square *d_squares, Prim_Box *d_array, Element *d_points, Triangle *d_triangles,
	int DivRayTubeNum, int RayTubeNum, Axis direction, float water_line)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= DivRayTubeNum)
		return;

	Direction tmp = d_rays[i + RayTubeNum];
	//tmp.caled = false;
	//tmp.flag = false;
	tmp.distance = 0;
	tmp.times = 0;
	tmp.triangle_index = -1;
	tmp.dir.x = -direction.x;
	tmp.dir.y = -direction.y;
	tmp.dir.z = -direction.z;
	SingleRayTrace(d_root, &tmp, d_array, d_points, d_triangles, water_line);
	d_rays[i + RayTubeNum] = tmp;

}

__global__ void CreateCenterRay_Div(Direction *d_rays, Square *d_squares, int RayTubeNum, int DivRayTubeNum, int* d_DivRayTubeNum, int* d_squares_pred)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;

	if (idx >= RayTubeNum)
		return;

	if (d_squares[idx].IsDivRayTube == true)
	{
		int index = d_DivRayTubeNum[blockIdx.x] + d_squares_pred[idx] + RayTubeNum * 2 + DivRayTubeNum;
		d_squares[idx].CenterRayIndex = index;
		d_rays[index].p = d_squares[idx].CenterRay;
		//d_rays[index].p.x = d_squares[idx].CenterRay.x;
		//d_rays[index].p.y = d_squares[idx].CenterRay.y;
		//d_rays[index].p.z = d_squares[idx].CenterRay.z;
	}
}

__global__ void CreateCenterRay(Direction *d_rays, Square *d_squares, int RayTubeNum, int* d_DivRayTubeNum, int* d_squares_pred)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;

	if (idx >= RayTubeNum)
		return;

	if (d_squares[idx].IsDivRayTube == true)
	{
		int index = d_DivRayTubeNum[blockIdx.x] + d_squares_pred[idx] + RayTubeNum;
		d_squares[idx].CenterRayIndex = index;
		d_rays[index].p = d_squares[idx].CenterRay;
		//d_rays[index].p[0] = d_squares[idx].CenterRay.x;
		//d_rays[index].p[1] = d_squares[idx].CenterRay.y;
		//d_rays[index].p[2] = d_squares[idx].CenterRay.z;
	}
}

__global__ void ExclusiveSumScan(int* d_squares_pred, int RayTubeNum)
{
	int tid = threadIdx.x;
	int idx = 2 * tid + 2 * blockDim.x * blockIdx.x;

	int offset = 1;
	int num = (RayTubeNum + 2 * blockDim.x - 1) / (2 * blockDim.x); //长度为array_length的数组被blockDim.x*2分割成了num个segment
	int len = num * 2 * blockDim.x;//补齐后数组长度为len

	__shared__ int temp[512];

	//边界填充
	for (int i = tid; i < blockDim.x; i += blockDim.x)
	{
		temp[2 * tid] = 0;
		temp[2 * tid + 1] = 0;
	}
	__syncthreads();

	if (idx < RayTubeNum)
	{
		temp[2 * tid] = d_squares_pred[idx];
	}

	if (idx + 1 < RayTubeNum)
	{
		temp[2 * tid + 1] = d_squares_pred[idx + 1];
	}
	__syncthreads();

	//up-sweep phase上行阶段
	for (int j = blockDim.x; j > 0; j >>= 1)
	{
		if (tid < j)
		{
			int ai = offset * (2 * tid + 1) - 1;
			int bi = offset * (2 * tid + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
		__syncthreads();
	}

	//down-sweep phase下行阶段
	if (tid == 0)
		temp[511] = 0;

	for (int j = 1; j < (blockDim.x * 2); j *= 2)
	{
		offset >>= 1;
		__syncthreads();
		if (tid < j)
		{
			int ai = offset * (2 * tid + 1) - 1;
			int bi = offset * (2 * tid + 2) - 1;

			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
		__syncthreads();
	}

	if (idx < RayTubeNum)
	{
		d_squares_pred[idx] = temp[2 * tid];
	}
	if (idx + 1 < RayTubeNum)
	{
		d_squares_pred[idx + 1] = temp[2 * tid + 1];
	}
}

__global__ void ScanInArray(Square *d_squares, int* d_squares_pred, int RayTubeNum)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;

	__shared__ bool sDivFlag[512];
	sDivFlag[tid] = (idx < RayTubeNum) ? d_squares[idx].IsDivRayTube : false;
	__syncthreads();

	if (idx < RayTubeNum)
	{
		d_squares_pred[idx] = (sDivFlag[tid] == true) ? 1 : 0;
	}
}

__global__ void BlellochScan1(int* d_DivRayTubeNum, int array_length, int* d_sum_gmem, int* d_sum_Gmem, int* count1_s)
{
	int tid = threadIdx.x;
	int idx = 2 * tid + 2 * blockDim.x * blockIdx.x;
	int id = tid + blockDim.x * blockIdx.x;
	
	int offset = 1;
	int num = (array_length + 2 * blockDim.x - 1) / (2 * blockDim.x); //长度为array_length的数组被blockDim.x*2分割成了num个segment
	int len = num * 2 * blockDim.x;//补齐后数组长度为len

	__shared__ int temp[512];

	//边界填充
	for (int i = tid; i < blockDim.x; i += blockDim.x)
	{
		temp[2 * tid] = 0;
		temp[2 * tid + 1] = 0;
	}
	__syncthreads();

	if (idx < array_length)
	{
		temp[2 * tid] = d_DivRayTubeNum[idx];
	}

	if (idx + 1 < array_length)
	{
		temp[2 * tid + 1] = d_DivRayTubeNum[idx + 1];
	}
	__syncthreads();

	//up-sweep phase上行阶段
	for (int j = blockDim.x; j > 0; j >>= 1)
	{
		if (tid < j)
		{
			int ai = offset * (2 * tid + 1) - 1;
			int bi = offset * (2 * tid + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
		__syncthreads();
	}

	//down-sweep phase下行阶段
	__shared__ int islast;
	if (tid == 0)
	{
		d_sum_gmem[blockIdx.x + 1] = temp[2 * blockDim.x - 1];
		temp[2 * blockDim.x - 1] = 0;
		__threadfence();

		int value = atomicAdd(count1_s, 1);
		islast = (value == gridDim.x - 1);
	}
	__syncthreads();

	if (islast)
	{
		int i = id - blockIdx.x * blockDim.x;

		if (i < gridDim.x)
		{
			int tmp = 0;
			for (int n = 0; n < i + 2; n++)
			{
				tmp += d_sum_gmem[n];
			}
			//__syncthreads();
			d_sum_Gmem[i + 1] = tmp;
		}
		__syncthreads();
	}

	for (int j = 1; j < (blockDim.x * 2); j *= 2)
	{
		offset >>= 1;
		__syncthreads();
		if (tid < j)
		{
			int ai = offset * (2 * tid + 1) - 1;
			int bi = offset * (2 * tid + 2) - 1;

			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
		__syncthreads();
	}

	if (idx < array_length)
	{
		//d_DivRayTubeNum[idx] = temp[2 * tid] + d_sum_gmem[blockIdx.x];
		d_DivRayTubeNum[idx] = temp[2 * tid];
	}
	if (idx + 1 < array_length)
	{
		//d_DivRayTubeNum[idx + 1] = temp[2 * tid + 1] + d_sum_gmem[blockIdx.x];
		d_DivRayTubeNum[idx + 1] = temp[2 * tid + 1];
	}
}

__global__ void DivRayTubeNumAdd(int* d_DivRayTubeNum, int* d_sum_gmem, int array_length)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < array_length)
	{
		d_DivRayTubeNum[idx] += d_sum_gmem[blockIdx.x];
	}
}

__global__ void IsValidRayTube_gpu(Direction *d_rays, Square *d_squares, int RayTubeNum, int* d_DivRayTubeNum)
{

	__shared__ bool sDivFlag[512];

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;

	if (idx >= RayTubeNum)
		return;

	int a = d_rays[d_squares[idx].CornerRayIndex.x].triangle_index;
	int b = d_rays[d_squares[idx].CornerRayIndex.y].triangle_index;
	int c = d_rays[d_squares[idx].CornerRayIndex.z].triangle_index;
	int d = d_rays[d_squares[idx].CornerRayIndex.w].triangle_index;
	d_squares[idx].right = ValidRayTube(a, b, c, d);//参与积分的声线管束
	d_squares[idx].IsDivRayTube = !(InvalidRayTube(a, b, c, d) ^ d_squares[idx].right);//需要分裂的声线管束

	sDivFlag[tid] = (idx < RayTubeNum) ? d_squares[idx].IsDivRayTube : false;
	__syncthreads();

	if (sDivFlag[tid] == true)
	{
		atomicAdd(&d_DivRayTubeNum[blockIdx.x], 1);
	}
}

__global__ void sideraytracekernel_1d_gpu(KD_Node_V *d_root, Direction *d_rays, Square *d_squares, Prim_Box *d_array, Element *d_points, Triangle *d_triangles,
	int totalraysnum, Axis direction, float water_line)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= totalraysnum)
		return;

	Direction tmp = d_rays[i];//2019-04-08
	//tmp.caled = false;
	//tmp.flag = false;
	tmp.distance = 0;
	tmp.times = 0;
	tmp.triangle_index = -1;
	tmp.dir.x = -direction.x;
	tmp.dir.y = -direction.y;
	tmp.dir.z = -direction.z;
	SingleRayTrace(d_root, &tmp, d_array, d_points, d_triangles, water_line);
	d_rays[i] = tmp;
}

void allraystrace_v2(Direction *d_rays, Square *d_squares, int width, int height, KD_Node_V *d_root, Prim_Box *d_array, Element *d_points, Triangle *d_triangles,
	int* d_DivRayTubeNum, int* DivRayTubeNum, int* d_sum_gmem, int* d_sum_Gmem, int* d_squares_pred, Axis direction, float angle, float water_line)
{
	const int gridSize1 = ((width + 1) * (height + 1) + TPB - 1) / TPB;
	const int blockSize1 = TPB;

	float temp = sqrt((direction.x * direction.x) + (direction.y * direction.y) + (direction.z * direction.z));
	direction.x /= temp;
	direction.y /= temp;
	direction.z /= temp;

	sideraytracekernel_1d_gpu << <gridSize1, blockSize1 >> >(d_root, d_rays, d_squares, d_array, d_points, d_triangles, (width + 1) * (height + 1), direction, water_line);
	HANDLE_ERROR(hipGetLastError());
	HANDLE_ERROR(hipDeviceSynchronize());

	dim3 threadSize(512, 1, 1);
	dim3 blockSize(width * height / 512 + 1, 1, 1);
	int array_length = blockSize.x;

	IsValidRayTube_gpu << <blockSize, threadSize >> >(d_rays, d_squares, width * height, d_DivRayTubeNum);
	HANDLE_ERROR(hipGetLastError());
	//HANDLE_ERROR(hipDeviceSynchronize());

	dim3 ThreadSize(256, 1, 1);
	dim3 BlockSize(array_length / 512 + 1, 1, 1);
	int ang = (int)angle;
	int x[38575];
	hipMemcpy(x,d_DivRayTubeNum,sizeof(int)*38575,hipMemcpyDeviceToHost);
	
	int *count1_s;
	hipMalloc((void**)&count1_s,sizeof(int));
	int value = 0;
	hipMemcpy(count1_s,&value,sizeof(int),hipMemcpyHostToDevice);
	BlellochScan1 << <BlockSize, ThreadSize >> >(d_DivRayTubeNum, array_length, d_sum_gmem, d_sum_Gmem, count1_s);
	HANDLE_ERROR(hipGetLastError());
	hipFree(count1_s);
	
	//HANDLE_ERROR(hipDeviceSynchronize());

/*
	int* sum_gmem;
	sum_gmem = (int*)malloc(35 * sizeof(int));
	printf("sum_gmem!\n");
	HANDLE_ERROR(hipMemcpy(sum_gmem, d_sum_gmem, 35 * sizeof(int), hipMemcpyDeviceToHost));
	for(int i=0; i<35; i++)
	{
	printf("%d ",sum_gmem[i]);
	}
	printf("\n");
	free(sum_gmem);
*/

	int DivRayTubeNumIdx = BlockSize.x;
	HANDLE_ERROR(hipMemcpy(DivRayTubeNum, &d_sum_Gmem[DivRayTubeNumIdx], sizeof(int), hipMemcpyDeviceToHost));
	//debug*********
	//printf("DivRayTubeNumIdx=%d\n",DivRayTubeNumIdx);
	//***********
	DivRayTubeNumAdd << <BlockSize, threadSize >> >(d_DivRayTubeNum, d_sum_Gmem, array_length);
	HANDLE_ERROR(hipGetLastError());
	HANDLE_ERROR(hipDeviceSynchronize());

	ScanInArray << <blockSize, threadSize >> >(d_squares, d_squares_pred, width * height);
	HANDLE_ERROR(hipGetLastError());
	
	dim3 Sizethread(256, 1, 1);
	dim3 Sizeblock(width * height / 512 + 1, 1, 1);
	ExclusiveSumScan << <Sizeblock, Sizethread >> >(d_squares_pred, width * height);
	HANDLE_ERROR(hipGetLastError());

	CreateCenterRay << <blockSize, threadSize >> >(d_rays, d_squares, (width + 1) * (height + 1), d_DivRayTubeNum, d_squares_pred);
	HANDLE_ERROR(hipGetLastError());
	const int gridSize2 = (*DivRayTubeNum + TPB - 1) / TPB;
	const int blockSize2 = TPB;
	centraytracekernel_1d_gpu << <gridSize2, blockSize2 >> >(d_root, d_rays, d_squares, d_array, d_points, d_triangles, *DivRayTubeNum, (width + 1) * (height + 1), direction, water_line);
	//HANDLE_ERROR(hipGetLastError());	
	HANDLE_ERROR(hipDeviceSynchronize());

}


__global__ void BlellochScan2(int* d_DivRayTubeNum, int array_length, int* d_sum_gmem, int* d_sum_Gmem, int* count2_s)
{
	int tid = threadIdx.x;
	int idx = 2 * tid + 2 * blockDim.x * blockIdx.x;
	int id = tid + blockDim.x * blockIdx.x;

	int offset = 1;
	int num = (array_length + 2 * blockDim.x - 1) / (2 * blockDim.x); //长度为array_length的数组被blockDim.x*2分割成了num个segment
	int len = num * 2 * blockDim.x;//补齐后数组长度为len

	__shared__ int temp[512];

	//边界填充
	for (int i = tid; i < blockDim.x; i += blockDim.x)
	{
		temp[2 * tid] = 0;
		temp[2 * tid + 1] = 0;
	}
	__syncthreads();

	if (idx < array_length)
	{
		temp[2 * tid] = d_DivRayTubeNum[idx];
	}

	if (idx + 1 < array_length)
	{
		temp[2 * tid + 1] = d_DivRayTubeNum[idx + 1];
	}
	__syncthreads();

	//up-sweep phase上行阶段
	for (int j = blockDim.x; j > 0; j >>= 1)
	{
		if (tid < j)
		{
			int ai = offset * (2 * tid + 1) - 1;
			int bi = offset * (2 * tid + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
		__syncthreads();
	}

	//down-sweep phase下行阶段
	__shared__ int islast;
	if (tid == 0)
	{
		d_sum_gmem[blockIdx.x + 1] = temp[2 * blockDim.x - 1];
		temp[2 * blockDim.x - 1] = 0;
		__threadfence();

		int value = atomicAdd(count2_s, 1);
		islast = (value == gridDim.x - 1);
	}
	__syncthreads();

	if (islast)
	{
		int i = id - blockIdx.x * blockDim.x;
		if (i < gridDim.x)
		{
			int tmp = 0;
			for (int n = 0; n < i + 2; n++)
			{
				tmp += d_sum_gmem[n];
			}
			//__syncthreads();
			d_sum_Gmem[i + 1] = tmp;
		}
		__syncthreads();
	}

	for (int j = 1; j < (blockDim.x * 2); j *= 2)
	{
		offset >>= 1;
		__syncthreads();
		if (tid < j)
		{
			int ai = offset * (2 * tid + 1) - 1;
			int bi = offset * (2 * tid + 2) - 1;

			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
		__syncthreads();
	}

	if (idx < array_length)
	{
		d_DivRayTubeNum[idx] = temp[2 * tid];
	}
	if (idx + 1 < array_length)
	{
		d_DivRayTubeNum[idx + 1] = temp[2 * tid + 1];
	}
}

__global__ void sideraytracekernel_DivRay_gpu(KD_Node_V *d_root, Direction *d_rays, Prim_Box *d_array, Element *d_points, Triangle *d_triangles,
	int d_totalDivRayNum, Axis direction, float water_line)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i >= d_totalDivRayNum)
		return;

	Direction tmp = d_rays[i];//2019-04-08
	//tmp.caled = false;
	//tmp.flag = false;
	tmp.distance = 0;
	tmp.times = 0;
	tmp.triangle_index = -1;
	tmp.dir.x = -direction.x;
	tmp.dir.y = -direction.y;
	tmp.dir.z = -direction.z;
	SingleRayTrace(d_root, &tmp, d_array, d_points, d_triangles, water_line);
	d_rays[i] = tmp;
}

__global__ void centraytracekernel_DivRay_gpu(KD_Node_V *d_root, Direction *d_rays, Square *d_squares, Prim_Box *d_array, Element *d_points, Triangle *d_triangles,
	int DivRayTubeNum, int d_totalDivRayNum, Axis direction, float water_line)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= DivRayTubeNum)
		return;

	Direction temp = d_rays[i + d_totalDivRayNum * 2 + DivRayTubeNum];
	//temp.caled = false;
	//temp.flag = false;
	temp.distance = 0;
	temp.times = 0;
	temp.triangle_index = -1;
	temp.dir.x = -direction.x;
	temp.dir.y = -direction.y;
	temp.dir.z = -direction.z;
	SingleRayTrace(d_root, &temp, d_array, d_points, d_triangles, water_line);
	d_rays[i + d_totalDivRayNum * 2 + DivRayTubeNum] = temp;
}

void allraystrace_DivRayFirst(Direction *d_rays2, Square *d_squares2, int d_totalDivRayNum, KD_Node_V *d_root, Prim_Box *d_array, Element *d_points, Triangle *d_triangles,
	int* d_DivRayTubeNum, int* DivRayTubeNum, int* d_sum_gmem, int* d_sum_Gmem, int* d_squares_pred, Axis direction, float angle, float water_line)
{
	d_totalDivRayNum *= 4;
	const int gridSize1 = (d_totalDivRayNum + TPB - 1) / TPB;
	const int blockSize1 = TPB;

	float temp = sqrt((direction.x * direction.x) + (direction.y * direction.y) + (direction.z * direction.z));
	direction.x /= temp;
	direction.y /= temp;
	direction.z /= temp;

	sideraytracekernel_DivRay_gpu << <gridSize1, blockSize1 >> >(d_root, d_rays2, d_array, d_points, d_triangles, d_totalDivRayNum, direction, water_line);
	HANDLE_ERROR(hipGetLastError());
	//HANDLE_ERROR(hipDeviceSynchronize());


	dim3 threadSize(512, 1, 1);
	dim3 blockSize(d_totalDivRayNum / 512 + 1, 1, 1);
	int array_length = blockSize.x;

	IsValidRayTube_gpu << <blockSize, threadSize >> >(d_rays2, d_squares2, d_totalDivRayNum, d_DivRayTubeNum);

	dim3 ThreadSize(256, 1, 1);
	dim3 BlockSize(array_length / 512 + 1, 1, 1);
	int ang = (int)angle;
	
	int *count2_s;
	hipMalloc((void**)&count2_s,sizeof(int));
	int value = 0;
	hipMemcpy(count2_s,&value,sizeof(int),hipMemcpyHostToDevice);
	BlellochScan2 << <BlockSize, ThreadSize >> >(d_DivRayTubeNum, array_length, d_sum_gmem, d_sum_Gmem, count2_s);
	hipFree(count2_s);

	int DivRayTubeNumIdx = BlockSize.x;
	hipMemcpy(DivRayTubeNum, &d_sum_Gmem[DivRayTubeNumIdx], sizeof(int), hipMemcpyDeviceToHost);

	DivRayTubeNumAdd << <BlockSize, threadSize >> >(d_DivRayTubeNum, d_sum_Gmem, array_length);

	//int DivRayTubeNumIdx = BlockSize.x;

	ScanInArray << <blockSize, threadSize >> >(d_squares2, d_squares_pred, d_totalDivRayNum);

	dim3 Sizethread(256, 1, 1);
	dim3 Sizeblock(d_totalDivRayNum / 512 + 1, 1, 1);
	ExclusiveSumScan << <Sizeblock, Sizethread >> >(d_squares_pred, d_totalDivRayNum);

	//hipMemcpy(DivRayTubeNum, &d_sum_gmem[DivRayTubeNumIdx], sizeof(int), hipMemcpyDeviceToHost);

	CreateCenterRay_Div << <blockSize, threadSize >> >(d_rays2, d_squares2, d_totalDivRayNum, *DivRayTubeNum, d_DivRayTubeNum, d_squares_pred);

	const int gridSize2 = (*DivRayTubeNum + TPB - 1) / TPB;
	const int blockSize2 = TPB;

	centraytracekernel_DivRay_gpu << <gridSize2, blockSize2 >> >(d_root, d_rays2, d_squares2, d_array, d_points, d_triangles, *DivRayTubeNum, d_totalDivRayNum, direction, water_line);

	HANDLE_ERROR(hipGetLastError());
	//HANDLE_ERROR(hipDeviceSynchronize());
}



__global__ void BlellochScan3(int* d_DivRayTubeNum, int array_length, int* d_sum_gmem, int* d_sum_Gmem, int* count3_s)
{
	int tid = threadIdx.x;
	int idx = 2 * tid + 2 * blockDim.x * blockIdx.x;
	int id = tid + blockDim.x * blockIdx.x;

	int offset = 1;
	int num = (array_length + 2 * blockDim.x - 1) / (2 * blockDim.x); //长度为array_length的数组被blockDim.x*2分割成了num个segment
	int len = num * 2 * blockDim.x;//补齐后数组长度为len

	__shared__ int temp[512];

	//边界填充
	for (int i = tid; i < blockDim.x; i += blockDim.x)
	{
		temp[2 * tid] = 0;
		temp[2 * tid + 1] = 0;
	}
	__syncthreads();

	if (idx < array_length)
	{
		temp[2 * tid] = d_DivRayTubeNum[idx];
	}

	if (idx + 1 < array_length)
	{
		temp[2 * tid + 1] = d_DivRayTubeNum[idx + 1];
	}
	__syncthreads();

	//up-sweep phase上行阶段
	for (int j = blockDim.x; j > 0; j >>= 1)
	{
		if (tid < j)
		{
			int ai = offset * (2 * tid + 1) - 1;
			int bi = offset * (2 * tid + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
		__syncthreads();
	}

	//down-sweep phase下行阶段
	__shared__ int islast;
	if (tid == 0)
	{
		d_sum_gmem[blockIdx.x + 1] = temp[2 * blockDim.x - 1];
		temp[2 * blockDim.x - 1] = 0;
		__threadfence();

		int value = atomicAdd(count3_s, 1);
		islast = (value == gridDim.x - 1);
	}
	__syncthreads();

	if (islast)
	{
		int i = id - blockIdx.x * blockDim.x;
		if (i < gridDim.x)
		{
			int tmp = 0;
			for (int n = 0; n < i + 2; n++)
			{
				tmp += d_sum_gmem[n];
			}
			//__syncthreads();
			d_sum_Gmem[i + 1] = tmp;
		}
		__syncthreads();
	}

	for (int j = 1; j < (blockDim.x * 2); j *= 2)
	{
		offset >>= 1;
		__syncthreads();
		if (tid < j)
		{
			int ai = offset * (2 * tid + 1) - 1;
			int bi = offset * (2 * tid + 2) - 1;

			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
		__syncthreads();
	}

	if (idx < array_length)
	{
		d_DivRayTubeNum[idx] = temp[2 * tid];
	}
	if (idx + 1 < array_length)
	{
		d_DivRayTubeNum[idx + 1] = temp[2 * tid + 1];
	}
}

void allraystrace_DivRaySecond(Direction *d_rays2, Square *d_squares2, int d_totalDivRayNum, KD_Node_V *d_root, Prim_Box *d_array, Element *d_points, Triangle *d_triangles,
	int* d_DivRayTubeNum, int* DivRayTubeNum, int* d_sum_gmem, int* d_sum_Gmem, int* d_squares_pred, Axis direction, float angle, float water_line)
{
	d_totalDivRayNum *= 4;
	const int gridSize1 = (d_totalDivRayNum + TPB - 1) / TPB;
	const int blockSize1 = TPB;

	float temp = sqrt((direction.x * direction.x) + (direction.y * direction.y) + (direction.z * direction.z));
	direction.x /= temp;
	direction.y /= temp;
	direction.z /= temp;

	sideraytracekernel_DivRay_gpu << <gridSize1, blockSize1 >> >(d_root, d_rays2, d_array, d_points, d_triangles, d_totalDivRayNum, direction, water_line);
	HANDLE_ERROR(hipGetLastError());
	//HANDLE_ERROR(hipDeviceSynchronize());


	dim3 threadSize(512, 1, 1);
	dim3 blockSize(d_totalDivRayNum / 512 + 1, 1, 1);
	int array_length = blockSize.x;

	IsValidRayTube_gpu << <blockSize, threadSize >> >(d_rays2, d_squares2, d_totalDivRayNum, d_DivRayTubeNum);

	dim3 ThreadSize(256, 1, 1);
	dim3 BlockSize(array_length / 512 + 1, 1, 1);
	int ang = (int)angle;
	int *count3_s;
	hipMalloc((void**)&count3_s,sizeof(int));
	int value = 0;
	hipMemcpy(count3_s,&value,sizeof(int),hipMemcpyHostToDevice);
	BlellochScan3 << <BlockSize, ThreadSize >> >(d_DivRayTubeNum, array_length, d_sum_gmem, d_sum_Gmem, count3_s);
	hipFree(count3_s);

	int DivRayTubeNumIdx = BlockSize.x;
	hipMemcpy(DivRayTubeNum, &d_sum_Gmem[DivRayTubeNumIdx], sizeof(int), hipMemcpyDeviceToHost);

	DivRayTubeNumAdd << <BlockSize, threadSize >> >(d_DivRayTubeNum, d_sum_Gmem, array_length);

	//int DivRayTubeNumIdx = BlockSize.x;

	ScanInArray << <blockSize, threadSize >> >(d_squares2, d_squares_pred, d_totalDivRayNum);

	dim3 Sizethread(256, 1, 1);
	dim3 Sizeblock(d_totalDivRayNum / 512 + 1, 1, 1);
	ExclusiveSumScan << <Sizeblock, Sizethread >> >(d_squares_pred, d_totalDivRayNum);

	//hipMemcpy(DivRayTubeNum, &d_sum_gmem[DivRayTubeNumIdx], sizeof(int), hipMemcpyDeviceToHost);

	CreateCenterRay_Div << <blockSize, threadSize >> >(d_rays2, d_squares2, d_totalDivRayNum, *DivRayTubeNum, d_DivRayTubeNum, d_squares_pred);

	const int gridSize2 = (*DivRayTubeNum + TPB - 1) / TPB;
	const int blockSize2 = TPB;

	centraytracekernel_DivRay_gpu << <gridSize2, blockSize2 >> >(d_root, d_rays2, d_squares2, d_array, d_points, d_triangles, *DivRayTubeNum, d_totalDivRayNum, direction, water_line);

	HANDLE_ERROR(hipGetLastError());
	//HANDLE_ERROR(hipDeviceSynchronize());
}

__global__ void centraytracekernel_DivRaySecond_gpu(Direction *d_rays, Square *d_squares, int d_totalDivRayNum)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= d_totalDivRayNum)
		return;

	int a = d_rays[d_squares[i].CornerRayIndex.x].triangle_index;
	int b = d_rays[d_squares[i].CornerRayIndex.y].triangle_index;
	int c = d_rays[d_squares[i].CornerRayIndex.z].triangle_index;
	int d = d_rays[d_squares[i].CornerRayIndex.w].triangle_index;
	d_squares[i].right = ValidRayTube(a, b, c, d);

}

void allraystrace_DivRayThird(Direction *d_rays2, Square *d_squares2, int d_totalDivRayNum, KD_Node_V *d_root, Prim_Box *d_array,
	Element *d_points, Triangle *d_triangles, Axis direction, float water_line)
{
	d_totalDivRayNum *= 4;

	/*Square* squares2;
	squares2 = (Square*)malloc(15976009 * sizeof(Square));
	HANDLE_ERROR(hipMemcpy(squares2, d_squares2, 15976009 * sizeof(Square), hipMemcpyDeviceToHost));
	int count=0;
	for(int i=9716; i< 9717; i++)
	{
	printf("%d %d %d %d %d\n", squares2[i].ray_index[0], squares2[i].ray_index[1], squares2[i].ray_index[2], squares2[i].ray_index[3], squares2[i].ray_index[4]);
	}
	Direction* rays2;
	rays2 = (Direction*)malloc(31960013 * sizeof(Direction));
	HANDLE_ERROR(hipMemcpy(rays2, d_rays2, 31960013 * sizeof(Direction), hipMemcpyDeviceToHost));
	printf("rays %d: %.7f %.7f %.7f\n", 9574360, rays2[9574360].p[0], rays2[9574360].p[1], rays2[9574360].p[2]);
	printf("rays %d: %.7f %.7f %.7f\n", 9716, rays2[9716].p[0], rays2[9716].p[1], rays2[9716].p[2]);
	printf("rays %d: %.7f %.7f %.7f\n", 19131717, rays2[19131717].p[0], rays2[19131717].p[1], rays2[19131717].p[2]);
	printf("rays %d: %.7f %.7f %.7f\n", 9719, rays2[9719].p[0], rays2[9719].p[1], rays2[9719].p[2]);
	//printf("rays %d: %.7f %.7f %.7f\n", 2735394, rays2[2735394].p[0], rays2[2735394].p[1], rays2[2735394].p[2]);
	printf("-------------------------------------------------------\n");*/

	const int gridSize = (d_totalDivRayNum + TPB - 1) / TPB;
	const int blockSize = TPB;

	float temp = sqrt((direction.x * direction.x) + (direction.y * direction.y) + (direction.z * direction.z));
	direction.x /= temp;
	direction.y /= temp;
	direction.z /= temp;

	sideraytracekernel_DivRay_gpu << <gridSize, blockSize >> >(d_root, d_rays2, d_array, d_points, d_triangles, d_totalDivRayNum, direction, water_line);

	/*HANDLE_ERROR(hipMemcpy(rays2, d_rays2, 31960013 * sizeof(Direction), hipMemcpyDeviceToHost));
	printf("rays %d: %.7f %.7f %.7f\n", 9574360, rays2[9574360].p[0], rays2[9574360].p[1], rays2[9574360].p[2]);
	printf("rays %d: %.7f %.7f %.7f\n", 9716, rays2[9716].p[0], rays2[9716].p[1], rays2[9716].p[2]);
	printf("rays %d: %.7f %.7f %.7f\n", 19131717, rays2[19131717].p[0], rays2[19131717].p[1], rays2[19131717].p[2]);
	printf("rays %d: %.7f %.7f %.7f\n", 9719, rays2[9719].p[0], rays2[9719].p[1], rays2[9719].p[2]);
	//printf("rays %d: %.7f %.7f %.7f\n", 2735394, rays2[2735394].p[0], rays2[2735394].p[1], rays2[2735394].p[2]);
	printf("-------------------------------------------------------\n");*/

	HANDLE_ERROR(hipGetLastError());
	//HANDLE_ERROR(hipDeviceSynchronize());

	centraytracekernel_DivRaySecond_gpu << <gridSize, blockSize >> >(d_rays2, d_squares2, d_totalDivRayNum);

	HANDLE_ERROR(hipGetLastError());
	//HANDLE_ERROR(hipDeviceSynchronize());
}
