#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<>
#include<hip/device_functions.h>
#include<string.h>
#include "scalfuc.h"
#include "handlerror.h"
#include "sm_20_atomic_functions.h"
#define  D2R 3.14159265358979/180.0
__global__ void reduce_add_sre(float* d_sum_re, RayBeamInfo* rays, int raysBeamNum)
{
	__shared__ float sdata[512];

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;

	sdata[tid] = (idx < raysBeamNum) ? rays[idx].parameter.p[1] : 0;
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		d_sum_re[blockIdx.x] = sdata[0];
	}
}
__global__ void reduce_sre(float* d_sum_re, float* d_in, int Num)
{
	__shared__ float sdata[512];

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;

	sdata[tid] = (idx < Num) ? d_in[idx] : 0;
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		d_sum_re[blockIdx.x] = sdata[0];
	}
}
__global__ void reduce_add_sim(float* d_sum_re, RayBeamInfo* rays, int raysBeamNum)
{
	__shared__ float sdata[512];

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;

	sdata[tid] = (idx < raysBeamNum) ? rays[idx].parameter.p[2] : 0;
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		d_sum_re[blockIdx.x] = sdata[0];
	}
}
__global__ void reduce_sim(float* d_sum_re, float* d_in, int Num)
{
	__shared__ float sdata[512];

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;

	sdata[tid] = (idx < Num) ? d_in[idx] : 0;
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		d_sum_re[blockIdx.x] = sdata[0];
	}
}


//以下四个函数cdiv cmul cabs d_ReflectCoeff_2为反射系数计算需要  姬梓遇20210913
#define A2R 3.1415926535897/180

__device__ __host__ comp cdiv(comp z1,comp z2)       
{ double x1,x2,y1,y2;
  comp z;
  x1=z1.re;
  x2=z2.re;
  y1=z1.im;
  y2=z2.im;
  z.re=(x1*x2+y1*y2)/(x2*x2+y2*y2);
  z.im=(x2*y1-y2*x1)/(x2*x2+y2*y2);
  return z;
}

__device__ __host__ comp cmul(comp z1,comp z2)
{ double x1,x2,y1,y2;
  comp z;
  x1=z1.re;
  x2=z2.re;
  y1=z1.im;
  y2=z2.im;
  z.re=x1*x2-y1*y2;
  z.im=x1*y2+y1*x2;
  return z;
}

__device__ __host__ double cabs(comp z)                      
{
	double x,y;
	x=z.re;
	y=z.im;
	return sqrt(x*x+y*y);
}

//计算双层层敷瓦T体的反射系数
__device__ float d_ReflectCoeff_2(float f, float theta)
{
	double theta0, theta1, theta2;
	double c0, c1, c2, c3;
	double rou0, rou1, rou2, rou3;
	double Ee2;
	double Z0, Z3, Z4;
	double eta1, eta2;
	double k1, k2, k3;
	double d1, d2, d3;
	double h1, h2, h3;
	double R, AR;
	double phi;
	double AR0, AR1;
	comp temp1, temp2, Z23, Z12, Z01, Zin, cw1, cw2, Z1, Z2;

	theta0 = theta;  //入射角度
	rou0 = 1000;     //水密度（单位kg/m^3）
	c0 = 1500;       //水中声速（单位m/s）
	Z0 = rou0 * c0;  //水的阻抗

	rou1 = 1039;     //橡胶（介质1）密度
	c1 = 1470;     //介质1中的等效声速
	eta1 = 0.4;      //损耗因子
	phi = atan(eta1);
	cw1.re = c1 * pow((1 * 1 + eta1*eta1), 0.25)*cos(phi / 2);
	cw1.im = c1 * pow((1 * 1 + eta1*eta1), 0.25)*sin(phi / 2);//粘弹材料波速
	Z1.re = rou1 * cw1.re;    //材料的纵波波阻抗
	Z1.im = rou1 * cw1.im;    //材料的纵波波阻抗
	k1 = 2 * 180 * A2R * f / c1;//介质1波数

	rou2 = 1090;     //橡胶（介质1）密度
	eta2 = 0.5;      //损耗因子
	phi = atan(eta2);
	Ee2 = 1e9;      //介质2的杨氏模量
	c2 = sqrt(Ee2 / rou2);//介质2中的等效声速
	cw2.re = c2 * pow((1 * 1 + eta2*eta2), 0.25)*cos(phi / 2);
	cw2.im = c2 * pow((1 * 1 + eta2*eta2), 0.25)*sin(phi / 2);//粘弹材料波速
	Z2.re = rou2 * cw2.re;    //材料的纵波波阻抗
	Z2.im = rou2 * cw2.im;    //材料的纵波波阻抗
	k2 = 2 * 180 * A2R * f / c2;//介质波数

	rou3 = 7850;     //钢（衬底）密度
	c3 = 5200;       //介质2中的等效声速
	Z3 = rou3 * c3;  //衬底(钢的阻抗大于水的20倍)
	k3 = 2 * 180 * A2R * f / c3;//介质波数,衬底为钢板时的传播速度

	d1 = 2e-3;       //介质1厚度
	d2 = 2e-3;       //介质2厚度
	d3 = 3e-3;       //衬底厚度

					 //////根据Snell折射定理计算各个介质层中的入射角度
	AR0 = sin(theta0) * c0 / c1;  //定义全反射系数
	if (fabs(AR0) > 1)
	{
		theta1 = 90 * A2R;
	}
	else
	{
		theta1 = asin(sin(theta0) * c0 / c1);
	}

	AR1 = sin(theta1) * c1 / c2;  //定义全反射系数
	if (fabs(AR1) > 1)
	{
		theta2 = 90 * A2R;
	}
	else
	{
		theta2 = asin(sin(theta1) * c1 / c2);
	}
	h1 = tan(k1 * cos(theta0) * d1);
	h2 = tan(k2 * cos(theta1) * d2);
	h3 = tan(k3 * cos(theta2) * d3);
	Z4 = Z0;

	temp1.im = Z3 * h3;
	temp1.re = Z4;
	temp2.im = Z4 * h3;
	temp2.re = Z3;
	Z23 = cdiv(temp1, temp2);
	Z23.re = Z3 * Z23.re;
	Z23.im = Z3 * Z23.im;


	temp1.im = Z23.im + Z2.re * h2;
	temp1.re = Z23.re - h2*Z2.im;
	temp2.im = Z23.re * h2 + Z2.im;
	temp2.re = Z2.re - Z23.im * h2;
	Z12 = cdiv(temp1, temp2);
	Z12 = cmul(Z12, Z2);

	temp1.im = Z12.im + Z1.re * h1;
	temp1.re = Z12.re - h1*Z1.im;
	temp2.im = Z1.im + Z12.re * h1;
	temp2.re = Z1.re - Z12.im * h1;
	Z01 = cdiv(temp1, temp2);
	Z01 = cmul(Z01, Z1);
	Zin = Z01;
	//Zin = Z1 * (Z3 * (Z2 - h1 * h2 * Z1) + jay * Z2 * (Z2 * h2 + h1 * Z1))/(Z2 * (Z1 - h1 * h2 * Z2) + jay * Z3 * (Z1 * h2 + h1 * Z2));  //输入阻抗
	R = fabs((cabs(Zin) - Z0) / (cabs(Zin) + Z0)); //反射系数
	return R;
}

__global__ void transMat(RayBeamInfo* rays, int raysBeamNum,  int ig, ConfigStruct config, Vector* d_center, Axis_slx New_receive_points) {
	float CSpeed = 1500.0;//shengsu
	float fend = config.time_end_frequency;//姬梓遇20210831
	float fbeg = config.time_start_frequency;//姬梓遇20210831
	float Tao = config.tao/1000.0; //单位:s 姬梓遇20210831
	float fs = config.sampling_frequency;//caiyanglv 姬梓遇20210831

	float velocity1 = config.velocity1;//目标速度1 姬梓遇
	float velocity2 = config.velocity2;//目标速度1 姬梓遇
    float velocity12 = velocity1 - velocity2;//相对投影速度 LV 20220720
	float dopGene = (CSpeed + velocity12) / (CSpeed - velocity12);//多普勒压缩因子
    float band = fend - fbeg;
	if(fabs(band) > 0.001)  //宽带信号则考虑脉冲宽度压缩
		Tao = Tao / dopGene;
	fbeg = config.time_start_frequency * dopGene;//调制后起始频率
	fend = fbeg + band * dopGene;
	band = fend - fbeg;
	float K0 = band / Tao;//wangying	
	//float Z0 ;//询问意义ZO->intel.cu
	int taosize = int(fs * Tao);
	//float wavenum=4.188;
	float start_alpha;

	if(config.continue_alpha == -1)
		start_alpha = config.start_alpha;
	else
		start_alpha = config.continue_alpha;
	
	
	//float tao0= ig  * (1 / fs);
	float wavenum= 2.0 * PI * fbeg / CSpeed;
    float f_ig= 0;
	float reflect_coeff= 1;
	//PointCoor point0, point1, point2;
	//Vector normal_vector;
	int pendZeroNum = 0;//wangying
	comp integralConst1 = { 0,0 };
	comp integralConst2 = { 0,0 };
	comp integralConst = { 0,0 };

	//float preTime = (ig - 1) * (1 / fs);
	//float furTime= (ig + 1) * (1 / fs);
    
    float tao0 = 0;  //snw
	float preTime =0 ;
	float furTime= 0;


	float Vector_proj_mod;
    float alpha00 = 0.0;
	for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < raysBeamNum; idx += gridDim.x * blockDim.x)//2018-12-11
	{

		if (rays[idx].p_cent_distance == 0)//20190116
			return;
        
		//*********************对每一块面片投影多普勒进行计算***********************20221229 snw//
		/*
        Axis_slx Vector_vobj;
		Vector_vobj.p[0] = 1.0 * config.velocity2 / cos(start_alpha * D2R - PI);
		Vector_vobj.p[1] = 0.0 * config.velocity2 / cos(start_alpha * D2R - PI);
		Vector_vobj.p[2] = 0.0 * config.velocity2 / cos(start_alpha * D2R - PI);

		Axis_slx Vector_proj;
	
		Vector_proj.p[0] = d_center[idx].x - New_receive_points.p[0];
		Vector_proj.p[1] = d_center[idx].y - New_receive_points.p[1];
		Vector_proj.p[2] = d_center[idx].z - New_receive_points.p[2];
		Vector_proj_mod = sqrt(Vector_proj.p[0]*Vector_proj.p[0] + Vector_proj.p[1]*Vector_proj.p[1] + Vector_proj.p[2]*Vector_proj.p[2]);
		Vector_proj.p[0] = Vector_proj.p[0] / Vector_proj_mod;
		Vector_proj.p[1] = Vector_proj.p[1] / Vector_proj_mod;
		Vector_proj.p[2] = Vector_proj.p[2] / Vector_proj_mod;
		// 修改前 LV 20230107 
		Axis_slx Vector_vtor;
		alpha00 = PI - start_alpha * D2R - config.H_axis_angle * D2R;     // 修改后 LV 20230707 
		//alpha00 = - (acos(Vector_proj.p[0]) - config.H_axis_angle * D2R); 
		Vector_vtor.p[0] = cos(alpha00) * config.velocity1 / cos(config.H_axis_angle * D2R);
		Vector_vtor.p[1] = 0.0;
		Vector_vtor.p[2] = sin(alpha00) * config.velocity1 / cos(config.H_axis_angle * D2R);

		// ****************
		velocity1 = Vector_vtor.p[0]*Vector_proj.p[0] + Vector_vtor.p[1]*Vector_proj.p[1] + Vector_vtor.p[2]*Vector_proj.p[2];
		velocity2 = Vector_vobj.p[0]*Vector_proj.p[0] + Vector_vobj.p[1]*Vector_proj.p[1] + Vector_vobj.p[2]*Vector_proj.p[2];

		velocity12 = velocity1 - velocity2;//相对投影速度 LV 20220720
		dopGene = (CSpeed + velocity12) / (CSpeed - velocity12);//多普勒压缩因子
		band = config.time_end_frequency - config.time_start_frequency;
		if(fabs(band) > 0.001)              //宽带信号则考虑脉冲宽度压缩
		{
			Tao = config.tao / 1000.0 / dopGene;    // 20221230 错误Tao 替换config.tao/1000
		}

		fbeg = config.time_start_frequency * dopGene;//调制后起始频率
		fend = fbeg + (config.time_end_frequency - config.time_start_frequency) * dopGene; // LV20230106 错误 * dopGene * dopGene ， 应该只有一个 * dopGene
		band = fend - fbeg;
		K0 = band / Tao;

		taosize = int(fs * Tao);*/

		//*********************对每一块面片投影多普勒进行计算***********************// 


		float recv_p_cent_distance = sqrt(pow(d_center[idx].x - New_receive_points.p[0], 2) + pow(d_center[idx].y - New_receive_points.p[1], 2) + pow(d_center[idx].z - New_receive_points.p[2], 2));
		pendZeroNum = int( (recv_p_cent_distance + rays[idx].p_cent_distance) / CSpeed * fs) ;//wangying  snw:pendZeroNum从2倍距离算起 jzy:发射点距离+接收点距离	
		//pendZeroNum = int(2 * rays[idx].p_cent_distance / CSpeed * fs) ;//wangying  snw:pendZeroNum从2倍距离算起
        
       f_ig = fbeg + (ig - pendZeroNum)  * K0 * (1.0 / fs);
       tao0= (ig - pendZeroNum) * (1.0 / fs);//snw
	   preTime = (ig - pendZeroNum - 1) * (1.0 / fs);
	   furTime= (ig - pendZeroNum + 1) * (1.0 / fs);


		if ((ig<pendZeroNum) || (ig>(pendZeroNum + taosize))) //wangying
		{
			rays[idx].parameter.p[0] = 0;
			rays[idx].parameter.p[1] = 0.0;
			rays[idx].parameter.p[2] = 0.0;
		} 
		else {
		/*	Z0 = 1.0f / 4.0f * (rays[idx].ray_index[0].p[0] + rays[idx].ray_index[1].p[0] + rays[idx].ray_index[2].p[0] + rays[idx].ray_index[3].p[0]);
			point0 = rays[idx].ray_index[0];
			point1 = rays[idx].ray_index[1];
			point2 = rays[idx].ray_index[2];
			normal_vector.x = (point0.p[1] - point2.p[1]) * (point1.p[2] - point0.p[2]) - (point1.p[1] - point0.p[1]) * (point0.p[2] - point2.p[2]);
			normal_vector.y = (point0.p[2] - point2.p[2]) * (point1.p[0] - point0.p[0]) - (point1.p[2] - point0.p[2]) * (point0.p[0] - point2.p[0]);
			normal_vector.z = (point0.p[0] - point2.p[0]) * (point1.p[1] - point0.p[1]) - (point1.p[0] - point0.p[0]) * (point0.p[1] - point2.p[1]);
			float gorden = sqrt(normal_vector.x * normal_vector.x + normal_vector.x * normal_vector.x + normal_vector.x * normal_vector.x);*///  gorden->intel.cu

			//rays[idx].parameter.p[0] = gorden;*/
            // LV 20221230 发现 wavenum 计算问题 需修改（原有代码）	
			wavenum = 2 * PI * fbeg / CSpeed;
			//wavenum= 2 * PI * f_ig / CSpeed;// LV 20221230 发现 wavenum 计算问题 修改（开始）
			integralConst1.re = cos(-2.0 * PI * (fbeg * preTime + 0.5 * K0 * tao0 * preTime) + 2.0 * wavenum * rays[idx].Z0); // 20221230 引入固定直流分量？？？  2 * 2 * pi * f_ig / CSpeed * rays[idx].Z0 ??
			integralConst1.im = sin(-2.0 * PI * (fbeg * preTime + 0.5 * K0 * tao0 * preTime) + 2.0 * wavenum * rays[idx].Z0); 

			integralConst2.re = cos(-2.0 * PI * (fbeg * furTime + 0.5 * K0 * tao0 * furTime) + 2.0 * wavenum * rays[idx].Z0);
			integralConst2.im = sin(-2.0 * PI * (fbeg * furTime + 0.5 * K0 * tao0 * furTime) + 2.0 * wavenum * rays[idx].Z0);

			/*wavenum = 2 * PI * f_ig / CSpeed;
			//wavenum= 2 * PI * f_ig / CSpeed;// LV 20221230 发现 wavenum 计算问题 修改（开始）
			integralConst1.re = cos(-2 * PI * (fbeg * preTime + 0.5 * K0 * ig / fs * preTime) + 2 * wavenum * rays[idx].Z0); // 20221230 引入固定直流分量？？？  2 * 2 * pi * f_ig / CSpeed * rays[idx].Z0 ??
			integralConst1.im = sin(-2 * PI * (fbeg * preTime + 0.5 * K0 * ig / fs * preTime) + 2 * wavenum * rays[idx].Z0); 

			integralConst2.re = cos(-2 * PI * (fbeg * furTime + 0.5 * K0 * ig / fs * furTime) + 2 * wavenum * rays[idx].Z0);
			integralConst2.im = sin(-2 * PI * (fbeg * furTime + 0.5 * K0 * ig / fs * furTime) + 2 * wavenum * rays[idx].Z0);*/
            
			//integralConst.re = (integralConst1.re - integralConst2.re) * rays[idx].gorden / rays[idx].p_cent_distance;
			//integralConst.im = (integralConst1.im - integralConst2.im) * rays[idx].gorden / rays[idx].p_cent_distance;
			integralConst.re = ((integralConst1.re - integralConst2.re) * rays[idx].gorden_re -  (integralConst1.im - integralConst2.im) * rays[idx].gorden_im) / rays[idx].p_cent_distance;
			integralConst.im = ((integralConst1.re - integralConst2.re) * rays[idx].gorden_im -  (integralConst1.im - integralConst2.im) * rays[idx].gorden_re) / rays[idx].p_cent_distance;
			reflect_coeff = config.reflect_coeff_Auto_flag ? d_ReflectCoeff_2(f_ig,rays[idx].Z0) : config.reflect_coeff; //反射系数 姬梓遇20210913
			rays[idx].parameter.p[1] = reflect_coeff * integralConst.re;//反射系数 姬梓遇20210913
			rays[idx].parameter.p[2] = reflect_coeff * integralConst.im;//反射系数 姬梓遇20210913
			
		}
	}
}

//__global__ void statis_sum(float *d_s_sum_re, float *d_sum_re, float *d_s_sum_im, float *d_sum_im, int ig, int j, int maxsize){
	//int idx = threadIdx.x + blockDim.x * blockIdx.x;
	//int tid = threadIdx.x;

	//d_s_sum_re[j * maxsize + ig]= *d_sum_re;
	//d_s_sum_im[j * maxsize + ig]= *d_sum_im;
//}



void scalfuc(RayBeamInfo* rays, int raysBeamNum, int ig, float* d_sum_sre, float* d_sum_sim, ConfigStruct config, Vector* d_center, Axis_slx New_receive_points) {

	dim3 threadSize(16, 1, 1);
	dim3 blockSize(raysBeamNum / 16+ 1, 1, 1);


	transMat << <blockSize, threadSize >> > (rays, raysBeamNum, ig, config, d_center, New_receive_points);//20191218 姬梓遇20210831
	int numItem = 0;



	dim3 THREADSIZE(512, 1, 1);
	dim3 BLOCKSIZE(raysBeamNum / 512 + 1, 1, 1);
	//ReduceAdd.
	reduce_add_sre << <BLOCKSIZE, THREADSIZE >> > (d_sum_sre, rays, raysBeamNum);
	hipDeviceSynchronize();
	reduce_add_sim << <BLOCKSIZE, THREADSIZE >> > (d_sum_sim, rays , raysBeamNum);
	hipDeviceSynchronize();
	numItem = BLOCKSIZE.x;
	BLOCKSIZE.x = numItem / THREADSIZE.x + 1;

	while (numItem > 1)
	{
		reduce_sre << <BLOCKSIZE, THREADSIZE >> > (d_sum_sre, d_sum_sre, numItem);
		hipDeviceSynchronize();
		reduce_sim << <BLOCKSIZE, THREADSIZE >> > (d_sum_sim, d_sum_sim, numItem);
		hipDeviceSynchronize();
		numItem = BLOCKSIZE.x;
		BLOCKSIZE.x = numItem / THREADSIZE.x + 1;
	}

	//statis_sum << <1 ,1 >> > (d_s_sum_re, d_sum_sre, d_s_sum_im, d_sum_sim, ig - minZeroNum + 200, j, maxsize);
	//float sum_re = 0;
	//float sum_im = 0;
	//HANDLE_ERROR(hipMemcpy(&sum_re, d_sum_sre, sizeof(float), hipMemcpyDeviceToHost));
	//HANDLE_ERROR(hipMemcpy(&sum_im, d_sum_sim, sizeof(float), hipMemcpyDeviceToHost));

	//comp sum_s = { sum_re, sum_im };

	//return sum_s;



}
