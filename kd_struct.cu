#include "hip/hip_runtime.h"
#include <stdio.h>
#include <malloc.h>
#include <math.h>
#include "kd_struct.h"

int Mid_Choose_Split_Axis(KD_Node *treenode, Prim_Box *array, int arr_length, Element* mid);
bool Cut_Blank(KD_Node *treenode, Prim_Box **array, Prim_Box **arr_left, Prim_Box **arr_right, int arr_length, int *nl, int *nr);
void Count_Prim(Prim_Box *array, int arr_length, DATA_Type Split_Pos, int *nl, int *nr, int *np, int *Axis);
void Count_Prim_Mid(Prim_Box *array, int arr_length, DATA_Type Split_Pos, int *nl, int *nr, int Axis);
int SAH_Choose_Split_Axis(KD_Node *treenode, Prim_Box *array, int arr_length,int *box_flag, DATA_Type *Position,int *np);
void Updata_Node(KD_Node *treenode, Prim_Box *array);
void StopBuild(KD_Node * treenode, Prim_Box *array, Prim_Box *out_array, int arr_length, int *out_arr_length);
//void Preprocessing_Triangles(FILE *fp, Triangle *Triangles, Element *Points, struct Prim_Box *Prim, int Triangle_Num);
void Max_Min(Prim_Box *array, int arr_length, int axis, DATA_Type *max, DATA_Type *min);
void Resize_Array(Prim_Box *array, Prim_Box *arr_left, Prim_Box *arr_right, int arr_length, int nl, int nr, int flag, DATA_Type Split_Pos, int Axis);
void Resize_Array(Prim_Box *array, Prim_Box *arr_left, Prim_Box *arr_right, int arr_length, int nl, int nr, DATA_Type Split_Pos, int Axis);
void Resize_Array_Mid(Prim_Box *array, Prim_Box *arr_left, Prim_Box *arr_right, int arr_length, int nl, int nr, DATA_Type Split_Pos, int Axis);


/********************************************************************************************************
�������ܣ���������Ԫ����Ԥ����ÿ����Ԫ��ʹ�ð�Χ�а�����������SAH����
�������壺1��Ҫ�򿪵��ļ�ָ��
2��������������Ԫ������
3�����涥����Ϣ
4�����潨������Ԫ��Χ��
********************************************************************************************************/
void Preprocessing_Triangles(FILE *fp, Triangle *Triangles, Element *Points, int *Node_Num,int *Triangle_Num)
{
	int i, j;
	int temp;//�������λ��������

/*-----��ȡdat�ļ�------*/
	int node_num;
	int face_num;
	int index = 0,num = 0;
	int tempnum1,tempnum2,tempnum3;

	fscanf(fp,"%d",&num);
	printf("node_num=%d\n",num);						// ���ڵ���;
	float *pointx = new float[num];					// �ڵ��x����
	float *pointy = new float[num];					// �ڵ��y����
	float *pointz = new float[num];					// �ڵ��z����
	while (num)
	{

	        fscanf(fp,"%d",&index);

		///////�������������/////
		fscanf(fp,"%f",&(pointx[index-1]));			// ��x����;     
		fscanf(fp,"%f",&(pointy[index-1]));			// ��y����;
		fscanf(fp,"%f",&(pointz[index-1]));			// ��z����;
                //////////////////////////
		
		///////��(benchmark)����/////
		//fscanf(fp,"%d",&index);
		//fscanf(fp,"%f",&(pointz[index-1]));			// ��x����;     
		//fscanf(fp,"%f",&(pointy[index-1]));			// ��y����;
		//fscanf(fp,"%f",&(pointx[index-1]));			// ��z����;
		//pointx[index-1] = pointx[index-1]  + 24.0*1.5;
		//pointx[index-1] = pointx[index-1] ;
		//pointy[index-1] = pointy[index-1] ;
		//pointz[index-1] = pointz[index-1] ;
		//////////////////////////////

		num--;
	}
	node_num = index;                                               //�ڵ����

	fscanf(fp,"%d",&num);	
	printf("ele_num=%d\n",num);					// ����Ԫ��;
	int *faceOne = new int[num];					// �ڵ��x����
	int *faceTwo = new int[num];					// �ڵ��y����
	int *faceThree = new int[num];					// �ڵ��z����
	while (num)
	{
		fscanf(fp,"%d",&index);
		fscanf(fp,"%d",&(faceOne[index-1]));			// ��x����;     
		fscanf(fp,"%d",&(faceTwo[index-1]));			// ��y����;
		fscanf(fp,"%d",&(faceThree[index-1]));			// ��z����;
		num--;
	}
	
	face_num = index;                                                //��Ԫ����




	for (i = 0; i < node_num; i++)
	{
		
			/*(Points + i)->point[0] = pointx[i];
			(Points + i)->point[1] = pointy[i];
			(Points + i)->point[2] = pointz[i];*/

	        Points[i].point[0] = pointx[i];
			Points[i].point[1] = pointy[i];
			Points[i].point[2] = pointz[i];

		//fscanf(fp,"%d",&(Points + i)->PointsIndex);//��������б��
		//(Points + i)->PointsIndex = i;
                Points[i].PointsIndex = i;
		//printf("\nRead  Point%d", (Points + i)->PointsIndex);
	}

	for (i = 0; i<face_num; i++)
	{

		/*(Triangles + i)->Points[0] = faceOne[i] ;
		(Triangles + i)->Points[1] = faceTwo[i] ;
		(Triangles + i)->Points[2] = faceThree[i] ;

		(Triangles + i)->TriangleIndex = i;*/

        Triangles[i].Points[0] = faceOne[i] - 1;
		Triangles[i].Points[1] = faceTwo[i] - 1;
		Triangles[i].Points[2] = faceThree[i] - 1;

		Triangles[i].TriangleIndex = i;


	}
	*Node_Num = node_num;
	*Triangle_Num = face_num;
	//Sort_Box(Points, Triangles, Prim, Triangle_Num);
}

/*void Preprocessing_Triangles(FILE *fp, Triangle *Triangles, Element *Points, Prim_Box *Prim, int Triangle_Num)
{
	int i, j;
	int temp;//�������λ��������


	for (i = 0; i < Points_Num; i++)
	{
		for (j = 0; j < 3; j++)
			fscanf(fp, "%f", &(Points + i)->point[j]);
		//fscanf(fp,"%d",&(Points + i)->PointsIndex);//��������б��
		(Points + i)->PointsIndex = i;
		//printf("\nRead  Point%d", (Points + i)->PointsIndex);
	}

	for (i = 0; i<Triangle_Num; i++)
	{
		for (j = 0; j < 3; j++)
		{
			fscanf(fp,"%x",&(Triangles + i)->Points[j]);//������Ԫ�ڴ洢�Ķ�����
			(Triangles + i)->Points[j] = (Triangles + i)->Points[j] -1 ;//���ڶ�������ֵ��1��ʼ�����Ҫ��һ
		}
		fscanf(fp, "%d", &temp);//������Ԫ�ڴ洢�Ķ�����
		fscanf(fp, "%d", &temp);//������Ԫ�ڴ洢�Ķ�����
		(Triangles + i)->TriangleIndex = i;
		//fscanf(fp,"%d",&(Triangles + i)->TriangleIndex);//�����α��
		//printf("\nRead  Triangle%d", (Triangles + i)->TriangleIndex);
	}

	Sort_Box(Points, Triangles, Prim, Triangle_Num);
}*/
/*******************************************************
�������ܣ��������������Ԫ�İ�Χ�е��������������ֵ
��������:1����������������Ԫ������
2���������ж��������
3����������Ԫ��Χ�й��ɵ�����
******************************************************/
void Sort_Box(Element points[], Triangle triangle[], Prim_Box *prim, int Triangle_Num)
{

	int i, j;
	DATA_Type Max, Min;
	for (i = 0; i < Triangle_Num; i++)
	{
		for (j = 0; j < 3; j++)
		{
			Max = (points + (triangle + i)->Points[0])->point[j] >(points + (triangle + i)->Points[1])->point[j] ? (points + (triangle + i)->Points[0])->point[j] : (points + (triangle + i)->Points[1])->point[j];
			Max = Max >(points + (triangle + i)->Points[2])->point[j] ? Max : (points + (triangle + i)->Points[2])->point[j];
			Min = (points + (triangle + i)->Points[0])->point[j] < (points + (triangle + i)->Points[1])->point[j] ? (points + (triangle + i)->Points[0])->point[j] : (points + (triangle + i)->Points[1])->point[j];
			Min = Min < (points + (triangle + i)->Points[2])->point[j] ? Min : (points + (triangle + i)->Points[2])->point[j];
			//if (Max != Min)
			//{
				(prim + i)->bmax[j] = Max;
				(prim + i)->bmin[j] = Min;
			//}
			//else//�������δ�ֱ��������ʱ������һ�����
			//{
			//	(prim + i)->bmax[j] = Max+Lambda/2;
			//	(prim + i)->bmin[j] = Min - Lambda/2;
			//}
		}
		(prim + i)->Box_Index = i;
		//ÿ����Χ���������ε�����
		(prim + i)->bmid[0] = ((points + (triangle + i)->Points[0])->point[0] + (points + (triangle + i)->Points[1])->point[0] + (points + (triangle + i)->Points[2])->point[0]) / 3;
		(prim + i)->bmid[1] = ((points + (triangle + i)->Points[0])->point[1] + (points + (triangle + i)->Points[1])->point[1] + (points + (triangle + i)->Points[2])->point[1]) / 3;
		(prim + i)->bmid[2] = ((points + (triangle + i)->Points[0])->point[2] + (points + (triangle + i)->Points[1])->point[2] + (points + (triangle + i)->Points[2])->point[2]) / 3;
	}

}
/****************************************************************************************
�������ܣ��Խڵ���г�ʼ��
�������壺1����ʼ�ڵ�
2��������Ԫ��Χ�й��ɵ�����
����ֵ���壺ѡ��ķָ��� 0.1.2�ֱ����x.y.z��
****************************************************************************************/
void KD_Node_init(struct KD_Node *kd_node, Prim_Box *array,int arr_length)
{//�ڵ��ʼ��
	//kd_node = (struct KD_Node*)(malloc( sizeof( struct KD_Node ) ));
	kd_node->Split_Axis = 0;
	kd_node->Depth = 1;
	kd_node->PrimCount = arr_length;
	kd_node->begin = 0;
	kd_node->end = arr_length - 1;
	//kd_node->NodeIndex = 0;
	for (int i = 0; i < 3; i++)
		kd_node->SplitPos.point[i] = 0;
	kd_node->SplitPos.PointsIndex = 0;
	kd_node->IsLeaf = false;
	kd_node->IsEmpty = false;
	for (int i = 0; i<6; i++)
		kd_node->rope[i] = NULL;
	//����ʼ�ڵ�İ�Χ��ȷ����Χ�����ҵ���Χ������ԭ���������Զ�ĵ�
	Max_Min(array, arr_length, 0, &kd_node->box.bmax[0], &kd_node->box.bmin[0]);
	Max_Min(array, arr_length, 1, &kd_node->box.bmax[1], &kd_node->box.bmin[1]);
	Max_Min(array, arr_length, 2, &kd_node->box.bmax[2], &kd_node->box.bmin[2]);
	return;
}
/******************************** 
 *��������swap 
 *���ã����������ṹ���ֵ 
 *�����������������ṹ�� 
 *����ֵ���� 
 ********************************/  
void swap(Prim_Box *a, Prim_Box *b)    
{  
    Prim_Box temp;  
    temp = *a;  
    *a = *b;  
    *b = temp;  
    return ;  
}  
  
/************************************ 
 *��������quicksort 
 *���ã����������㷨����С��������
 *������ ������Ľṹ������ �������е���ʼλ�ã������н���λ�ã���Ҫ�����ά��
 *����ֵ���� 
 ************************************/  
void QuickSort(Prim_Box* array,  int begin, int end,int dim) 
{  
    int i, j;  
    if(begin < end)  
    {  
        i = begin + 1;  // ��array[begin]��Ϊ��׼������˴�array[begin+1]��ʼ���׼���Ƚϣ�  
        j = end;        // array[end]����������һλ  
            
        while(i < j)  
        {  
            if(  (array+i)->bmid[dim] > (array+begin) ->bmid[dim])  // ����Ƚϵ�����Ԫ�ش��ڻ�׼�����򽻻�λ�á�  
            {  
                swap(&array[i],&array[j]);  // ���������ṹ��
                j--;  
            }  
            else  
            {  
                i++;  // �����������һλ���������׼���Ƚϡ�  
            }  
        }  
		/* ����whileѭ����i = j�� 
         * ��ʱ���鱻�ָ����������  -->  array[begin+1] ~ array[i-1] < array[begin] 
         *                           -->  array[i+1] ~ array[end] > array[begin] 
         * ���ʱ������array�ֳ��������֣��ٽ�array[i]��array[begin]���бȽϣ�����array[i]��λ�á� 
         * ���array[i]��array[begin]���������������ָ�ֵ������Դ����ƣ�ֱ�����i = j�������������˳��� 
         */  
        if( (array+i)->bmid[dim] >= (array+begin)->bmid[dim])  // �������Ҫȡ�ȡ�>=������������Ԫ������ͬ��ֵʱ������ִ���  
        {  
            i--;  
        }  
        swap(&array[begin], &array[i]);  // ����array[i]��array[begin]  
        QuickSort(array, begin, i,dim);  
        QuickSort(array, j, end,dim);  
    }  
}  

/****************************************************************************************
�������ܣ�ʹ���зַ�ѡ��ָ���
�������壺1���ɰ�Χ�й��ɵ�����
2��3��begin��end��ʾ���������������е���ʼ��
4��mid��ʾ�ָ���ֵ
����ֵ���壺ѡ��ķָ��� 0.1.2�ֱ����x.y.z��
****************************************************************************************/
int Mid_Choose_Split_Axis(KD_Node *treenode, Prim_Box *array, int arr_length, Element* mid)
{
	//���������ϵĳ���
	DATA_Type x_dim = 0;
	DATA_Type y_dim = 0;
	DATA_Type z_dim = 0;
	//���������ϵ��м�ֵ�����ָ��
	DATA_Type x_mid = 0;
	DATA_Type y_mid = 0;
	DATA_Type z_mid = 0;
	int nl,nr;
	//��������������������
	//int x_y_z = 0;//0��ʾx����1��ʾy��2��ʾz
	/***********************************************************
	������ά���ϵĳ��ȣ����ҳ�����Ǹ����򣬲���¼�е�ֵ
	***********************************************************/

	x_dim = fabs(treenode->box.bmax[0] - treenode->box.bmin[0]);
	x_mid = x_dim / 2 + treenode->box.bmin[0];

	y_dim = fabs(treenode->box.bmax[1] - treenode->box.bmin[1]);
	y_mid = y_dim / 2 + treenode->box.bmin[1];

	z_dim = fabs(treenode->box.bmax[2] - treenode->box.bmin[2]);
	z_mid = z_dim / 2 + treenode->box.bmin[2];

	if (x_dim >= y_dim && x_dim >= z_dim)
	{
		mid->point[0] = x_mid;
		//Count_Prim_Mid(array, begin, end, x_mid, &nl, &nr, 0);
		//treenode->SplitPos.PointsIndex = nl;
		//Resize_Array_Mid(array, begin, end, nl, nr, x_mid, 0);
		return 0;
	}
	else if (y_dim >= x_dim && y_dim >= z_dim)
	{
		mid->point[1] = y_mid;
		//Count_Prim_Mid(array, begin, end, y_mid, &nl, &nr, 1);
		//treenode->SplitPos.PointsIndex = nl;
		//Resize_Array_Mid(array, begin, end, nl, nr, y_mid, 1);
		return 1;
	}
	else if (z_dim >= y_dim && z_dim >= x_dim)
	{
		mid->point[2] = z_mid;
		//Count_Prim_Mid(array, begin, end, z_mid, &nl, &nr, 2);
		//treenode->SplitPos.PointsIndex = nl;
		//Resize_Array_Mid(array, begin, end, nl, nr, z_mid, 2);
		return 2;
	}
	return -1;
}
/*********************************************************
�������ܣ��޳���Χ���еĿհײ���

*********************************************************/
bool Cut_Blank(KD_Node *treenode, Prim_Box **array, Prim_Box **arr_left, Prim_Box **arr_right, int arr_length, int *nl, int *nr)
{
	DATA_Type max, min, blank, length,empty_rate;
	empty_rate = 0.20;
	for (int i = 0; i < 3; i++)//����ά�����ҿհײ��֣���������ֵʱ���зָ�
	{
		Max_Min(*array, arr_length, i, &max, &min);
		length = max - min;
		blank = fabs(max - treenode->box.bmax[i]);//x�����Һ���Ϊ�սڵ�
		if (blank / length > empty_rate)
		{
			treenode->RightChild->IsEmpty = true;
			treenode->RightChild->PrimCount = 0;

			treenode->LeftChild->PrimCount = arr_length;
			//arr_left = (Prim_Box*)malloc(arr_length*sizeof(Prim_Box));
			*arr_left = *array;//�Һ���Ϊ�գ�����ǰ����ֱ��ת������
			*nl = arr_length;
			*nr = 0;
			treenode->SplitPos.point[i] = max;
			treenode->Split_Axis = i;
			Updata_Node(treenode, *array);
			return true;
		}
		blank = fabs(min - treenode->box.bmin[i]);
		if (blank / length > empty_rate)
		{
			treenode->LeftChild->IsEmpty = true;
			treenode->LeftChild->PrimCount = 0;
			//treenode->RightChild->begin = treenode->begin;
			//treenode->RightChild->end = treenode->end;
			treenode->RightChild->PrimCount = arr_length;
			//arr_right = (Prim_Box*)malloc(arr_length*sizeof(Prim_Box));
			*arr_right = *array;
			*nl = 0;
			*nr = arr_length;
			treenode->SplitPos.point[i] = min;
			treenode->Split_Axis = i;
			Updata_Node(treenode, *array);
			return true;
		}

	}
	return false;
}
/*******************************************************
�������ܣ��ҳ������е������Сֵ
�������壺
1��������
2�����������
3�������ҷָ���
4�����ֵ
5����Сֵ
********************************************************/
void Max_Min(Prim_Box *array, int arr_length, int axis, DATA_Type *max, DATA_Type *min)
{

	*min = array ->bmin[axis];
	*max =array ->bmax[axis];
	for (int i = 0; i < arr_length; i++)
	{
		if ((array + i)->bmin[axis] < *min)
			*min = (array + i)->bmin[axis];
		if ((array + i)->bmax[axis] > *max)
			*max = (array + i)->bmax[axis];
	}
}
/****************************************************************************************
�������ܣ�����ָ�ƽ�������Լ����ָ�ƽ�洩�����Ӱ�Χ�е�����
�������壺1���ɰ�Χ�й��ɵ�����
2��3��begin��end��ʾ���������������е���ʼ��
4��Poisition�ڰ����˷ָ��λ���Լ��ָ�ƽ�����ڰ�Χ�еı��
5���ýڵ��а�Χ�����ֵС�ڷָ�����Ԫ����
6���ýڵ��а�Χ�����ֵ���ڷָ�����Ԫ����
7���ýڵ��а�Χ�б��ָ�ƽ��ָ����Ԫ����
8���ָ���
����ֵ���壺�޷���ֵ
****************************************************************************************/
void Count_Prim(Prim_Box *array, int arr_length, DATA_Type Split_Pos, int *nl, int *nr, int *np, int *Axis)
{
	*nr = 0;
	*nl = 0;
	*np = 0;

	for (Prim_Box *itr = array; itr != (array + arr_length); itr++)
	{
		if (itr->bmax[*Axis] == itr->bmin[*Axis] && (itr->bmax[*Axis]) == Split_Pos)
		{
			//(*nl)++;
			//(*nr)++;
			continue;
		}
		else
		{
			(*nl) += ((itr->bmax[*Axis]) <= Split_Pos ? 1 : 0);//���޸�
			(*nr) += ((itr->bmin[*Axis]) >= Split_Pos ? 1 : 0);
		}

	}
	*np = arr_length - *nr - *nl ;
	return;
}
/****************************************************************************************
�������ܣ�����ָ�ƽ�������Լ����ָ�ƽ�洩�����Ӱ�Χ�е�����
�������壺1���ɰ�Χ�й��ɵ�����
2��3��begin��end��ʾ���������������е���ʼ��
4��Poisition�ڰ����˷ָ��λ���Լ��ָ�ƽ�����ڰ�Χ�еı��
5���ýڵ��а�Χ�����ֵС�ڷָ�����Ԫ����
6���ýڵ��а�Χ�����ֵ���ڷָ�����Ԫ����
7���ýڵ��а�Χ�б��ָ�ƽ��ָ����Ԫ����
8���ָ���
����ֵ���壺�޷���ֵ
****************************************************************************************/
void Count_Prim_Mid(Prim_Box *array, int arr_length, DATA_Type Split_Pos, int *nl, int *nr, int Axis)
{
	*nr = 0;
	*nl = 0;
	for (Prim_Box *itr = array ; itr != (array + arr_length); itr++)
	{
		(*nl) += ((itr->bmin[Axis]) <= Split_Pos ? 1 : 0);//���޸�
		(*nr) += ((itr->bmax[Axis]) >= Split_Pos ? 1 : 0);
	}
	return;
}
/****************************************************************************************
�������ܣ������鰴���ʷ�ƽ�������з֣�������ԭ����λ��
�������壺
1��������
2��3�����������Ԫ�������е���ʼλ������ֹλ��
4��5���ֱ��������ҽڵ���Ԫ������
6���ص���Ԫ������߻����ұߵı�־λ 0����� 1���ұ�
7���ָ�λ��
8���ָ���
****************************************************************************************/
void Resize_Array(Prim_Box *array, Prim_Box *arr_left, Prim_Box *arr_right, int arr_length, int nl, int nr, int flag, DATA_Type Split_Pos, int Axis)
{
	//struct Prim_Box *left = (Prim_Box*)malloc(nl*(sizeof(Prim_Box)));
	//struct Prim_Box *right = (Prim_Box*)malloc(nr*(sizeof(Prim_Box)));
	//memset(left, 0, nl*sizeof(struct Prim_Box));//��ʼ��Ϊ0
	//memset(right, 0, nr*sizeof(struct Prim_Box));//��ʼ��Ϊ0

	if (!flag)//�ص���Ԫ�����
		for (int i = 0; i< arr_length; i++)
		{
			if (((array + i)->bmin[Axis]) >= Split_Pos)
			{
				*arr_right = *(array + i);
				arr_right++;
			}
			else
			{
				*arr_left = *(array + i);
				arr_left++;
			}
		}
	else
		for (int i = 0; i < arr_length; i++)
		{
			if ((array + i)->bmax[Axis] <= Split_Pos)
			{
				*arr_left = *(array + i);
				arr_left++;
			}
			else
			{
				*arr_right = *(array + i);
				arr_right++;
			}
		}
	//array = NULL;
	//free(array);
}
void Resize_Array(Prim_Box *array, Prim_Box *arr_left, Prim_Box *arr_right, int arr_length, int nl, int nr, DATA_Type Split_Pos, int Axis)
{
	//struct Prim_Box *left = (Prim_Box*)malloc(nl*(sizeof(Prim_Box)));
	//struct Prim_Box *right = (Prim_Box*)malloc(nr*(sizeof(Prim_Box)));
	//memset(left, 0, nl*sizeof(struct Prim_Box));//��ʼ��Ϊ0
	//memset(right, 0, nr*sizeof(struct Prim_Box));//��ʼ��Ϊ0
	int l_index, r_index;
	l_index = 0;
	r_index = 0;
	for (int i = 0; i< arr_length; i++)
	{
		if ((array[i].bmax[Axis]) == array[i].bmin[Axis] && (array[i].bmax[Axis]) == Split_Pos)
		{
			arr_right[r_index] = array[i];
			r_index++;
			arr_left[l_index] = array[i];
			l_index++;
			continue;
		}
		else if ((array[i].bmax[Axis]) <= Split_Pos)
		{
			arr_left[l_index] = array[i];
			l_index++;
		}
		else if (array[i].bmin[Axis] >= Split_Pos)
		{
			arr_right[r_index] = array[i];
			r_index++;
		}
		else
		{
			arr_right[r_index] = array[i];
			r_index++;
			arr_left[l_index] = array[i];
			l_index++;
		}
	}
	//printf(" ");
}
void Resize_Array_Mid(Prim_Box *array, Prim_Box *arr_left, Prim_Box *arr_right, int arr_length, int nl, int nr,DATA_Type Split_Pos, int Axis)
{
	//struct Prim_Box *left = (Prim_Box*)malloc(nl*(sizeof(Prim_Box)));
	//struct Prim_Box *right = (Prim_Box*)malloc(nr*(sizeof(Prim_Box)));
	//memset(left, 0, nl*sizeof(struct Prim_Box));//��ʼ��Ϊ0
	//memset(right, 0, nr*sizeof(struct Prim_Box));//��ʼ��Ϊ0
	int l_index, r_index;
	l_index = 0;
	r_index = 0;
	for (int i = 0; i< arr_length; i++)
		{
			if ((array[i].bmax[Axis]) >= Split_Pos)
			{
				arr_right[r_index] = array [i];
				r_index++;
			}
			if (array[i].bmin[Axis]<= Split_Pos)
			{
				arr_left[l_index] = array[i];
				l_index++;
			}
		}
}
/****************************************************************************************
�������ܣ�ʹ��SAH�㷨ѡ��ָ���
�������壺1���ɰ�Χ�й��ɵ�����
2��3��begin��end��ʾ���������������е���ʼ��
4��Poisition�ڰ����˷ָ��λ���Լ��ָ�ƽ�����ڰ�Χ�еı��
����ֵ���壺ѡ��ķָ��� 0.1.2�ֱ����x.y.z��
****************************************************************************************/
int SAH_Choose_Split_Axis(KD_Node *treenode, Prim_Box *array, int arr_length, int *box_flag,struct Element *Position,int *np)
{
	int flag = -1;//ѡȡ��������
	//int box_flag = 0;
	int temp_flag = 0;
	DATA_Type temp1, temp2;//ѡ��ڵ����Ҳ�ı�־
	int c_hit, c_walk,NL,NR,NP;
	c_hit = 19;
	c_walk = 9;//??????????????????????????????????????????
	//int alpha_k = 5;//alpha_k=c_hit/c_walk;
	DATA_Type cost = 0;
	DATA_Type min_cost, min_cost_L, min_cost_R;//ѡ����ʷ�����ѡ�а�Χ�е���߻����ұ�
	DATA_Type Surface,Surface_L,Surface_R;
	DATA_Type H, L, W,
					   H_L,L_L,W_L,
					   H_R,L_R,W_R;
	DATA_Type max, min;
		//��ǰ�ڵ��Χ�еĳ����
		//printf("\nStart Split");
	//Max_Min(array, begin, end, 0, &max, &min);
	L = DataAbs(treenode->box.bmax[0] - treenode->box.bmin[0]);
	//treenode->box.bmin[0] = min;
	//treenode->box.bmax[0] = max;
	//Max_Min(array, begin, end, 1, &max, &min);
	W = DataAbs(treenode->box.bmax[1] - treenode->box.bmin[1]);
	//treenode->box.bmin[1] = min;
	//treenode->box.bmax[1] = max;
	//Max_Min(array, begin, end, 2, &max, &min);
	H = DataAbs(treenode->box.bmax[2] - treenode->box.bmin[2]);
	//treenode->box.bmin[2] = min;
	//treenode->box.bmax[2] = max;
	Surface = (H*L+H*W+W*L);
	min_cost = DATA_Type(c_hit * 10 * (arr_length)+c_walk);//��ʼ����С�Ĵ���Ϊ���ֵ
	for (int i = 0; i < 3; i++)
	{
		min_cost_L = min_cost;
		min_cost_R = min_cost;
		//QuickSort(array, begin, end, i);
		for (int j = 0; j <arr_length; j++)
		{
			if ((array + j)->bmin[i] < treenode->box.bmin[i] || (array + j)->bmax[i] > treenode->box.bmax[i])
				continue;
			switch (i)//�԰�Χ�е���ƽ����Ϊ�ʷ��棬��ѡ���ʷֽڵ�İ�Χ�б������ҽڵ���
			{
				case 0:
				{
					L_L = DataAbs((array + j)->bmin[0] - treenode->box.bmin[0]);
					L_R = DataAbs(treenode->box.bmax[0] - (array + j)->bmin[0]);
					Surface_L =  (H*W + H*L_L + L_L*W);
					Surface_R =  (H*W + H*L_R + L_R*W);
					break;
				}
				case 1:
				{
					W_L = DataAbs((array + j)->bmin[1] - treenode->box.bmin[1]);
					W_R = DataAbs(treenode->box.bmax[1] - (array + j)->bmin[1]);
					Surface_L =  (H*W_L + H*L + L*W_L);
					Surface_R =  (H*W_R + H*L + L* W_R); 
					break;
				}
				case 2:
				{
					H_L = DataAbs((array + j)->bmin[2] - treenode->box.bmin[2]);
					H_R = DataAbs(treenode->box.bmax[2] - (array + j)->bmin[2]);
					Surface_L =  (H_L*W + H_L*L + L*W);
					Surface_R =  (H_R*W + H_R*L + L*W);
					break;
				}
			}
			//printf("\nStart count prim:%d",j);
			Count_Prim(array, arr_length, (array + j)->bmin[i], &NL, &NR, &NP, &i);
			temp1 = (c_walk + (Surface_L*(NL + NP)*c_hit + Surface_R*NR*c_hit) / Surface);
			temp2 = (c_walk + (Surface_L*NL*c_hit + Surface_R*(NR + NP)*c_hit) / Surface);
			if (temp1 <= temp2 )//�ص��İ�Χ���ǻ�����߻����ұ�
			{
				min_cost_L = temp1;
				//NL += NP;
				temp_flag = 0;//�ص���Χ�л��ֱ�־
			}
			else
			{
				min_cost_L = temp2;
				temp_flag = 1;
			}
			if (min_cost_L < min_cost)
			{
				Position->point[i] = (array + j)->bmin[i];//��¼�ָ��λ
				Position->PointsIndex = NL ;//���ص��ڵ���ൽ��ߣ����Χ�д�С��ԭ����һ�£��߽���
				*np = NP;
				min_cost = min_cost_L;
				flag = i;//����ĳά���ϳ��ָ�С�Ĵ���ʱ����¼��ά��
				*box_flag = temp_flag;
			}
			//min_cost_L = min_cost_L <= cost ? min_cost_L : cost;
			switch (i)//�԰�Χ�е���ƽ����Ϊ�ʷ��棬��ѡ���ʷֽڵ�İ�Χ�б�������ڵ���
			{
				case 0:
				{
					L_L = DataAbs((array + j)->bmax[0] - treenode->box.bmin[0]);
					L_R = DataAbs(treenode->box.bmax[0] - (array + j)->bmax[0]);
					Surface_L =  (H*W + H*L_L + L_L*W);
					Surface_R =  (H*W + H*L_R + L_R*W);
					break;
				}
				case 1:
				{
					W_L = DataAbs((array + j)->bmax[1] - treenode->box.bmin[1]);
					W_R = DataAbs(treenode->box.bmax[1] - (array + j)->bmax[1]);
					Surface_L =  (H*W_L + H*L + L*W_L);
					Surface_R =  (H*W_R + H*L + L* W_R);
					break;
				} 
				case 2:
				{
					H_L = DataAbs((array + j)->bmax[2] - treenode->box.bmin[2]);
					H_R = DataAbs(treenode->box.bmax[2] - (array + j)->bmax[2]);
					Surface_L =  (H_L*W + H_L*L + L*W);
					Surface_R =  (H_R*W + H_R*L + L*W);
					break;
				}
			}
			Count_Prim(array, arr_length, (array + j)->bmax[i], &NL, &NR, &NP, &i);
			temp1 = (c_walk + (Surface_L*(NL + NP)*c_hit + Surface_R*NR*c_hit) / Surface);
			temp2 = (c_walk + (Surface_L*NL*c_hit + Surface_R*(NR + NP)*c_hit) / Surface);
			if (temp1 <= temp2)//�ص��İ�Χ���ǻ�����߻����ұ�
			{
				min_cost_R = temp1;
				//NL += NP;
				temp_flag = 0;//�ص���Χ�л��ֱ�־

			}
			else
			{
				min_cost_R = temp2;
				temp_flag = 1;
			}
			if (min_cost_R < min_cost)
			{
				min_cost = min_cost_R;
				Position->point[i] = (array + j)->bmax[i];//��¼�ָ��λ
				Position->PointsIndex =  NL ;//��������ߵ���Ԫ����
				*np = NP;
				flag = i;//����ĳά���ϳ��ָ�С�Ĵ���ʱ����¼��ά��
				*box_flag = temp_flag;
			}
		}
	}
	return flag;//���طָ�ά��
}
/****************************************************************************************
�������ܣ����½ڵ��ڵİ�Χ������ֵ����Ϣ
�������壺
����ֵ���壺ѡ��ķָ��� 0.1.2�ֱ����x.y.z��
****************************************************************************************/
void Updata_Node(KD_Node *treenode, Prim_Box *array)
{
	for (int i = 0; i < 6; i++)//��ǰ�ڵ�̳и��ڵ������ֵ,
	{
		treenode->LeftChild->rope[i] = treenode->rope[i];
		treenode->RightChild->rope[i] = treenode->rope[i];
	}
	//���Һ��Ӽ̳и��ڵ�İ�Χ�е���Ϣ
	for (int i = 0; i < 3; i++)
	{
		treenode->LeftChild->box.bmin[i] = treenode->box.bmin[i];
		treenode->LeftChild->box.bmax[i] = treenode->box.bmax[i];
		treenode->RightChild->box.bmin[i] = treenode->box.bmin[i];
		treenode->RightChild->box.bmax[i] = treenode->box.bmax[i];
	}

	if (treenode->Split_Axis == -1)
	{
		printf("error when choose split");
		//StopBuild(treenode, array);
		return;
	}

	else if (treenode->Split_Axis == 0)
	{
		treenode->LeftChild->rope[1] = treenode->RightChild;
		treenode->RightChild->rope[0] = treenode->LeftChild;
		treenode->LeftChild->box.bmax[0] = treenode->SplitPos.point[treenode->Split_Axis];
		treenode->RightChild->box.bmin[0] = treenode->SplitPos.point[treenode->Split_Axis];
	}
	else if (treenode->Split_Axis == 1)
	{
		treenode->LeftChild->rope[3] = treenode->RightChild;
		treenode->RightChild->rope[2] = treenode->LeftChild;
		treenode->LeftChild->box.bmax[1] = treenode->SplitPos.point[treenode->Split_Axis];
		treenode->RightChild->box.bmin[1] = treenode->SplitPos.point[treenode->Split_Axis];
	}
	else if (treenode->Split_Axis == 2)
	{
		treenode->LeftChild->rope[5] = treenode->RightChild;
		treenode->RightChild->rope[4] = treenode->LeftChild;
		treenode->LeftChild->box.bmax[2] = treenode->SplitPos.point[treenode->Split_Axis];
		treenode->RightChild->box.bmin[2] = treenode->SplitPos.point[treenode->Split_Axis];
	}
	else printf("\nerror when get Split Axis\n");
}
//void Build_Tree(struct KD_Node * treenode,struct Prim_Box *array)
//{//�������� 
//	int i;
//	DATA_Type max, min;
//	if (treenode->PrimCount  < Prim_Min || treenode->end - treenode->begin  < Prim_Min || treenode->Depth >= 20 || treenode->IsEmpty)
//	{
//		StopBuild(treenode, array);
//		return;
//	}
//	
//	else
//	{
//		treenode->PrimCount = treenode -> end - treenode -> begin + 1;//���½ڵ���������Ԫ����
//		treenode->LeftChild = (KD_Node*)malloc(sizeof(KD_Node));
//		treenode->RightChild = (KD_Node*)malloc(sizeof(KD_Node));
//		treenode->LeftChild->IsLeaf = false;
//		treenode->RightChild->IsLeaf = false;
//		treenode->LeftChild->IsEmpty = false;
//		treenode->RightChild->IsEmpty = false;
//		treenode->LeftChild->Depth = treenode->Depth+1;//������������
//		treenode->RightChild->Depth = treenode->Depth+1;//�����Һ�������
//		
//		for( i = 0; i < 6 ;i++)//��ǰ�ڵ�̳и��ڵ������ֵ,
//		{
//			treenode->LeftChild->rope[i] = treenode->rope[i];
//			treenode->RightChild->rope[i] = treenode->rope[i];
//		}
////��������ά�ȷֱ�����,ѡ��ָ��Ტ�ҳ��ָ��
//
//
//		////ִ�еĺ���
//		treenode->Split_Axis = SAH_Choose_Split_Axis(treenode, array, treenode->begin, treenode->end, &treenode->SplitPos);
//		//Max_Min(array, treenode->begin, treenode->end, treenode->Split_Axis, &max, &min);
//
////���Һ��Ӽ̳и��ڵ�İ�Χ�е���Ϣ
//		for (i = 0; i < 3; i++)
//		{
//			treenode->LeftChild->box.bmin[i] = treenode->box.bmin[i];
//			treenode->LeftChild->box.bmax[i] = treenode->box.bmax[i];
//			treenode->RightChild->box.bmin[i] = treenode->box.bmin[i];
//			treenode->RightChild->box.bmax[i] = treenode->box.bmax[i];
//		}
//		for (int i = 0; i < 3; i++)
//		{
//			if (treenode->box.bmax[i] < treenode->box.bmin[i])
//				printf("Box split error");
//		}
//		//QuickSort(array, treenode->begin, treenode->end, treenode->Split_Axis);//���˳�ǰ���վ�����С�ɱ���ά����������
//
////�жϰ���ѡ���ķָ����зָ�Ƿ񲻿��ٷָ�
//		if (treenode->SplitPos.PointsIndex == 0)//�ָ�λ���ڸýڵ�İ�Χ���������Ԫ
//		{
//			if (treenode->SplitPos.point[treenode->Split_Axis] == treenode->box.bmin[treenode->Split_Axis])//�����ŷָ�λ�ò�����,ֹͣ�ָ�
//			{
//				StopBuild(treenode, array);
//				return;
//			}
//			else //�սڵ�j
//			{
//				treenode->LeftChild->IsEmpty = true;
//				treenode->LeftChild->PrimCount = 0;
//				treenode->RightChild->begin = treenode->begin;
//				treenode->RightChild->end = treenode->end;
//				treenode->RightChild->PrimCount = treenode->end - treenode->begin + 1;
//			}
//		}
//		else if (treenode->SplitPos.PointsIndex == treenode->PrimCount)
//		{
//			if (treenode->SplitPos.point[treenode->Split_Axis] == treenode->box.bmax[treenode->Split_Axis])
//			{
//					StopBuild(treenode, array);
//					return;
//			}
//			else
//			{
//				treenode->RightChild->IsEmpty = true;
//				treenode->RightChild->PrimCount = 0;
//				treenode->LeftChild->begin = treenode->begin;
//				treenode->LeftChild->end = treenode->end;
//				treenode->LeftChild->PrimCount = treenode->end - treenode->begin + 1;
//			}
//		}
//		else//�����ӽڵ������Ԫ����ʼ��Χ
//		{
//			treenode->LeftChild->begin = treenode->begin;
//			treenode->LeftChild->end =treenode->begin + treenode->SplitPos.PointsIndex-1;
//			treenode->RightChild->begin = treenode->begin + treenode->SplitPos.PointsIndex;
//			treenode->RightChild->end = treenode->end;
//			treenode->LeftChild->PrimCount = treenode->LeftChild->end - treenode->LeftChild->begin+1;
//			treenode->RightChild->PrimCount = treenode->RightChild->end - treenode->RightChild->begin + 1;
//		}
//		/*==================
//		���ݷָ���������Һ��ӵ�����ֵ�Ͱ�Χ�е�ֵ
//		===================*/
//		if (treenode->Split_Axis == -1)
//		{
//			StopBuild(treenode, array);
//			return;
//		}
//			//printf("\nerror when Choose Split Axis\n");
//		else if (treenode->Split_Axis == 0)
//		{
//			treenode->LeftChild->rope[1] = treenode->RightChild;
//			treenode->RightChild->rope[0] = treenode->LeftChild;
//			treenode->LeftChild->box.bmax[0] = treenode->SplitPos.point[treenode->Split_Axis];
//			treenode->RightChild->box.bmin[0] = treenode->SplitPos.point[treenode->Split_Axis];
//		}
//		else if (treenode->Split_Axis == 1)
//		{
//			treenode->LeftChild->rope[3] = treenode->RightChild;
//			treenode->RightChild->rope[2] = treenode->LeftChild;
//			treenode->LeftChild->box.bmax[1] = treenode->SplitPos.point[treenode->Split_Axis];
//			treenode->RightChild->box.bmin[1] = treenode->SplitPos.point[treenode->Split_Axis];
//		}
//		else if (treenode->Split_Axis == 2)
//		{
//			treenode->LeftChild->rope[5] = treenode->RightChild;
//			treenode->RightChild->rope[4] = treenode->LeftChild;
//			treenode->LeftChild->box.bmax[2] = treenode->SplitPos.point[treenode->Split_Axis];
//			treenode->RightChild->box.bmin[2] = treenode->SplitPos.point[treenode->Split_Axis];
//		}
//		else printf("\nerror when get Split Axis\n");
//
//	}
//
//	Build_Tree(treenode->LeftChild, array);//�ݹ�����Һ���
//	Build_Tree(treenode->RightChild, array);
//}
void Build_BigNode(struct KD_Node * treenode, struct Prim_Box *array ,struct Prim_Box *out_array,int arr_length,int *out_arr_length)
{
	int i,nl,nr,np,box_flag;
	DATA_Type max, min;
	Prim_Box *arr_left, *arr_right;
	arr_left = NULL;
	arr_right = NULL;
	if (treenode == NULL)
	{
		free(treenode);
		return;
	}
	if ((arr_length  < Prim_Min) || (treenode->IsEmpty) || (treenode->Depth == Max_Depth))
	{
		StopBuild(treenode, array,out_array,arr_length,out_arr_length);
		return;
	}
	else
	{
		//treenode->PrimCount = arr_length;//���½ڵ���������Ԫ����
		treenode->LeftChild = (KD_Node*)malloc(sizeof(KD_Node));
		treenode->RightChild = (KD_Node*)malloc(sizeof(KD_Node));
		treenode->LeftChild->IsLeaf = false;
		treenode->RightChild->IsLeaf = false;
		treenode->LeftChild->IsEmpty = false;
		treenode->RightChild->IsEmpty = false;
		treenode->LeftChild->Depth = treenode->Depth + 1;//������������
		treenode->RightChild->Depth = treenode->Depth + 1;//�����Һ�������
		//�зַ�ѡ�ָ���
		if (treenode->PrimCount > BigNodeNum)
		{
			if (!Cut_Blank(treenode, &array,&arr_left,&arr_right,arr_length,&nl,&nr))
			{
				treenode->Split_Axis = Mid_Choose_Split_Axis(treenode, array, arr_length, &treenode->SplitPos);
				
				Count_Prim_Mid(array, arr_length, treenode->SplitPos.point[treenode->Split_Axis], &nl, &nr, treenode->Split_Axis);
				
				arr_left = (Prim_Box*)malloc(nl*sizeof(Prim_Box));
				arr_right = (Prim_Box*)malloc(nr*sizeof(Prim_Box));

				Resize_Array_Mid(array, arr_left, arr_right, arr_length, nl, nr, treenode->SplitPos.point[treenode->Split_Axis], treenode->Split_Axis);
				treenode->LeftChild->PrimCount = nl;
				treenode->RightChild->PrimCount = nr;
				Updata_Node(treenode, array);
				//free(array);
				//array=NULL;
			}
		}
		else
		{
			treenode->Split_Axis = SAH_Choose_Split_Axis(treenode, array, arr_length, &box_flag,&treenode->SplitPos,&np);
			//�жϰ���ѡ���ķָ����зָ�Ƿ񲻿��ٷָ�

			//if ((treenode->SplitPos.PointsIndex + np) == 0)//�ָ�λ���ڸýڵ�İ�Χ���������Ԫ
			//{
				if (treenode->SplitPos.point[treenode->Split_Axis] == treenode->box.bmin[treenode->Split_Axis])//�����ŷָ�λ�ò�����,ֹͣ�ָ�
				{
					StopBuild(treenode, array,out_array,arr_length,out_arr_length);
					free(treenode->LeftChild);
					free(treenode->RightChild);
					treenode->LeftChild = NULL;
					treenode->RightChild = NULL;
					return;
				}
			//	else //�սڵ�j
			//	{
			//		treenode->LeftChild->IsEmpty = true;
			//		treenode->LeftChild->PrimCount = 0;
			//		treenode->RightChild->PrimCount = arr_length;
			//		nl = 0;
			//		nr = arr_length;
			//		arr_right = array;
			//	}
			//}
			//else if (((treenode->SplitPos.PointsIndex + np) == treenode->PrimCount)) 
			//{
				else if (treenode->SplitPos.point[treenode->Split_Axis] == treenode->box.bmax[treenode->Split_Axis])
				{
					StopBuild(treenode, array, out_array, arr_length, out_arr_length);
					free(treenode->LeftChild);
					free(treenode->RightChild);
					treenode->LeftChild = NULL;
					treenode->RightChild = NULL;
					return;
				}
				//else
				//{
				//	treenode->RightChild->IsEmpty = true;
				//	treenode->RightChild->PrimCount = 0;
				//	treenode->LeftChild->PrimCount = arr_length;
				//	nl = arr_length;
				//	nr = 0;
				//	arr_left = array;
				//}
			//}
			else//�����ӽڵ������Ԫ����ʼ��Χ
			{
				//treenode->LeftChild->begin = treenode->begin;
				//treenode->LeftChild->end = treenode->begin + treenode->SplitPos.PointsIndex - 1;
				//treenode->RightChild->begin = treenode->begin + treenode->SplitPos.PointsIndex;
				//treenode->RightChild->end = treenode->end;
				//Count_Prim(array, arr_length, treenode->SplitPos.point[treenode->Split_Axis], &nl, &nr, &np, &treenode->Split_Axis);
				nl = treenode->SplitPos.PointsIndex;
				nr = arr_length - nl - np;
				arr_left = (Prim_Box*)malloc((nl + np)*sizeof(Prim_Box));
				arr_right = (Prim_Box*)malloc((nr + np)*sizeof(Prim_Box));
				Resize_Array(array, arr_left, arr_right, arr_length, (nl+np), (nr+np), treenode->SplitPos.point[treenode->Split_Axis], treenode->Split_Axis);
				treenode->LeftChild->PrimCount = (nl + np);
				treenode->RightChild->PrimCount = (nr + np);
				
			}
			/*==================
			���ݷָ���������Һ��ӵ�����ֵ�Ͱ�Χ�е�ֵ
			===================*/
			Updata_Node(treenode, array);
			//free(array);
			//array=NULL;
		}
	}
	Build_BigNode(treenode->LeftChild, arr_left, out_array, treenode->LeftChild->PrimCount, out_arr_length);//�ݹ�����Һ���
	Build_BigNode(treenode->RightChild, arr_right, out_array, treenode->RightChild->PrimCount, out_arr_length);
	//free(array);
	//free(arr_right);
	//array= NULL;
	//arr_right = NULL;
}

/***************************************************
�������ܣ�����Ҷ�ڵ㣬����ڵ��ڵ�������Ԫ��������
���г��ڵ���������Ԫ���б����ڸýڵ�
��������Ԫ��ֵ�����ڵ��ڲ��Ľṹ�岢���б���
�������壺
****************************************************/
void StopBuild(KD_Node * treenode, Prim_Box *array, Prim_Box *out_array,int arr_length,int *out_arr_length)
{
	if (treenode->IsEmpty == true)
	{
		//free(treenode->LeftChild);
		//free(treenode->RightChild);
		treenode->LeftChild = NULL;
		treenode->RightChild = NULL;
		treenode->PrimCount = 0;
		return;
	}
	else
	{
		treenode->IsLeaf = true;
		treenode->PrimCount = arr_length;
		for (int i = 0; i < arr_length; i++)
		{
			*(out_array + i + *out_arr_length) = *(array + i);
		}
		treenode->begin = *out_arr_length;
		treenode->end = *out_arr_length + arr_length - 1;
		*out_arr_length = *out_arr_length + arr_length;
		free(array);
		//free(treenode->LeftChild);
		//free(treenode->RightChild);
		treenode->LeftChild = NULL;
		treenode->RightChild = NULL;
		array = NULL;
	
	}
	return;
		
}
void Optimization_Rope(KD_Node *treenode)
{
	if (treenode->LeftChild == NULL || treenode->LeftChild->IsEmpty == true || treenode->LeftChild->IsLeaf == true 
		|| treenode->RightChild == NULL || treenode->RightChild->IsEmpty == true || treenode->RightChild->IsLeaf == true)
		return;
	if (treenode->LeftChild->Split_Axis == treenode->RightChild->Split_Axis && treenode->Split_Axis == treenode->LeftChild->Split_Axis)//���ڵ������ӽڵ�ָ�����ͬ
	{
		treenode->LeftChild->RightChild->rope[2 * treenode->Split_Axis + 1] = treenode->RightChild->LeftChild;
		treenode->RightChild->LeftChild->rope[2 * treenode->Split_Axis] = treenode->LeftChild->RightChild;
	}
	else if (treenode->LeftChild->Split_Axis == treenode->RightChild->Split_Axis && treenode->Split_Axis != treenode->LeftChild->Split_Axis)
	{
		if (treenode->RightChild->SplitPos.point[treenode->RightChild->Split_Axis] < treenode->LeftChild->SplitPos.point[treenode->LeftChild->Split_Axis])//���ӽڵ�ָ�λ��С�����ӽڵ�ָ�λ��
		{
			treenode->LeftChild->RightChild->rope[2 * treenode->Split_Axis + 1] = treenode->RightChild->RightChild;
			treenode->RightChild->LeftChild->rope[2 * treenode->Split_Axis] = treenode->LeftChild->LeftChild;
		}
		else if (treenode->RightChild->SplitPos.point[treenode->RightChild->Split_Axis] > treenode->LeftChild->SplitPos.point[treenode->LeftChild->Split_Axis])
		{
			treenode->LeftChild->LeftChild->rope[2 * treenode->Split_Axis + 1] = treenode->RightChild->LeftChild;
			treenode->RightChild->RightChild->rope[2 * treenode->Split_Axis] = treenode->LeftChild->RightChild;
		}
	}
	else if (treenode->Split_Axis == treenode->LeftChild->Split_Axis && treenode->LeftChild->Split_Axis != treenode->RightChild->Split_Axis)
	{
		treenode->RightChild->LeftChild->rope[2 * treenode->Split_Axis] = treenode->LeftChild->RightChild;
		treenode->RightChild->RightChild->rope[2 * treenode->Split_Axis] = treenode->LeftChild->RightChild;
	}
	else if (treenode->Split_Axis == treenode->RightChild->Split_Axis && treenode->LeftChild->Split_Axis != treenode->RightChild->Split_Axis)
	{
		treenode->LeftChild->RightChild->rope[2 * treenode->Split_Axis + 1] = treenode->RightChild->LeftChild;
		treenode->LeftChild->LeftChild->rope[2 * treenode->Split_Axis + 1] = treenode->RightChild->LeftChild;
	}
	Optimization_Rope(treenode->LeftChild);
	Optimization_Rope(treenode->RightChild);
}
//
//void Destroy_Tree(KD_Node *root)
//{
//	if(root)
//	{
//		Destroy_Tree(root->LeftChild);
//		Destroy_Tree(root->RightChild);
//	}
//	free(root);
//	root=NULL;
//}




