
#include <hip/hip_runtime.h>

//�������cpu�汾
//���룺x,y��˵�������һά���顣	rowNum��colNum�ֱ�Ϊ�������ά���� resultΪ������˽����һά����
//����ֵ��-1ʧ��   1�ɹ�
int martixMulti(float* x,int rowNum_X,int colNum_X,float* y,int rowNum_Y,int colNum_Y,float* result)
{
	if(colNum_X != rowNum_Y)
		return -1;
	
	memset(result,0,sizeof(float) * rowNum_X*colNum_Y);

	for(int i=0;i<rowNum_X;i++)
	{
		for(int j=0;j<colNum_Y;j++)
		{
			for(int idx=0;idx<colNum_X;idx++)
			{
				result[i*colNum_Y +j] += x[i*colNum_X + idx] * y[idx*colNum_Y + j];
			}
		}
	}
	return 1;
}
