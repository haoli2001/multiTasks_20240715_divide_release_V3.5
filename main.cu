#include <stdio.h>
#include <pthread.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <semaphore.h>
#ifdef linux
#include <sys/types.h>
#include <netinet/in.h>
#include <sys/socket.h>
#include <sys/wait.h>
#include <unistd.h>
#include <netinet/tcp.h>
pthread_mutex_t socket_mutex;//lihao 20240711传输锁
#endif
#ifdef _UNIX
#include <sys/types.h>
#include <netinet/in.h>
#include <sys/socket.h>
#include <sys/wait.h>
#include <unistd.h>
#include <netinet/tcp.h>
#endif
#ifdef __WINDOWS_
#include <winsock2.h>

#endif
#ifdef _WIN32
#include <winsock2.h>

#endif


#pragma comment(lib, "Ws2_32.lib")

#include "socketFunctions.h"
#include "common_struct.h"
#include "recvThreadFunction.h"
#include "GPUWatchThreadFun.h"

#define PORT 3490  //定义默认端口为3490端口


//服务器socket和连接的客户端socket 
int socketSevice;


int main()
{
   
	pthread_t recvThread;       //计算线程；
	pthread_t GPUWatchThread;   //GPU 监听线程
	int socketClient[10];       //客户端socket
	int curIndex = 0;           //当前连接的id
    
    
	//socket的初始化，绑定，监听以及等待连接
	init_socket();
    //创建套接字
	socketSevice = create_socket();
    //将创建的套接字绑定到指定端口
	if (-1 == bind_listen(socketSevice, PORT))
	{ 
		printf("bind&listen wrong!\n");
		return 0;
	}
	pthread_mutex_init(&socket_mutex,NULL);//lihao 20240711 init the mutex
    //循环监听连接信息
    while(true) 
    {
        //监听连接，当建立建立后accept_client 函数返回，否则阻塞
        socketClient[curIndex] = accept_client(socketSevice);
        
        int deviceCount;
        //获取服务器GPU设备数
        hipGetDeviceCount(&deviceCount); 
        for (int dev = 0; dev < deviceCount; dev++)
        {
            hipSetDevice(dev);
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, dev);
            Frame frame;
            DeviceInfo gpuInfo;
            strcpy(frame.command, "DeviceInfo");
            gpuInfo.deviceID = dev;
            gpuInfo.deviceCount = deviceCount;
            strcpy(gpuInfo.deviceName, deviceProp.name);
            //获取设备每个多处理器的核心数量
            gpuInfo.coresPreMutiprocess = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor); 
            //多处理器数量 
            gpuInfo.mutiprocessCount = deviceProp.multiProcessorCount;          
            frame.length = sizeof(DeviceInfo);
            memcpy(frame.data, (char*)&gpuInfo, sizeof(gpuInfo));
            //向客户端发送设备信息
#ifdef linux
			pthread_mutex_lock(&socket_mutex);//lihao 20240711 
#endif
            send_frame(socketClient[curIndex], (char*)&frame, sizeof(Frame));   
#ifdef linux
			pthread_mutex_unlock(&socket_mutex);//lihao 20240711  
#endif
        }
        pthread_create(&recvThread, NULL, recvThreadFunction, (void*)&(socketClient[curIndex]));    //创建接收数据的线程
        pthread_create(&GPUWatchThread,NULL , GPUWatchThreadFun, (void*)&(socketClient[curIndex]));  //创建GPU实时监控的线程
        curIndex++;
    }
	
    //计算结束，关闭socket
#ifdef linux
	for(int i = curIndex-1;i>=0;i--)
	{	
		shutdown(socketClient[i], SHUT_RDWR);
		close(socketClient[i]);
	}
	shutdown(socketSevice, SHUT_RDWR);
	close(socketSevice);
	pthread_mutex_destroy(&socket_mutex);//lihao 20240711 销毁锁
#endif
#ifdef _UNIX
	for(int i = curIndex-1;i>=0;i--)
	{	
		shutdown(socketClient[i], SHUT_RDWR);
		close(socketClient[i]);
	}
	shutdown(socketClient, SHUT_RDWR);
	shutdown(socketSevice, SHUT_RDWR);
	close(socketClient);
	close(socketSevice);
#endif
#ifdef __WINDOWS_
	for(int i = curIndex-1;i>=0;i--)
	{	
		close(socketClient[i]);
	}
	close(socketSevice);
#endif
#ifdef _WIN32
	for(int i = curIndex-1;i>=0;i--)
	{	
		close(socketClient[i]);
	}
	closesocket(socketSevice);
#endif
	return 0;
}
