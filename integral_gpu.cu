#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<>
#include<hip/device_functions.h>
#include<string.h>
#include "integral_gpu.h"
#include "handlerror.h"
#include "sm_20_atomic_functions.h"

__global__ void reduce_add_re(float* d_sum_re, ReimOutput* d_in, int raysBeamNum)
{
	__shared__ float sdata[512];

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;

	sdata[tid] = (idx < raysBeamNum) ? d_in[idx].re : 0;
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s){
			sdata[tid] +=  sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		d_sum_re[blockIdx.x] = sdata[0];
	}
}

__global__ void reduce_re(float* d_sum_re, float* d_in, int Num)
{
	__shared__ float sdata[512];

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;

	sdata[tid] = (idx < Num) ? d_in[idx] : 0;
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s){
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		d_sum_re[blockIdx.x] = sdata[0];
	}
}

__global__ void reduce_add_im(float* d_sum_im, ReimOutput* d_in, int raysBeamNum)
{
	__shared__ float sdata[512];

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;

	sdata[tid] = (idx < raysBeamNum) ? d_in[idx].im : 0;
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s){
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		d_sum_im[blockIdx.x] = sdata[0];
	}
}

__global__ void reduce_im(float* d_sum_im, float* d_in, int Num)
{
	__shared__ float sdata[512];

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;

	sdata[tid] = (idx < Num) ? d_in[idx] : 0;
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s){
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		d_sum_im[blockIdx.x] = sdata[0];
	}
}

//�������
comp product(comp c1, comp c2)
{
	comp x = {0,0};
	x.re = c1.re * c2.re - c1.im * c2.im;
	x.im = c1.im * c2.re + c2.im * c1.re;
	return x;
}

__device__ comp complex_product(comp c1, comp c2)
{
	comp x = {0,0};
	x.re = c1.re * c2.re - c1.im * c2.im;
	x.im = c1.im * c2.re + c2.im * c1.re;
	return x;
}

__device__ comp complex_add(comp c1, comp c2)
{
	comp sum = {0,0};
	sum.re = c1.re + c2.re;
	sum.im = c1.im + c2.im;
	return sum;
}

__device__ float sa_function(float f1, float f2, float wavenum)
{
	float result = 0;
	if (f2 == 0)
	{
		result = f1;
	}
	else
	{
		result = f1 * (sinf(wavenum / 2 * f2) / (wavenum / 2 * f2));
	}
	return result;
}

__global__ void integral(RayBeamInfo* d_effrays, int raysBeamNum, float wavenumber, ReimOutput* d_reim)
{
	float WaveNumber = 2 * wavenumber;
	float d_omgdelt1, d_omgdelt2, tmp;
	comp d_ctemp, d_reim_tmp = {0,0};

	PointCoor point0, point1, point2;
	Vector normal_vector, center_vecter;//向量求面积的参数，声束管线中心点位置
	PointCoor launch_point, nomal_launch_point;//声源点位置，归一化之后声源点位置20201014
	float launch_point_model;
	
	for (int idx = threadIdx.x + blockIdx.x*blockDim.x; idx < raysBeamNum; idx += blockDim.x*gridDim.x)
	{
		if(d_effrays[idx].p_cent_distance == 0)//20190116
			return;
			
		PointCoor3 parameter = d_effrays[idx].parameter;
		float p_cent_distance = d_effrays[idx].p_cent_distance;
		PointCoor p_2d[5];
		p_2d[0] = d_effrays[idx].point_2D[0];
		p_2d[1] = d_effrays[idx].point_2D[1];
		p_2d[2] = d_effrays[idx].point_2D[2];
		p_2d[3] = d_effrays[idx].point_2D[3];
		p_2d[4] = d_effrays[idx].point_2D[0];
			
		for (int i = 0; i < 4; i++)
		{
			d_omgdelt1 = parameter.p[0] * (p_2d[i + 1].p[0] - p_2d[i].p[0]) + parameter.p[1] * (p_2d[i + 1].p[1] - p_2d[i].p[1]);
			d_omgdelt2 = parameter.p[1] * (p_2d[i + 1].p[0] - p_2d[i].p[0]) - parameter.p[0] * (p_2d[i + 1].p[1] - p_2d[i].p[1]);
			tmp = sa_function(d_omgdelt2, d_omgdelt1, WaveNumber);
			d_ctemp.re = 0;
			d_ctemp.im = -(parameter.p[0] * WaveNumber * (p_2d[i + 1].p[0] + p_2d[i].p[0]) / 2 + parameter.p[1] * WaveNumber * (p_2d[i + 1].p[1] + p_2d[i].p[1]) / 2);
			d_ctemp.re = cosf(d_ctemp.im) * tmp;
			d_ctemp.im = sinf(d_ctemp.im) * tmp;
			d_reim_tmp = complex_add(d_reim_tmp, d_ctemp);
		}
		parameter.p[0] = (parameter.p[0] == 0) ? 9999999999 : parameter.p[0];
		parameter.p[1] = (parameter.p[1] == 0) ? 9999999999 : parameter.p[1];

		d_reim_tmp.re = d_reim_tmp.re / WaveNumber / (parameter.p[0] * parameter.p[0] + parameter.p[1] * parameter.p[1]);
		d_reim_tmp.im = d_reim_tmp.im / WaveNumber / (parameter.p[0] * parameter.p[0] + parameter.p[1] * parameter.p[1]);

		d_ctemp.re = 0;
		d_ctemp.im = 2 * wavenumber * p_cent_distance;
		d_ctemp.re = cosf(d_ctemp.im);
		d_ctemp.im = sinf(d_ctemp.im);

		d_reim_tmp = complex_product(d_reim_tmp, d_ctemp);
		d_ctemp.re = -1 / p_cent_distance / p_cent_distance / p_cent_distance / (-2 * PI);
		d_ctemp.im = wavenumber / p_cent_distance / p_cent_distance / (-2 * PI);
		d_reim_tmp = complex_product(d_reim_tmp, d_ctemp);

		d_reim[idx].re = d_reim_tmp.re * parameter.p[2];
		d_reim[idx].im = d_reim_tmp.im * parameter.p[2];
		//2023.2.9
		d_effrays[idx].gorden_re = d_reim[idx].re;
		d_effrays[idx].gorden_im = d_reim[idx].im;

		launch_point = d_effrays[idx].launch_point;
		launch_point_model = sqrt(launch_point.p[0] * launch_point.p[0] + launch_point.p[1] * launch_point.p[1] + launch_point.p[2] * launch_point.p[2]);
		nomal_launch_point.p[0] = launch_point.p[0] / launch_point_model;
		nomal_launch_point.p[1] = launch_point.p[1] / launch_point_model;
		nomal_launch_point.p[2] = launch_point.p[2] / launch_point_model;
		center_vecter.x = 1.0f / 4.0f * (d_effrays[idx].ray_index[0].p[0] + d_effrays[idx].ray_index[1].p[0] + d_effrays[idx].ray_index[2].p[0] + d_effrays[idx].ray_index[3].p[0]);
		center_vecter.y = 1.0f / 4.0f * (d_effrays[idx].ray_index[0].p[1] + d_effrays[idx].ray_index[1].p[1] + d_effrays[idx].ray_index[2].p[1] + d_effrays[idx].ray_index[3].p[1]);
		center_vecter.z = 1.0f / 4.0f * (d_effrays[idx].ray_index[0].p[2] + d_effrays[idx].ray_index[1].p[2] + d_effrays[idx].ray_index[2].p[2] + d_effrays[idx].ray_index[3].p[2]);
		d_effrays[idx].Z0 = center_vecter.x * nomal_launch_point.p[0] + center_vecter.y * nomal_launch_point.p[1] + center_vecter.z * nomal_launch_point.p[2];


		point0 = d_effrays[idx].ray_index[0];
		point1 = d_effrays[idx].ray_index[1];
		point2 = d_effrays[idx].ray_index[2];
		normal_vector.x = (point0.p[1] - point2.p[1]) * (point1.p[2] - point0.p[2]) - (point1.p[1] - point0.p[1]) * (point0.p[2] - point2.p[2]);
		normal_vector.y = (point0.p[2] - point2.p[2]) * (point1.p[0] - point0.p[0]) - (point1.p[2] - point0.p[2]) * (point0.p[0] - point2.p[0]);
		normal_vector.z = (point0.p[0] - point2.p[0]) * (point1.p[1] - point0.p[1]) - (point1.p[0] - point0.p[0]) * (point0.p[1] - point2.p[1]);
		d_effrays[idx].gorden = sqrt(normal_vector.x * normal_vector.x + normal_vector.y * normal_vector.y + normal_vector.z * normal_vector.z);
	}
}

__global__ void parameter(MatStruct* transMat, RayBeamInfo* d_effrays, int raysBeamNum)
{
	float tmp_data;
	
	for (int idx = threadIdx.x + blockIdx.x*blockDim.x; idx < raysBeamNum; idx += blockDim.x*gridDim.x)
	{
		if(d_effrays[idx].p_cent_distance == 0)
			return;
			
		MatStruct resg_mat = transMat[idx];
		PointCoor tmp_ray = d_effrays[idx].launch_point;
		float dis = d_effrays[idx].p_cent_distance;
		for (int k = 0; k < 3; k++)
		{
			tmp_data = 0;
			for (int i = 0; i < 4; i++)
			{
				tmp_data += resg_mat.p[(4 * k) + i] * tmp_ray.p[i];
			}
			d_effrays[idx].parameter.p[k] = tmp_data / dis;
		}
	}
}

__global__ void coordinate2D(MatStruct* transMat, RayBeamInfo* d_effrays, int raysBeamNum)
{
	float tmp_data;
	PointCoor tmp_ray;
	__shared__ MatStruct s_mat[8];

	int tid = threadIdx.x;
	int bdx = blockIdx.x; 

	if ((bdx * 8 + tid / 4) >= raysBeamNum || d_effrays[bdx * 8 + tid / 4].p_cent_distance == 0)
		return;

	s_mat[tid / 4] = transMat[bdx * 8 + tid / 4];
	__syncthreads();

	tmp_ray = d_effrays[bdx * 8 + tid / 4].ray_index[tid % 4];
	for (int k = 0; k < 3; k++)
	{
		tmp_data = 0;
		for (int i = 0; i < 4; i++)
		{
			tmp_data += s_mat[tid/4].p[(4 * k) + i] * tmp_ray.p[i];
		}
		d_effrays[bdx * 8 + tid / 4].point_2D[tid % 4].p[k] = tmp_data;
	}
	
	
	/*MatStruct resg_mat;//20190427
	float tmp_data;
	PointCoor tmp_ray;
	for (int idx = threadIdx.x + blockIdx.x*blockDim.x; idx < raysBeamNum; idx += blockDim.x*gridDim.x)
	{
		//if (d_effrays[idx].p_cent_distance == 0)//20190116
		//	return;
			
		resg_mat = transMat[idx];

		for (int j = 0; j < 4; j++)
		{
			tmp_ray = d_effrays[idx].ray_index[j];
			for (int k = 0; k < 3; k++)
			{
				tmp_data = 0;
				for (int i = 0; i < 4; i++)
				{
					tmp_data += resg_mat.p[(4 * k) + i] * tmp_ray.p[i];
				}
				d_effrays[idx].point_2D[j].p[k] = tmp_data;
			}
		}
	}*/

}


__global__ void build_transMat(RayBeamInfo* rays, int raysBeamNum, Vector* d_axis, Vector* d_center, MatStruct* transMat)
{

	//20191218

	float tmp;
	Vector vector_x, vector_y, vector_z;
	Vector normal_vector;

	PointCoor point0, point1, point2;
	Vector point_center;
	
	for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < raysBeamNum; idx += gridDim.x * blockDim.x)//2018-12-11
	{

		if(rays[idx].p_cent_distance == 0)//20190116
			return;

		point0 = rays[idx].ray_index[0];
		point1 = rays[idx].ray_index[1];
		point2 = rays[idx].ray_index[2];

		normal_vector.x = (point0.p[1] - point2.p[1]) * (point1.p[2] - point0.p[2]) - (point1.p[1] - point0.p[1]) * (point0.p[2] - point2.p[2]);
		normal_vector.y = (point0.p[2] - point2.p[2]) * (point1.p[0] - point0.p[0]) - (point1.p[2] - point0.p[2]) * (point0.p[0] - point2.p[0]);
		normal_vector.z = (point0.p[0] - point2.p[0]) * (point1.p[1] - point0.p[1]) - (point1.p[0] - point0.p[0]) * (point0.p[1] - point2.p[1]);
		tmp = normal_vector.x * normal_vector.x + normal_vector.y * normal_vector.y + normal_vector.z * normal_vector.z;
		vector_z.x = normal_vector.x / sqrtf(tmp);
		vector_z.y = normal_vector.y / sqrtf(tmp);
		vector_z.z = normal_vector.z / sqrtf(tmp);//z

		point_center = d_center[idx];

		tmp = (point0.p[0] - point_center.x)*(point0.p[0] - point_center.x) \
			+ (point0.p[1] - point_center.y)*(point0.p[1] - point_center.y) \
			+ (point0.p[2] - point_center.z)*(point0.p[2] - point_center.z);

		//ͶӰ����ϵx��
		vector_x.x = (point0.p[0] - point_center.x) / sqrtf(tmp);
		vector_x.y = (point0.p[1] - point_center.y) / sqrtf(tmp);
		vector_x.z = (point0.p[2] - point_center.z) / sqrtf(tmp);//x

		//ͶӰ����ϵy��
		vector_y.x = -(vector_x.y * vector_z.z - vector_z.y * vector_x.z);
		vector_y.y = -(vector_x.z * vector_z.x - vector_z.z * vector_x.x);
		vector_y.z = -(vector_x.x * vector_z.y - vector_z.x * vector_x.y);//y

		transMat[idx].p[0] = vector_x.x;//x
		transMat[idx].p[1] = vector_x.y;
		transMat[idx].p[2] = vector_x.z;
		transMat[idx].p[3] = -point_center.x * vector_x.x - point_center.y * vector_x.y - point_center.z * vector_x.z;
		transMat[idx].p[4] = vector_y.x;//y
		transMat[idx].p[5] = vector_y.y;
		transMat[idx].p[6] = vector_y.z;
		transMat[idx].p[7] = -point_center.x * vector_y.x - point_center.y * vector_y.y - point_center.z * vector_y.z;
		transMat[idx].p[8] = vector_z.x;//z
		transMat[idx].p[9] = vector_z.y;
		transMat[idx].p[10] = vector_z.z;
		transMat[idx].p[11] = -point_center.x * vector_z.x - point_center.y * vector_z.y - point_center.z * vector_z.z;
	}
}

__global__ void copy_data_gpu(Direction *d_rays, Square *d_squares, RayBeamInfo* d_rays_save, int raysBeamNum, Vector* d_center, PointCoor launchpoint_tmp, ReimOutput* d_reim, ConfigStruct* config)
{
	float CSpeed = 1500.0;//shengsu 

	//float fs = config->sampling_frequency;//caiyanglv 姬梓遇20210831
	for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < raysBeamNum; idx += gridDim.x * blockDim.x)//2018-12-11
	{
		//if ((d_squares[idx].right == true) && (d_rays[d_squares[idx].ray_index[4]].triangle_index != -1))
		if ((d_squares[idx].right == true))
		{
			float ray00 = d_rays[d_squares[idx].CornerRayIndex.x].p.x;
			float ray01 = d_rays[d_squares[idx].CornerRayIndex.x].p.y;
			float ray02 = d_rays[d_squares[idx].CornerRayIndex.x].p.z;
			int triangle_index = d_rays[d_squares[idx].CornerRayIndex.x].triangle_index;
			float ray10 = d_rays[d_squares[idx].CornerRayIndex.y].p.x;
			float ray11 = d_rays[d_squares[idx].CornerRayIndex.y].p.y;
			float ray12 = d_rays[d_squares[idx].CornerRayIndex.y].p.z;
			float ray20 = d_rays[d_squares[idx].CornerRayIndex.z].p.x;
			float ray21 = d_rays[d_squares[idx].CornerRayIndex.z].p.y;
			float ray22 = d_rays[d_squares[idx].CornerRayIndex.z].p.z;
			float ray30 = d_rays[d_squares[idx].CornerRayIndex.w].p.x;
			float ray31 = d_rays[d_squares[idx].CornerRayIndex.w].p.y;
			float ray32 = d_rays[d_squares[idx].CornerRayIndex.w].p.z;
			d_center[idx].x = 1.0f / 4.0f * (ray00 + ray10 + ray20 + ray30);
			d_center[idx].y = 1.0f / 4.0f * (ray01 + ray11 + ray21 + ray31);
			d_center[idx].z = 1.0f / 4.0f * (ray02 + ray12 + ray22 + ray32);
			d_rays_save[idx].p_cent_distance = sqrt(pow(d_center[idx].x - launchpoint_tmp.p[0], 2) + pow(d_center[idx].y - launchpoint_tmp.p[1], 2) + pow(d_center[idx].z - launchpoint_tmp.p[2], 2));
			d_rays_save[idx].launch_point = launchpoint_tmp;
			d_rays_save[idx].ray_index[0].p[0] = ray00;
			d_rays_save[idx].ray_index[0].p[1] = ray01;
			d_rays_save[idx].ray_index[0].p[2] = ray02;
			d_rays_save[idx].ray_index[0].p[3] = 1.0f;
			d_rays_save[idx].ray_index[1].p[0] = ray10;
			d_rays_save[idx].ray_index[1].p[1] = ray11;
			d_rays_save[idx].ray_index[1].p[2] = ray12;
			d_rays_save[idx].ray_index[1].p[3] = 1.0f;
			d_rays_save[idx].ray_index[2].p[0] = ray20;
			d_rays_save[idx].ray_index[2].p[1] = ray21;
			d_rays_save[idx].ray_index[2].p[2] = ray22;
			d_rays_save[idx].ray_index[2].p[3] = 1.0f;
			d_rays_save[idx].ray_index[3].p[0] = ray30;
			d_rays_save[idx].ray_index[3].p[1] = ray31;
			d_rays_save[idx].ray_index[3].p[2] = ray32;
			d_rays_save[idx].ray_index[3].p[3] = 1.0f;
			d_reim[idx].triangle_index = triangle_index;
			//d_rays_save[idx].launch_point.p[0] = x;//20190423
			//d_rays_save[idx].launch_point.p[1] = y;
			//d_rays_save[idx].launch_point.p[2] = z;
			//d_rays_save[idx].launch_point.p[3] = 1.0f;
			/*d_rays_save[idx].launch_point = launchpoint_tmp;
			d_rays_save[idx].ray_index[0].p[0] = d_rays[d_squares[idx].ray_index[0]].p[0];
			d_rays_save[idx].ray_index[0].p[1] = d_rays[d_squares[idx].ray_index[0]].p[1];
			d_rays_save[idx].ray_index[0].p[2] = d_rays[d_squares[idx].ray_index[0]].p[2];
			d_rays_save[idx].ray_index[0].p[3] = 1.0f;
			d_rays_save[idx].ray_index[1].p[0] = d_rays[d_squares[idx].ray_index[1]].p[0];
			d_rays_save[idx].ray_index[1].p[1] = d_rays[d_squares[idx].ray_index[1]].p[1];
			d_rays_save[idx].ray_index[1].p[2] = d_rays[d_squares[idx].ray_index[1]].p[2];
			d_rays_save[idx].ray_index[1].p[3] = 1.0f;
			d_rays_save[idx].ray_index[2].p[0] = d_rays[d_squares[idx].ray_index[2]].p[0];
			d_rays_save[idx].ray_index[2].p[1] = d_rays[d_squares[idx].ray_index[2]].p[1];
			d_rays_save[idx].ray_index[2].p[2] = d_rays[d_squares[idx].ray_index[2]].p[2];
			d_rays_save[idx].ray_index[2].p[3] = 1.0f;
			d_rays_save[idx].ray_index[3].p[0] = d_rays[d_squares[idx].ray_index[3]].p[0];
			d_rays_save[idx].ray_index[3].p[1] = d_rays[d_squares[idx].ray_index[3]].p[1];
			d_rays_save[idx].ray_index[3].p[2] = d_rays[d_squares[idx].ray_index[3]].p[2];
			d_rays_save[idx].ray_index[3].p[3] = 1.0f;
			d_center[idx].x = 1.0f / 4.0f * (d_rays_save[idx].ray_index[0].p[0] + d_rays_save[idx].ray_index[1].p[0] + d_rays_save[idx].ray_index[2].p[0] + d_rays_save[idx].ray_index[3].p[0]);
			d_center[idx].y = 1.0f / 4.0f * (d_rays_save[idx].ray_index[0].p[1] + d_rays_save[idx].ray_index[1].p[1] + d_rays_save[idx].ray_index[2].p[1] + d_rays_save[idx].ray_index[3].p[1]);
			d_center[idx].z = 1.0f / 4.0f * (d_rays_save[idx].ray_index[0].p[2] + d_rays_save[idx].ray_index[1].p[2] + d_rays_save[idx].ray_index[2].p[2] + d_rays_save[idx].ray_index[3].p[2]);
			d_rays_save[idx].p_cent_distance = sqrt(pow(d_center[idx].x - launchpoint_tmp.p[0], 2) + pow(d_center[idx].y - launchpoint_tmp.p[1], 2) + pow(d_center[idx].z - launchpoint_tmp.p[2], 2));*/
		}
	}
}

comp sound_field_integral_gpu(Direction *d_rays, Square *d_squares, float wavelength, int raysBeamNum, RayBeamInfo* d_effrays, Vector* d_center, Vector* d_axis, 
	MatStruct* d_transMat, ReimOutput* d_reim, float* d_sum_re, float* d_sum_im, float fi, float si,  ConfigStruct* config)
{
	float wavenumber = (2 * PI) / wavelength;

	
	float x = config->far_distance * sin(PI / 180 * fi)*cos(PI / 180 * si);
	float y = config->far_distance * sin(PI / 180 * fi)*sin(PI / 180 * si);
	float z = config->far_distance * cos(PI / 180 * fi);
	PointCoor launchpoint_tmp = {x, y, z, 1.0f};

	dim3 threadSize(16, 1, 1);
	dim3 blockSize(raysBeamNum / 16 + 1, 1, 1);
	
	//��ʱ
	//hipEvent_t start, stop;
	//HANDLE_ERROR(hipEventCreate(&start));
	//HANDLE_ERROR(hipEventCreate(&stop));
	//HANDLE_ERROR(hipEventRecord(start, 0));
	//���ݿ�������

	copy_data_gpu << <blockSize, threadSize >> >(d_rays, d_squares, d_effrays, raysBeamNum, d_center, launchpoint_tmp, d_reim, config);


	//HANDLE_ERROR(hipDeviceSynchronize());
	//HANDLE_ERROR(hipEventRecord(stop, 0));
	//HANDLE_ERROR(hipEventSynchronize(stop));
	//float elapsedTime;
	//HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	//printf("Time to copy data at %d: %fms\n", degree, elapsedTime);
	//HANDLE_ERROR(hipEventDestroy(start));
	//HANDLE_ERROR(hipEventDestroy(stop));

	build_transMat << <blockSize, threadSize >> >(d_effrays, raysBeamNum, d_axis, d_center, d_transMat);//20191218
	//HANDLE_ERROR(hipDeviceSynchronize());

	//compute_axis_x_gpu << <blockSize, threadSize >> >(d_effrays, raysBeamNum, d_center, d_axis);
	//HANDLE_ERROR(hipDeviceSynchronize());

	//build_transMat << <blockSize, threadSize >> >(d_effrays, d_transMat, raysBeamNum, d_center, d_axis);
	//HANDLE_ERROR(hipDeviceSynchronize());

	dim3 threadSize1(32, 1, 1);
	dim3 blockSize1(raysBeamNum/8+1, 1, 1);
	coordinate2D << <blockSize1, threadSize1 >> >(d_transMat, d_effrays, raysBeamNum);
	//HANDLE_ERROR(hipDeviceSynchronize());
	
	//coordinate2D << <raysBeamNum / 64 + 1, 64 >> >(d_transMat, d_effrays, raysBeamNum);

	parameter << <blockSize, threadSize >> >(d_transMat, d_effrays, raysBeamNum);
	//HANDLE_ERROR(hipDeviceSynchronize());

	integral << <blockSize, threadSize >> >(d_effrays, raysBeamNum, wavenumber, d_reim);
	//HANDLE_ERROR(hipDeviceSynchronize());

	int numItem = 0;
	
	dim3 THREADSIZE(512, 1, 1);
	dim3 BLOCKSIZE(raysBeamNum / 512 + 1, 1, 1);
	//ReduceAdd.
	reduce_add_re<<<BLOCKSIZE,THREADSIZE>>>(d_sum_re, d_reim, raysBeamNum);
	hipDeviceSynchronize();
	reduce_add_im<<<BLOCKSIZE,THREADSIZE>>>(d_sum_im, d_reim, raysBeamNum);
	hipDeviceSynchronize();
	numItem = BLOCKSIZE.x;
	BLOCKSIZE.x = numItem / THREADSIZE.x + 1;

	while (numItem > 1)
	{
		reduce_re<<<BLOCKSIZE,THREADSIZE>>>(d_sum_re, d_sum_re, numItem);
		hipDeviceSynchronize();
		reduce_im<<<BLOCKSIZE,THREADSIZE>>>(d_sum_im, d_sum_im, numItem);
		hipDeviceSynchronize();
		numItem = BLOCKSIZE.x;
		BLOCKSIZE.x = numItem / THREADSIZE.x + 1;
	}

	float sum_re = 0;
	float sum_im = 0;

     

	HANDLE_ERROR(hipMemcpy(&sum_re, d_sum_re, sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(&sum_im, d_sum_im, sizeof(float), hipMemcpyDeviceToHost));



	//����ǿ�ȼ���
	comp sum = { sum_re, sum_im };

	return sum;
}

comp sound_field_integral_gpu_DivRay(Direction *d_rays, Square *d_squares, float wavelength, int raysBeamNum,
	RayBeamInfo* d_effrays, Vector* d_center, Vector* d_axis, MatStruct* d_transMat, ReimOutput* d_reim, float* d_sum_re, float* d_sum_im, float fi, float si,  ConfigStruct* config)
{
	raysBeamNum *= 4;
	
	float x = config->far_distance * sin(PI / 180 * fi)*cos(PI / 180 * si);
	float y = config->far_distance * sin(PI / 180 * fi)*sin(PI / 180 * si);
	float z = config->far_distance * cos(PI / 180 * fi);
	PointCoor launchpoint_tmp = {x, y, z, 1.0f};
	
	float wavenumber = (2 * PI) / wavelength;

	dim3 threadSize(16, 1, 1);
	dim3 blockSize(raysBeamNum / 16 + 1, 1, 1);

	copy_data_gpu<< <blockSize, threadSize >> >(d_rays, d_squares, d_effrays, raysBeamNum, d_center, launchpoint_tmp, d_reim, config);
	//HANDLE_ERROR(hipDeviceSynchronize());

	/*RayBeamInfo * h_effrays;
	h_effrays = (RayBeamInfo*)malloc(15976009 * sizeof(RayBeamInfo));
	HANDLE_ERROR(hipMemcpy(h_effrays, d_effrays, 15976009 * sizeof(RayBeamInfo), hipMemcpyDeviceToHost));
	
	printf("�ĸ��Ƕ�����---------------------\n");
	for(int m=1050; m<1100;m++)
	{
		printf("%d ",m);
		for (int i = 0; i < 4; i++)
		{
			for (int j = 0; j < 4; j++)
			{
				printf("%.7f ", h_effrays[m].ray_index[i].p[j]);
			}
			printf("\n");
		}
	}
	printf("---------------------\n");*/

	build_transMat << <blockSize, threadSize >> >(d_effrays, raysBeamNum, d_axis, d_center, d_transMat);
	//HANDLE_ERROR(hipDeviceSynchronize());

	//compute_axis_x_gpu << <blockSize, threadSize >> >(d_effrays, raysBeamNum, d_center, d_axis);
	//HANDLE_ERROR(hipDeviceSynchronize());
	
	//build_transMat << <blockSize, threadSize >> >(d_effrays, d_transMat, raysBeamNum, d_center, d_axis);
	//HANDLE_ERROR(hipDeviceSynchronize());

	//float* h_transMat;
	//h_transMat = (float*)malloc(320356 * 12 * sizeof(float));
	//HANDLE_ERROR(hipMemcpy2D(h_transMat, 12 * sizeof(float), d_transMat, pitch, 12 * sizeof(float), 320356, hipMemcpyDeviceToHost));
	//
	//printf("�任����---------------------\n");
	//for (int i = 164; i < 165; i++)
	//{
	//	for (int j = 0; j < 12; ++j)
	//	{
	//		printf("%.7f ", h_transMat[i*(12) + j]);
	//		if ((j == 3) || (j == 7) || (j == 11))
	//			printf("\n");
	//	}
	//}
	//printf("---------------------\n");

	//dim3 ThreadPerBlock(1, 1, 1);
	//dim3 BlockPerGrid(raysBeamNum, 1, 1);
	//coordinate2D << <blockSize, threadSize >> >(d_transMat, d_effrays, raysBeamNum);
	dim3 threadSize1(32, 1, 1);
	dim3 blockSize1(raysBeamNum / 8 + 1, 1, 1);
	coordinate2D << <blockSize1, threadSize1 >> >(d_transMat, d_effrays, raysBeamNum);
	//HANDLE_ERROR(hipDeviceSynchronize());

	//coordinate2D << <blockSize, threadSize >> >(d_transMat, d_effrays, raysBeamNum);

	parameter << <blockSize, threadSize >> >(d_transMat, d_effrays, raysBeamNum);
	//HANDLE_ERROR(hipDeviceSynchronize());

	/*HANDLE_ERROR(hipMemcpy(h_effrays, d_effrays, 15976009 * sizeof(RayBeamInfo), hipMemcpyDeviceToHost));
	printf("u,v,w = ");
	for(int m=1050; m<1100;m++)
	{
		printf("%d ",m);
		for (int j = 0; j < 3; j++)
		{
			printf("%.7f ", h_effrays[m].parameter.p[j]);
		}
		printf("\n");
	}
	printf("\n---------------------\n");*/

	integral << <blockSize, threadSize >> >(d_effrays, raysBeamNum, wavenumber, d_reim);
	//HANDLE_ERROR(hipDeviceSynchronize());
	
	/*float sum_reim_re = 0;
	float sum_reim_im = 0;
	comp* h_reim;
	h_reim = (comp*)malloc(15976009 * sizeof(comp));
	HANDLE_ERROR(hipMemcpy(h_reim, d_reim, 15976009 * sizeof(comp), hipMemcpyDeviceToHost));
	for (int i = 0; i < raysBeamNum; i++)
	{
		sum_reim_re += h_reim[i].re;
		sum_reim_im += h_reim[i].im;
		printf("%d reim = %.16f,%.16f\n",i, h_reim[i].re, h_reim[i].im);
	}
	printf("reim = %.16f,%.16f\n", sum_reim_re, sum_reim_im) ;*/
	
	int numItem = 0;
	dim3 THREADSIZE(512, 1, 1);
	dim3 BLOCKSIZE(raysBeamNum / 512 + 1, 1, 1);
	//ReduceAdd.
	reduce_add_re << <BLOCKSIZE, THREADSIZE >> >(d_sum_re, d_reim, raysBeamNum);
	hipDeviceSynchronize();
	reduce_add_im << <BLOCKSIZE, THREADSIZE >> >(d_sum_im, d_reim, raysBeamNum);
	hipDeviceSynchronize();
	numItem = BLOCKSIZE.x;
	BLOCKSIZE.x = numItem / THREADSIZE.x + 1;

	while (numItem > 1)
	{
		reduce_re << <BLOCKSIZE, THREADSIZE >> >(d_sum_re, d_sum_re, numItem);
		hipDeviceSynchronize();
		reduce_im << <BLOCKSIZE, THREADSIZE >> >(d_sum_im, d_sum_im, numItem);
		hipDeviceSynchronize();
		numItem = BLOCKSIZE.x;
		BLOCKSIZE.x = numItem / THREADSIZE.x + 1;
	}

	float sum_re = 0;
	float sum_im = 0;
	HANDLE_ERROR(hipMemcpy(&sum_re, d_sum_re, sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(&sum_im, d_sum_im, sizeof(float), hipMemcpyDeviceToHost));

	//����ǿ�ȼ���
	comp sum = { sum_re, sum_im };
	//printf("integral1: %.16f,%.16f\n", sum.re, sum.im);
	return sum;
}

comp sound_field_integral_gpu_DivRay3(Direction *d_rays, Square *d_squares, float wavelength, int raysBeamNum,
	RayBeamInfo* d_effrays, Vector* d_center, Vector* d_axis, MatStruct* d_transMat, ReimOutput* d_reim, float* d_sum_re, float* d_sum_im, float fi, float si,  ConfigStruct* config)
{
	raysBeamNum *= 4;
	
	float x = config->far_distance * sin(PI / 180 * fi)*cos(PI / 180 * si);  //snw
	float y = config->far_distance * sin(PI / 180 * fi)*sin(PI / 180 * si);
	float z = config->far_distance * cos(PI / 180 * fi);
	PointCoor launchpoint_tmp = {x, y, z, 1.0f};
	
	float wavenumber = (2 * PI) / wavelength;

	dim3 threadSize(16, 1, 1);
	dim3 blockSize(raysBeamNum / 16 + 1, 1, 1);

	copy_data_gpu<< <blockSize, threadSize >> >(d_rays, d_squares, d_effrays, raysBeamNum, d_center, launchpoint_tmp, d_reim, config);
	//HANDLE_ERROR(hipDeviceSynchronize());

	/*RayBeamInfo * h_effrays;
	h_effrays = (RayBeamInfo*)malloc(15976009 * sizeof(RayBeamInfo));
	HANDLE_ERROR(hipMemcpy(h_effrays, d_effrays, 15976009 * sizeof(RayBeamInfo), hipMemcpyDeviceToHost));
	
	printf("�ĸ��Ƕ�����---------------------\n");
	for(int m=9716; m<9717;m++)
	{
		//printf("%d ",m);
		for (int i = 0; i < 4; i++)
		{
			for (int j = 0; j < 4; j++)
			{
				printf("%.7f ", h_effrays[m].ray_index[i].p[j]);
			}
			printf("\n");
		}
	}
	printf("---------------------\n");
	printf("distance:%f\n",h_effrays[9716].p_cent_distance);*/
	
	/*Vector * h_center;
	h_center = (Vector*)malloc(15976009 * sizeof(Vector));
	HANDLE_ERROR(hipMemcpy(h_center, d_center, 15976009 * sizeof(Vector), hipMemcpyDeviceToHost));
	printf("���ĵ�����---------------------\n");
	for(int m=9716; m<9717;m++)
	{
		printf("%.7f %.7f %.7f\n", h_center[m].x, h_center[m].y, h_center[m].z);
	}
	printf("---------------------\n");*/

	build_transMat << <blockSize, threadSize >> >(d_effrays, raysBeamNum, d_axis, d_center, d_transMat);
	//HANDLE_ERROR(hipDeviceSynchronize());
	
	/*Vector * h_axis;
	h_axis = (Vector*)malloc(3 * 15976009 * sizeof(Vector));
	HANDLE_ERROR(hipMemcpy(h_axis, d_axis, 3 * 15976009 * sizeof(Vector), hipMemcpyDeviceToHost));
	printf("---------------------\n");
	for(int m=9716; m<9717;m++)
	{
		printf("%.7f %.7f %.7f\n", h_axis[m].x, h_axis[m].y, h_axis[m].z);
	}
	printf("---------------------\n");*/

	//compute_axis_x_gpu << <blockSize, threadSize >> >(d_effrays, raysBeamNum, d_center, d_axis);
	//HANDLE_ERROR(hipDeviceSynchronize());

	/*HANDLE_ERROR(hipMemcpy(h_axis, d_axis, 3 * 15976009 * sizeof(Vector), hipMemcpyDeviceToHost));
	printf("---------------------\n");
	for(int m=9716+9564644*2; m<9717+9564644*2;m++)
	{
		printf("%.7f %.7f %.7f\n", h_axis[m].x, h_axis[m].y, h_axis[m].z);
	}
	for(int m=9716+9564644; m<9717+9564644;m++)
	{
		printf("%.7f %.7f %.7f\n", h_axis[m].x, h_axis[m].y, h_axis[m].z);
	}
	printf("---------------------\n");*/
	
	//build_transMat << <blockSize, threadSize >> >(d_effrays, d_transMat, raysBeamNum, d_center, d_axis);
	//HANDLE_ERROR(hipDeviceSynchronize());

	/*MatStruct* h_transMat;
	h_transMat = (MatStruct*)malloc(15976009 * sizeof(MatStruct));
	HANDLE_ERROR(hipMemcpy(h_transMat, d_transMat, sizeof(MatStruct) * 15976009, hipMemcpyDeviceToHost));
	
	printf("�任����---------------------\n");
	for (int i = 9716; i < 9717; i++)
	{
		for (int j = 0; j < 12; ++j)
		{
			printf("%.7f ", h_transMat[i].p[j]);
			if ((j == 3) || (j == 7) || (j == 11))
				printf("\n");
		}
	}
	printf("---------------------\n");*/

	//dim3 ThreadPerBlock(1, 1, 1);
	//dim3 BlockPerGrid(raysBeamNum, 1, 1);
	//coordinate2D << <blockSize, threadSize >> >(d_transMat, d_effrays, raysBeamNum);
	dim3 threadSize1(32, 1, 1);
	dim3 blockSize1(raysBeamNum / 8 + 1, 1, 1);
	coordinate2D << <blockSize1, threadSize1 >> >(d_transMat, d_effrays, raysBeamNum);
	//HANDLE_ERROR(hipDeviceSynchronize());
	
	//coordinate2D << <blockSize, threadSize >> >(d_transMat, d_effrays, raysBeamNum);

	parameter << <blockSize, threadSize >> >(d_transMat, d_effrays, raysBeamNum);
	//HANDLE_ERROR(hipDeviceSynchronize());

	/*HANDLE_ERROR(hipMemcpy(h_effrays, d_effrays, 15976009 * sizeof(RayBeamInfo), hipMemcpyDeviceToHost));
	printf("u,v,w = ");
	for(int m=9716; m<9717;m++)
	{
		printf("%d ",m);
		for (int j = 0; j < 3; j++)
		{
			printf("%.7f ", h_effrays[m].parameter.p[j]);
		}
		printf("\n");
	}
	printf("\n---------------------\n");*/

	integral << <blockSize, threadSize >> >(d_effrays, raysBeamNum, wavenumber, d_reim);
	//HANDLE_ERROR(hipDeviceSynchronize());
	
	/*float sum_reim_re = 0;
	float sum_reim_im = 0;
	comp* h_reim;
	h_reim = (comp*)malloc(15976009 * sizeof(comp));
	HANDLE_ERROR(hipMemcpy(h_reim, d_reim, 15976009 * sizeof(comp), hipMemcpyDeviceToHost));
	for (int i = 9716; i < 9717; i++)
	{
		//sum_reim_re += h_reim[i].re;
		//sum_reim_im += h_reim[i].im;
		printf("%d reim = %.16f,%.16f\n",i, h_reim[i].re, h_reim[i].im);
	}
	//printf("reim = %.16f,%.16f\n", sum_reim_re, sum_reim_im) ;*/
	
	int numItem = 0;
	dim3 THREADSIZE(512, 1, 1);
	dim3 BLOCKSIZE(raysBeamNum / 512 + 1, 1, 1);
	//ReduceAdd.
	reduce_add_re << <BLOCKSIZE, THREADSIZE >> >(d_sum_re, d_reim, raysBeamNum);
	hipDeviceSynchronize();
	reduce_add_im << <BLOCKSIZE, THREADSIZE >> >(d_sum_im, d_reim, raysBeamNum);
	hipDeviceSynchronize();
	numItem = BLOCKSIZE.x;
	BLOCKSIZE.x = numItem / THREADSIZE.x + 1;

	while (numItem > 1)
	{
		reduce_re << <BLOCKSIZE, THREADSIZE >> >(d_sum_re, d_sum_re, numItem);
		hipDeviceSynchronize();
		reduce_im << <BLOCKSIZE, THREADSIZE >> >(d_sum_im, d_sum_im, numItem);
		hipDeviceSynchronize();
		numItem = BLOCKSIZE.x;
		BLOCKSIZE.x = numItem / THREADSIZE.x + 1;
	}

	float sum_re = 0;
	float sum_im = 0;
	HANDLE_ERROR(hipMemcpy(&sum_re, d_sum_re, sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(&sum_im, d_sum_im, sizeof(float), hipMemcpyDeviceToHost));

	//����ǿ�ȼ���
	comp sum = { sum_re, sum_im };
	//printf("integral1: %.16f,%.16f\n", sum.re, sum.im);
	return sum;
}

float TS_compute(float far_dis,comp sum, float wavelength)  //����ǿ�ȼ���
{
	comp reim = { 0, 0 };
	float mag = 0;
	float intensity = 0;
	reim.re = 0;
	reim.im = -(2 * PI) / wavelength * far_dis;
	reim.re = cosf(reim.im);
	reim.im = sinf(reim.im);
	sum = product(sum, reim);
	mag = sqrt(sum.re * sum.re + sum.im * sum.im);
	if (mag < 99999999999.0)
		intensity = 20 * log10f(mag * far_dis * far_dis);
	return intensity;
}

//float sound_field_integral_gpu_f(Direction *d_rays, Square *d_squares, float wavelength, int width, int height)
//{
//	//Direction   * d_rays;
//	//Square      * d_squares;
//	RayBeamInfo * d_effrays;
//	Vector      * d_center;
//	Vector      * d_axis_z;
//	PointCoor   * ac;
//	PointCoor   * bd;
//	float       * nv_length;
//	Vector      * normal_vector;
//	Vector      * d_axis_x;
//	float       * d_vectorX_length;
//	Vector      * d_axis_y;
//	size_t      pitch;
//	size_t      pitch1;
//	size_t 	    pitch2;
//	float       * d_transMat;
//	float intensity = 0;
//	int nrow = 4;
//	int ncol = 4;
//	float wavenumber = (2 * PI) / wavelength;
//	int num = 0;
//	int raysNum = (width + 1) * (height + 1) + width * height;
//	int raysBeamNum = width * height;
//
//	/*HANDLE_ERROR(hipMalloc((void**)&d_rays, raysNum * sizeof(Direction)));
//	HANDLE_ERROR(hipMalloc((void**)&d_squares, raysBeamNum * sizeof(Square)));*/
//	HANDLE_ERROR(hipMalloc((void**)&d_effrays, raysBeamNum * sizeof(RayBeamInfo)));
//	HANDLE_ERROR(hipMalloc((void**)&d_center, raysBeamNum * sizeof(Vector)));
//	HANDLE_ERROR(hipMalloc((void**)&d_axis_z, raysBeamNum * sizeof(Vector)));
//	HANDLE_ERROR(hipMalloc((void**)&ac, raysBeamNum * sizeof(PointCoor)));
//	HANDLE_ERROR(hipMalloc((void**)&bd, raysBeamNum * sizeof(PointCoor)));
//	HANDLE_ERROR(hipMalloc((void**)&nv_length, raysBeamNum * sizeof(float)));
//	HANDLE_ERROR(hipMalloc((void**)&normal_vector, raysBeamNum * sizeof(Vector)));
//	HANDLE_ERROR(hipMalloc((void**)&d_axis_x, raysBeamNum * sizeof(Vector)));
//	HANDLE_ERROR(hipMalloc((void**)&d_vectorX_length, raysBeamNum * sizeof(float)));
//	HANDLE_ERROR(hipMalloc((void**)&d_axis_y, raysBeamNum * sizeof(Vector)));
//	HANDLE_ERROR(hipMallocPitch((void**)&d_transMat, &pitch, nrow * ncol * sizeof(float), raysBeamNum));
//
//	/*HANDLE_ERROR(hipMemcpy(d_rays, rays, raysNum * sizeof(Direction), hipMemcpyHostToDevice));
//	HANDLE_ERROR(hipMemcpy(d_squares, squares, raysBeamNum * sizeof(Square), hipMemcpyHostToDevice));*/
//
//	HANDLE_ERROR(hipMemset(d_effrays, 0, raysBeamNum * sizeof(RayBeamInfo)));
//	HANDLE_ERROR(hipMemset(d_center, 0, raysBeamNum * sizeof(Vector)));
//	HANDLE_ERROR(hipMemset(d_axis_z, 0, raysBeamNum * sizeof(Vector)));
//	HANDLE_ERROR(hipMemset(ac, 0, raysBeamNum * sizeof(PointCoor)));
//	HANDLE_ERROR(hipMemset(bd, 0, raysBeamNum * sizeof(PointCoor)));
//	HANDLE_ERROR(hipMemset(nv_length, 0, raysBeamNum * sizeof(float)));
//	HANDLE_ERROR(hipMemset(normal_vector, 0, raysBeamNum * sizeof(Vector)));
//	HANDLE_ERROR(hipMemset(d_axis_x, 0, raysBeamNum * sizeof(Vector)));
//	HANDLE_ERROR(hipMemset(d_vectorX_length, 0, raysBeamNum * sizeof(float)));
//	HANDLE_ERROR(hipMemset(d_axis_y, 0, raysBeamNum * sizeof(Vector)));
//	HANDLE_ERROR(hipMemset2D(d_transMat, pitch, 0, nrow * ncol * sizeof(float), raysBeamNum));
//
//	dim3 threadSize(512, 1, 1);
//	dim3 blockSize(width * height / 512 + 1, 1, 1);
//
//	//��ʱ
//	//hipEvent_t start, stop;
//	//HANDLE_ERROR(hipEventCreate(&start));
//	//HANDLE_ERROR(hipEventCreate(&stop));
//	//HANDLE_ERROR(hipEventRecord(start, 0));
//	//���ݿ�������
//	copy_data_gpu << <blockSize, threadSize >> >(d_rays, d_squares, d_effrays, width, height);
//	hipDeviceSynchronize();
//
//
//	//HANDLE_ERROR(hipEventRecord(stop, 0));
//	//HANDLE_ERROR(hipEventSynchronize(stop));
//	//float elapsedTime;
//	//HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
//	//printf("Time to copy data at %d: %fms\n", degree, elapsedTime);
//	//HANDLE_ERROR(hipEventDestroy(start));
//	//HANDLE_ERROR(hipEventDestroy(stop));
//
//	//RayBeamInfo * h_effrays;
//	//h_effrays = (RayBeamInfo*)malloc(raysBeamNum*sizeof(RayBeamInfo));
//	//HANDLE_ERROR(hipMemcpy(h_effrays, d_effrays, raysBeamNum*sizeof(RayBeamInfo), hipMemcpyDeviceToHost));
//
//	//printf("�ĸ��Ƕ�����---------------------\n");
//	//for (int i = 0; i < 4; i++)
//	//{
//	//for (int j = 0; j < 4; j++)
//	//{
//	//printf("%.7f ", h_effrays[30078].ray_index[i].p[j]);
//	//}
//	//printf("\n");
//	//}
//	//printf("---------------------\n");
//
//	compute_axis_z_gpu << <blockSize, threadSize >> >(d_effrays, width, height, ac, bd, nv_length, normal_vector, d_axis_z);
//	hipDeviceSynchronize();
//	hipFree(ac);
//	hipFree(bd);
//	hipFree(nv_length);
//	hipFree(normal_vector);
//
//	compute_axis_x_gpu << <blockSize, threadSize >> >(d_effrays, width, height, d_center, d_vectorX_length, d_axis_x);
//	hipDeviceSynchronize();
//
//	//Vector * h_center;
//	//h_center = (Vector*)malloc(raysBeamNum*sizeof(Vector));
//	//HANDLE_ERROR(hipMemcpy(h_center, d_center, raysBeamNum*sizeof(Vector), hipMemcpyDeviceToHost));
//	//printf("center = %.7f, %.7f, %.7f\n", h_center[30078].x, h_center[30078].y, h_center[30078].z);
//	//free(h_center);
//
//	//float * h_vectorX_length;
//	//h_vectorX_length = (float*)malloc(raysBeamNum*sizeof(float));
//	//HANDLE_ERROR(hipMemcpy(h_vectorX_length, d_vectorX_length, raysBeamNum*sizeof(float), hipMemcpyDeviceToHost));
//	//printf("length = %.7f\n", h_vectorX_length[30078]);
//	//free(h_vectorX_length);
//
//	hipFree(d_vectorX_length);
//
//	compute_axis_y_gpu << <blockSize, threadSize >> >(d_axis_x, d_axis_z, width, height, d_axis_y);
//	hipDeviceSynchronize();
//
//	build_transMat << <blockSize, threadSize >> >(d_transMat, width, height, d_axis_x, d_axis_y, d_axis_z, d_center, pitch);
//	hipDeviceSynchronize();
//	//float* h_transMat;
//	//h_transMat = (float*)malloc(raysBeamNum * nrow * ncol * sizeof(float));
//	//HANDLE_ERROR(hipMemcpy2D(h_transMat, nrow * ncol * sizeof(float), d_transMat, pitch, nrow * ncol * sizeof(float), raysBeamNum, hipMemcpyDeviceToHost));
//
//	//printf("�任����---------------------\n");
//	//for (int i = 30078; i < 30079; i++)
//	//{
//	//for (int j = 0; j < 16; ++j)
//	//{
//	//printf("%.7f ", h_transMat[i*(nrow * ncol) + j]);
//	//if ((j == 3) || (j == 7) || (j == 11) || (j == 15))
//	//printf("\n");
//	//}
//	//}
//	//printf("---------------------\n");
//
//	hipFree(d_axis_z);
//	hipFree(d_axis_x);
//	hipFree(d_axis_y);
//	hipFree(d_center);
//
//	float* tempvar;
//	HANDLE_ERROR(hipMallocPitch((void**)&tempvar, &pitch1, nrow * ncol * sizeof(float), raysBeamNum));
//	HANDLE_ERROR(hipMemset2D(tempvar, pitch1, 0, nrow * ncol * sizeof(float), raysBeamNum));
//	coordinate2D << <blockSize, threadSize >> >(d_transMat, d_effrays, tempvar, raysBeamNum, pitch, pitch1);
//	hipDeviceSynchronize();
//	hipFree(tempvar);
//
//	distance << <blockSize, threadSize >> >(d_effrays, raysBeamNum);
//	hipDeviceSynchronize();
//
//	float* tempVar;
//	HANDLE_ERROR(hipMallocPitch((void**)&tempVar, &pitch2, ncol * sizeof(float), raysBeamNum));
//	HANDLE_ERROR(hipMemset2D(tempVar, pitch2, 0, ncol * sizeof(float), raysBeamNum));
//	parameter << <blockSize, threadSize >> >(d_transMat, d_effrays, tempVar, raysBeamNum, pitch, pitch2);
//	hipDeviceSynchronize();
//	hipFree(d_transMat);
//	hipFree(tempVar);
//
//	//HANDLE_ERROR(hipMemcpy(h_effrays, d_effrays, raysBeamNum*sizeof(RayBeamInfo), hipMemcpyDeviceToHost));
//	//printf("u,v,w = ");
//	//for (int j = 0; j < 3; j++)
//	//{
//	//printf("%.7f ", h_effrays[30078].parameter.p[j]);
//	//}
//	//printf("\n---------------------\n");
//
//	float       * d_omgdelt1;
//	float       * d_omgdelt2;
//	float       * d_tempVar;
//	comp        * d_ctemp;
//	comp        * d_reim;
//	HANDLE_ERROR(hipMalloc((void**)&d_omgdelt1, raysBeamNum * sizeof(float)));
//	HANDLE_ERROR(hipMalloc((void**)&d_omgdelt2, raysBeamNum * sizeof(float)));
//	HANDLE_ERROR(hipMalloc((void**)&d_tempVar, raysBeamNum * sizeof(float)));
//	HANDLE_ERROR(hipMalloc((void**)&d_ctemp, raysBeamNum * sizeof(comp)));
//	HANDLE_ERROR(hipMalloc((void**)&d_reim, raysBeamNum * sizeof(comp)));
//	HANDLE_ERROR(hipMemset(d_omgdelt1, 0, raysBeamNum * sizeof(float)));
//	HANDLE_ERROR(hipMemset(d_omgdelt2, 0, raysBeamNum * sizeof(float)));
//	HANDLE_ERROR(hipMemset(d_tempVar, 0, raysBeamNum * sizeof(float)));
//	HANDLE_ERROR(hipMemset(d_ctemp, 0, raysBeamNum * sizeof(comp)));
//	HANDLE_ERROR(hipMemset(d_reim, 0, raysBeamNum * sizeof(comp)));
//
//	integral << <blockSize, threadSize >> >(d_effrays, raysBeamNum, wavenumber, d_omgdelt1, d_omgdelt2, d_tempVar, d_ctemp, d_reim);
//	hipDeviceSynchronize();
//
//	//comp* h_reim;
//	//h_reim = (comp*)malloc(raysBeamNum*sizeof(comp));
//	//HANDLE_ERROR(hipMemcpy(h_reim, d_reim, raysBeamNum*sizeof(comp), hipMemcpyDeviceToHost));
//	//for(int i=15000;i<15500;i++)
//	//{
//	//printf("%d reim = %.10f,%.10f\n",i, h_reim[i].re, h_reim[i].im);
//	//}
//
//	hipFree(d_omgdelt1);
//	hipFree(d_omgdelt2);
//	hipFree(d_tempVar);
//	hipFree(d_ctemp);
//
//	int numItem = 0;
//	float* d_sum_re;
//	float* d_sum_im;
//	HANDLE_ERROR(hipMalloc((void**)&d_sum_re, blockSize.x * sizeof(float)));
//	HANDLE_ERROR(hipMalloc((void**)&d_sum_im, blockSize.x * sizeof(float)));
//	HANDLE_ERROR(hipMemset(d_sum_re, 0, blockSize.x * sizeof(float)));
//	HANDLE_ERROR(hipMemset(d_sum_im, 0, blockSize.x * sizeof(float)));
//
//	//ReduceAdd.
//	reduce_add_re << <blockSize, threadSize >> >(d_sum_re, d_reim, raysBeamNum);
//	hipDeviceSynchronize();
//	reduce_add_im << <blockSize, threadSize >> >(d_sum_im, d_reim, raysBeamNum);
//	hipDeviceSynchronize();
//	numItem = blockSize.x;
//	blockSize.x = numItem / threadSize.x + 1;
//
//	while (numItem > 1)
//	{
//		reduce_re << <blockSize, threadSize >> >(d_sum_re, d_sum_re, numItem);
//		hipDeviceSynchronize();
//		reduce_im << <blockSize, threadSize >> >(d_sum_im, d_sum_im, numItem);
//		hipDeviceSynchronize();
//		numItem = blockSize.x;
//		blockSize.x = numItem / threadSize.x + 1;
//	}
//
//	float sum_re = 0;
//	float sum_im = 0;
//	HANDLE_ERROR(hipMemcpy(&sum_re, d_sum_re, sizeof(float), hipMemcpyDeviceToHost));
//	HANDLE_ERROR(hipMemcpy(&sum_im, d_sum_im, sizeof(float), hipMemcpyDeviceToHost));
//	//printf("sum_re = %.10f, sum_im = %.10f\n", sum_re, sum_im);
//
//	//����ǿ�ȼ���
//	comp Reim;
//	comp sum = { sum_re, sum_im };
//	float mag = 0;
//
//	Reim.re = 0;
//	Reim.im = -wavenumber * 1000;
//	Reim.re = cosf(Reim.im);
//	Reim.im = sinf(Reim.im);
//	//printf("Reim.re=%f, Reim.im=%f\n",Reim.re, Reim.im);
//	sum = product(sum, Reim);
//	mag = sqrt(sum.re * sum.re + sum.im * sum.im);
//	//printf("mag = %f\n",mag);
//	if (mag < 99999999999.0)
//		intensity = 20 * log10f(mag * 1000 * 1000);
//	//printf("The sound field intensity is %f\n", intensity);
//
//	//hipFree(d_rays);
//	//hipFree(d_squares);
//	hipFree(d_effrays);
//	hipFree(d_reim);
//	hipFree(d_sum_re);
//	hipFree(d_sum_im);
//	//free(h_transMat);
//	//free(h_effrays);
//	//free(h_reim);
//
//	return intensity;
//}
